#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
// Copyright (c) 2010, Paul Furgale, Chi Hay Tong
//
// The original code was written by Paul Furgale and Chi Hay Tong
// and later optimized and prepared for integration into OpenCV by Itseez.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"
#include "opencv2/gpu/device/utility.hpp"
#include "opencv2/gpu/device/functional.hpp"
#include "opencv2/gpu/device/filters.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace surf
    {
        ////////////////////////////////////////////////////////////////////////
        // Global parameters

        // The maximum number of features (before subpixel interpolation) that memory is reserved for.
        __constant__ int c_max_candidates;
        // The maximum number of features that memory is reserved for.
        __constant__ int c_max_features;
        // The image size.
        __constant__ int c_img_rows;
        __constant__ int c_img_cols;
        // The number of layers.
        __constant__ int c_nOctaveLayers;
        // The hessian threshold.
        __constant__ float c_hessianThreshold;

        // The current octave.
        __constant__ int c_octave;
        // The current layer size.
        __constant__ int c_layer_rows;
        __constant__ int c_layer_cols;

        void loadGlobalConstants(int maxCandidates, int maxFeatures, int img_rows, int img_cols, int nOctaveLayers, float hessianThreshold)
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_max_candidates), &maxCandidates, sizeof(maxCandidates)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_max_features), &maxFeatures, sizeof(maxFeatures)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_img_rows), &img_rows, sizeof(img_rows)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_img_cols), &img_cols, sizeof(img_cols)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_nOctaveLayers), &nOctaveLayers, sizeof(nOctaveLayers)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_hessianThreshold), &hessianThreshold, sizeof(hessianThreshold)) );
        }

        void loadOctaveConstants(int octave, int layer_rows, int layer_cols)
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_octave), &octave, sizeof(octave)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_layer_rows), &layer_rows, sizeof(layer_rows)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_layer_cols), &layer_cols, sizeof(layer_cols)) );
        }

        ////////////////////////////////////////////////////////////////////////
        // Integral image texture

        texture<unsigned char, 2, hipReadModeElementType> imgTex(0, hipFilterModePoint, hipAddressModeClamp);
        texture<unsigned int, 2, hipReadModeElementType> sumTex(0, hipFilterModePoint, hipAddressModeClamp);
        texture<unsigned int, 2, hipReadModeElementType> maskSumTex(0, hipFilterModePoint, hipAddressModeClamp);

        void bindImgTex(DevMem2Db img)
        {
            bindTexture(&imgTex, img);
        }
        void bindSumTex(DevMem2D_<uint> sum)
        {
            bindTexture(&sumTex, sum);
        }
        void bindMaskSumTex(DevMem2D_<uint> maskSum)
        {
            bindTexture(&maskSumTex, maskSum);
        }

        template <int N> __device__ float icvCalcHaarPatternSum(const float src[][5], int oldSize, int newSize, int y, int x)
        {
        #if __CUDA_ARCH__ >= 200
            typedef double real_t;
        #else
            typedef float  real_t;
        #endif

            float ratio = (float)newSize / oldSize;

            real_t d = 0;

            #pragma unroll
            for (int k = 0; k < N; ++k)
            {
                int dx1 = __float2int_rn(ratio * src[k][0]);
                int dy1 = __float2int_rn(ratio * src[k][1]);
                int dx2 = __float2int_rn(ratio * src[k][2]);
                int dy2 = __float2int_rn(ratio * src[k][3]);

                real_t t = 0;
                t += tex2D(sumTex, x + dx1, y + dy1);
                t -= tex2D(sumTex, x + dx1, y + dy2);
                t -= tex2D(sumTex, x + dx2, y + dy1);
                t += tex2D(sumTex, x + dx2, y + dy2);

                d += t * src[k][4] / ((dx2 - dx1) * (dy2 - dy1));
            }

            return (float)d;
        }

        ////////////////////////////////////////////////////////////////////////
        // Hessian

        __constant__ float c_DX [3][5] = { {0, 2, 3, 7, 1}, {3, 2, 6, 7, -2}, {6, 2, 9, 7, 1} };
        __constant__ float c_DY [3][5] = { {2, 0, 7, 3, 1}, {2, 3, 7, 6, -2}, {2, 6, 7, 9, 1} };
        __constant__ float c_DXY[4][5] = { {1, 1, 4, 4, 1}, {5, 1, 8, 4, -1}, {1, 5, 4, 8, -1}, {5, 5, 8, 8, 1} };

        __host__ __device__ __forceinline__ int calcSize(int octave, int layer)
        {
            /* Wavelet size at first layer of first octave. */
            const int HAAR_SIZE0 = 9;

            /* Wavelet size increment between layers. This should be an even number,
             such that the wavelet sizes in an octave are either all even or all odd.
             This ensures that when looking for the neighbours of a sample, the layers
             above and below are aligned correctly. */
            const int HAAR_SIZE_INC = 6;

            return (HAAR_SIZE0 + HAAR_SIZE_INC * layer) << octave;
        }

        __global__ void icvCalcLayerDetAndTrace(PtrStepf det, PtrStepf trace)
        {
            // Determine the indices
            const int gridDim_y = gridDim.y / (c_nOctaveLayers + 2);
            const int blockIdx_y = blockIdx.y % gridDim_y;
            const int blockIdx_z = blockIdx.y / gridDim_y;

            const int j = threadIdx.x + blockIdx.x * blockDim.x;
            const int i = threadIdx.y + blockIdx_y * blockDim.y;
            const int layer = blockIdx_z;

            const int size = calcSize(c_octave, layer);

            const int samples_i = 1 + ((c_img_rows - size) >> c_octave);
            const int samples_j = 1 + ((c_img_cols - size) >> c_octave);

            // Ignore pixels where some of the kernel is outside the image
            const int margin = (size >> 1) >> c_octave;

            if (size <= c_img_rows && size <= c_img_cols && i < samples_i && j < samples_j)
            {
                const float dx  = icvCalcHaarPatternSum<3>(c_DX , 9, size, i << c_octave, j << c_octave);
                const float dy  = icvCalcHaarPatternSum<3>(c_DY , 9, size, i << c_octave, j << c_octave);
                const float dxy = icvCalcHaarPatternSum<4>(c_DXY, 9, size, i << c_octave, j << c_octave);

                det.ptr(layer * c_layer_rows + i + margin)[j + margin] = dx * dy - 0.81f * dxy * dxy;
                trace.ptr(layer * c_layer_rows + i + margin)[j + margin] = dx + dy;
            }
        }

        void icvCalcLayerDetAndTrace_gpu(const PtrStepf& det, const PtrStepf& trace, int img_rows, int img_cols, int octave, int nOctaveLayers)
        {
            const int min_size = calcSize(octave, 0);
            const int max_samples_i = 1 + ((img_rows - min_size) >> octave);
            const int max_samples_j = 1 + ((img_cols - min_size) >> octave);

            dim3 threads(16, 16);

            dim3 grid;
            grid.x = divUp(max_samples_j, threads.x);
            grid.y = divUp(max_samples_i, threads.y) * (nOctaveLayers + 2);

            icvCalcLayerDetAndTrace<<<grid, threads>>>(det, trace);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////
        // NONMAX

        __constant__ float c_DM[5] = {0, 0, 9, 9, 1};

        struct WithMask
        {
            static __device__ bool check(int sum_i, int sum_j, int size)
            {
                float ratio = (float)size / 9.0f;

                float d = 0;

                int dx1 = __float2int_rn(ratio * c_DM[0]);
                int dy1 = __float2int_rn(ratio * c_DM[1]);
                int dx2 = __float2int_rn(ratio * c_DM[2]);
                int dy2 = __float2int_rn(ratio * c_DM[3]);

                float t = 0;
                t += tex2D(maskSumTex, sum_j + dx1, sum_i + dy1);
                t -= tex2D(maskSumTex, sum_j + dx1, sum_i + dy2);
                t -= tex2D(maskSumTex, sum_j + dx2, sum_i + dy1);
                t += tex2D(maskSumTex, sum_j + dx2, sum_i + dy2);

                d += t * c_DM[4] / ((dx2 - dx1) * (dy2 - dy1));

                return (d >= 0.5f);
            }
        };

        template <typename Mask>
        __global__ void icvFindMaximaInLayer(const PtrStepf det, const PtrStepf trace, int4* maxPosBuffer, unsigned int* maxCounter)
        {
            #if __CUDA_ARCH__ >= 110

            extern __shared__ float N9[];

            // The hidx variables are the indices to the hessian buffer.
            const int gridDim_y = gridDim.y / c_nOctaveLayers;
            const int blockIdx_y = blockIdx.y % gridDim_y;
            const int blockIdx_z = blockIdx.y / gridDim_y;

            const int layer = blockIdx_z + 1;

            const int size = calcSize(c_octave, layer);

            // Ignore pixels without a 3x3x3 neighbourhood in the layer above
            const int margin = ((calcSize(c_octave, layer + 1) >> 1) >> c_octave) + 1;

            const int j = threadIdx.x + blockIdx.x * (blockDim.x - 2) + margin - 1;
            const int i = threadIdx.y + blockIdx_y * (blockDim.y - 2) + margin - 1;

            // Is this thread within the hessian buffer?
            const int zoff = blockDim.x * blockDim.y;
            const int localLin = threadIdx.x + threadIdx.y * blockDim.x + zoff;
            N9[localLin - zoff] = det.ptr(c_layer_rows * (layer - 1) + ::min(::max(i, 0), c_img_rows - 1))[::min(::max(j, 0), c_img_cols - 1)];
            N9[localLin       ] = det.ptr(c_layer_rows * (layer    ) + ::min(::max(i, 0), c_img_rows - 1))[::min(::max(j, 0), c_img_cols - 1)];
            N9[localLin + zoff] = det.ptr(c_layer_rows * (layer + 1) + ::min(::max(i, 0), c_img_rows - 1))[::min(::max(j, 0), c_img_cols - 1)];
            __syncthreads();

            if (i < c_layer_rows - margin && j < c_layer_cols - margin && threadIdx.x > 0 && threadIdx.x < blockDim.x - 1 && threadIdx.y > 0 && threadIdx.y < blockDim.y - 1)
            {
                float val0 = N9[localLin];

                if (val0 > c_hessianThreshold)
                {
                    // Coordinates for the start of the wavelet in the sum image. There
                    // is some integer division involved, so don't try to simplify this
                    // (cancel out sampleStep) without checking the result is the same
                    const int sum_i = (i - ((size >> 1) >> c_octave)) << c_octave;
                    const int sum_j = (j - ((size >> 1) >> c_octave)) << c_octave;

                    if (Mask::check(sum_i, sum_j, size))
                    {
                        // Check to see if we have a max (in its 26 neighbours)
                        const bool condmax = val0 > N9[localLin - 1 - blockDim.x - zoff]
                        &&                   val0 > N9[localLin     - blockDim.x - zoff]
                        &&                   val0 > N9[localLin + 1 - blockDim.x - zoff]
                        &&                   val0 > N9[localLin - 1              - zoff]
                        &&                   val0 > N9[localLin                  - zoff]
                        &&                   val0 > N9[localLin + 1              - zoff]
                        &&                   val0 > N9[localLin - 1 + blockDim.x - zoff]
                        &&                   val0 > N9[localLin     + blockDim.x - zoff]
                        &&                   val0 > N9[localLin + 1 + blockDim.x - zoff]

                        &&                   val0 > N9[localLin - 1 - blockDim.x]
                        &&                   val0 > N9[localLin     - blockDim.x]
                        &&                   val0 > N9[localLin + 1 - blockDim.x]
                        &&                   val0 > N9[localLin - 1             ]
                        &&                   val0 > N9[localLin + 1             ]
                        &&                   val0 > N9[localLin - 1 + blockDim.x]
                        &&                   val0 > N9[localLin     + blockDim.x]
                        &&                   val0 > N9[localLin + 1 + blockDim.x]

                        &&                   val0 > N9[localLin - 1 - blockDim.x + zoff]
                        &&                   val0 > N9[localLin     - blockDim.x + zoff]
                        &&                   val0 > N9[localLin + 1 - blockDim.x + zoff]
                        &&                   val0 > N9[localLin - 1              + zoff]
                        &&                   val0 > N9[localLin                  + zoff]
                        &&                   val0 > N9[localLin + 1              + zoff]
                        &&                   val0 > N9[localLin - 1 + blockDim.x + zoff]
                        &&                   val0 > N9[localLin     + blockDim.x + zoff]
                        &&                   val0 > N9[localLin + 1 + blockDim.x + zoff]
                        ;

                        if(condmax)
                        {
                            unsigned int ind = atomicInc(maxCounter,(unsigned int) -1);

                            if (ind < c_max_candidates)
                            {
                                const int laplacian = (int) copysignf(1.0f, trace.ptr(layer * c_layer_rows + i)[j]);

                                maxPosBuffer[ind] = make_int4(j, i, layer, laplacian);
                            }
                        }
                    }
                }
            }

            #endif
        }

        void icvFindMaximaInLayer_gpu(const PtrStepf& det, const PtrStepf& trace, int4* maxPosBuffer, unsigned int* maxCounter,
            int img_rows, int img_cols, int octave, bool use_mask, int nOctaveLayers)
        {
            const int layer_rows = img_rows >> octave;
            const int layer_cols = img_cols >> octave;

            const int min_margin = ((calcSize(octave, 2) >> 1) >> octave) + 1;

            dim3 threads(16, 16);

            dim3 grid;
            grid.x = divUp(layer_cols - 2 * min_margin, threads.x - 2);
            grid.y = divUp(layer_rows - 2 * min_margin, threads.y - 2) * nOctaveLayers;

            const size_t smem_size = threads.x * threads.y * 3 * sizeof(float);

            if (use_mask)
                icvFindMaximaInLayer<WithMask><<<grid, threads, smem_size>>>(det, trace, maxPosBuffer, maxCounter);
            else
                icvFindMaximaInLayer<WithOutMask><<<grid, threads, smem_size>>>(det, trace, maxPosBuffer, maxCounter);

            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////
        // INTERPOLATION

        __global__ void icvInterpolateKeypoint(const PtrStepf det, const int4* maxPosBuffer,
            float* featureX, float* featureY, int* featureLaplacian, int* featureOctave, float* featureSize, float* featureHessian,
            unsigned int* featureCounter)
        {
            #if __CUDA_ARCH__ >= 110

            const int4 maxPos = maxPosBuffer[blockIdx.x];

            const int j = maxPos.x - 1 + threadIdx.x;
            const int i = maxPos.y - 1 + threadIdx.y;
            const int layer = maxPos.z - 1 + threadIdx.z;

            __shared__ float N9[3][3][3];

            N9[threadIdx.z][threadIdx.y][threadIdx.x] = det.ptr(c_layer_rows * layer + i)[j];
            __syncthreads();

            if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
            {
                __shared__ float dD[3];

                //dx
                dD[0] = -0.5f * (N9[1][1][2] - N9[1][1][0]);
                //dy
                dD[1] = -0.5f * (N9[1][2][1] - N9[1][0][1]);
                //ds
                dD[2] = -0.5f * (N9[2][1][1] - N9[0][1][1]);

                __shared__ float H[3][3];

                //dxx
                H[0][0] = N9[1][1][0] - 2.0f * N9[1][1][1] + N9[1][1][2];
                //dxy
                H[0][1]= 0.25f * (N9[1][2][2] - N9[1][2][0] - N9[1][0][2] + N9[1][0][0]);
                //dxs
                H[0][2]= 0.25f * (N9[2][1][2] - N9[2][1][0] - N9[0][1][2] + N9[0][1][0]);
                //dyx = dxy
                H[1][0] = H[0][1];
                //dyy
                H[1][1] = N9[1][0][1] - 2.0f * N9[1][1][1] + N9[1][2][1];
                //dys
                H[1][2]= 0.25f * (N9[2][2][1] - N9[2][0][1] - N9[0][2][1] + N9[0][0][1]);
                //dsx = dxs
                H[2][0] = H[0][2];
                //dsy = dys
                H[2][1] = H[1][2];
                //dss
                H[2][2] = N9[0][1][1] - 2.0f * N9[1][1][1] + N9[2][1][1];

                __shared__ float x[3];

                if (solve3x3(H, dD, x))
                {
                    if (::fabs(x[0]) <= 1.f && ::fabs(x[1]) <= 1.f && ::fabs(x[2]) <= 1.f)
                    {
                        // if the step is within the interpolation region, perform it

                        const int size = calcSize(c_octave, maxPos.z);

                        const int sum_i = (maxPos.y - ((size >> 1) >> c_octave)) << c_octave;
                        const int sum_j = (maxPos.x - ((size >> 1) >> c_octave)) << c_octave;

                        const float center_i = sum_i + (float)(size - 1) / 2;
                        const float center_j = sum_j + (float)(size - 1) / 2;

                        const float px = center_j + x[0] * (1 << c_octave);
                        const float py = center_i + x[1] * (1 << c_octave);

                        const int ds = size - calcSize(c_octave, maxPos.z - 1);
                        const float psize = roundf(size + x[2] * ds);

                        /* The sampling intervals and wavelet sized for selecting an orientation
                         and building the keypoint descriptor are defined relative to 's' */
                        const float s = psize * 1.2f / 9.0f;

                        /* To find the dominant orientation, the gradients in x and y are
                         sampled in a circle of radius 6s using wavelets of size 4s.
                         We ensure the gradient wavelet size is even to ensure the
                         wavelet pattern is balanced and symmetric around its center */
                        const int grad_wav_size = 2 * __float2int_rn(2.0f * s);

                        // check when grad_wav_size is too big
                        if ((c_img_rows + 1) >= grad_wav_size && (c_img_cols + 1) >= grad_wav_size)
                        {
                            // Get a new feature index.
                            unsigned int ind = atomicInc(featureCounter, (unsigned int)-1);

                            if (ind < c_max_features)
                            {
                                featureX[ind] = px;
                                featureY[ind] = py;
                                featureLaplacian[ind] = maxPos.w;
                                featureOctave[ind] = c_octave;
                                featureSize[ind] = psize;
                                featureHessian[ind] = N9[1][1][1];
                            }
                        } // grad_wav_size check
                    } // If the subpixel interpolation worked
                }
            } // If this is thread 0.

            #endif
        }

        void icvInterpolateKeypoint_gpu(const PtrStepf& det, const int4* maxPosBuffer, unsigned int maxCounter,
            float* featureX, float* featureY, int* featureLaplacian, int* featureOctave, float* featureSize, float* featureHessian,
            unsigned int* featureCounter)
        {
            dim3 threads;
            threads.x = 3;
            threads.y = 3;
            threads.z = 3;

            dim3 grid;
            grid.x = maxCounter;

            icvInterpolateKeypoint<<<grid, threads>>>(det, maxPosBuffer, featureX, featureY, featureLaplacian, featureOctave, featureSize, featureHessian, featureCounter);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////
        // Orientation

        #define ORI_SEARCH_INC 5
        #define ORI_WIN        60
        #define ORI_SAMPLES    113

        __constant__ float c_aptX[ORI_SAMPLES] = {-6, -5, -5, -5, -5, -5, -5, -5, -4, -4, -4, -4, -4, -4, -4, -4, -4, -3, -3, -3, -3, -3, -3, -3, -3, -3, -3, -3, -2, -2, -2, -2, -2, -2, -2, -2, -2, -2, -2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 4, 4, 4, 4, 4, 4, 4, 4, 4, 5, 5, 5, 5, 5, 5, 5, 6};
        __constant__ float c_aptY[ORI_SAMPLES] = {0, -3, -2, -1, 0, 1, 2, 3, -4, -3, -2, -1, 0, 1, 2, 3, 4, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, -6, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, 6, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, -4, -3, -2, -1, 0, 1, 2, 3, 4, -3, -2, -1, 0, 1, 2, 3, 0};
        __constant__ float c_aptW[ORI_SAMPLES] = {0.001455130288377404f, 0.001707611023448408f, 0.002547456417232752f, 0.003238451667129993f, 0.0035081731621176f, 0.003238451667129993f, 0.002547456417232752f, 0.001707611023448408f, 0.002003900473937392f, 0.0035081731621176f, 0.005233579315245152f, 0.00665318313986063f, 0.00720730796456337f, 0.00665318313986063f, 0.005233579315245152f, 0.0035081731621176f, 0.002003900473937392f, 0.001707611023448408f, 0.0035081731621176f, 0.006141661666333675f, 0.009162282571196556f, 0.01164754293859005f, 0.01261763460934162f, 0.01164754293859005f, 0.009162282571196556f, 0.006141661666333675f, 0.0035081731621176f, 0.001707611023448408f, 0.002547456417232752f, 0.005233579315245152f, 0.009162282571196556f, 0.01366852037608624f, 0.01737609319388866f, 0.0188232995569706f, 0.01737609319388866f, 0.01366852037608624f, 0.009162282571196556f, 0.005233579315245152f, 0.002547456417232752f, 0.003238451667129993f, 0.00665318313986063f, 0.01164754293859005f, 0.01737609319388866f, 0.02208934165537357f, 0.02392910048365593f, 0.02208934165537357f, 0.01737609319388866f, 0.01164754293859005f, 0.00665318313986063f, 0.003238451667129993f, 0.001455130288377404f, 0.0035081731621176f, 0.00720730796456337f, 0.01261763460934162f, 0.0188232995569706f, 0.02392910048365593f, 0.02592208795249462f, 0.02392910048365593f, 0.0188232995569706f, 0.01261763460934162f, 0.00720730796456337f, 0.0035081731621176f, 0.001455130288377404f, 0.003238451667129993f, 0.00665318313986063f, 0.01164754293859005f, 0.01737609319388866f, 0.02208934165537357f, 0.02392910048365593f, 0.02208934165537357f, 0.01737609319388866f, 0.01164754293859005f, 0.00665318313986063f, 0.003238451667129993f, 0.002547456417232752f, 0.005233579315245152f, 0.009162282571196556f, 0.01366852037608624f, 0.01737609319388866f, 0.0188232995569706f, 0.01737609319388866f, 0.01366852037608624f, 0.009162282571196556f, 0.005233579315245152f, 0.002547456417232752f, 0.001707611023448408f, 0.0035081731621176f, 0.006141661666333675f, 0.009162282571196556f, 0.01164754293859005f, 0.01261763460934162f, 0.01164754293859005f, 0.009162282571196556f, 0.006141661666333675f, 0.0035081731621176f, 0.001707611023448408f, 0.002003900473937392f, 0.0035081731621176f, 0.005233579315245152f, 0.00665318313986063f, 0.00720730796456337f, 0.00665318313986063f, 0.005233579315245152f, 0.0035081731621176f, 0.002003900473937392f, 0.001707611023448408f, 0.002547456417232752f, 0.003238451667129993f, 0.0035081731621176f, 0.003238451667129993f, 0.002547456417232752f, 0.001707611023448408f, 0.001455130288377404f};

        __constant__ float c_NX[2][5] = {{0, 0, 2, 4, -1}, {2, 0, 4, 4, 1}};
        __constant__ float c_NY[2][5] = {{0, 0, 4, 2, 1}, {0, 2, 4, 4, -1}};

        __global__ void icvCalcOrientation(const float* featureX, const float* featureY, const float* featureSize, float* featureDir)
        {
            __shared__ float s_X[128];
            __shared__ float s_Y[128];
            __shared__ float s_angle[128];

            __shared__ float s_sumx[32 * 4];
            __shared__ float s_sumy[32 * 4];

            /* The sampling intervals and wavelet sized for selecting an orientation
             and building the keypoint descriptor are defined relative to 's' */
            const float s = featureSize[blockIdx.x] * 1.2f / 9.0f;

            /* To find the dominant orientation, the gradients in x and y are
             sampled in a circle of radius 6s using wavelets of size 4s.
             We ensure the gradient wavelet size is even to ensure the
             wavelet pattern is balanced and symmetric around its center */
            const int grad_wav_size = 2 * __float2int_rn(2.0f * s);

            // check when grad_wav_size is too big
            if ((c_img_rows + 1) < grad_wav_size || (c_img_cols + 1) < grad_wav_size)
                return;

            // Calc X, Y, angle and store it to shared memory
            const int tid = threadIdx.y * blockDim.x + threadIdx.x;

            float X = 0.0f, Y = 0.0f, angle = 0.0f;

            if (tid < ORI_SAMPLES)
            {
                const float margin = (float)(grad_wav_size - 1) / 2.0f;
                const int x = __float2int_rn(featureX[blockIdx.x] + c_aptX[tid] * s - margin);
                const int y = __float2int_rn(featureY[blockIdx.x] + c_aptY[tid] * s - margin);

                if (y >= 0 && y < (c_img_rows + 1) - grad_wav_size &&
                    x >= 0 && x < (c_img_cols + 1) - grad_wav_size)
                {
                    X = c_aptW[tid] * icvCalcHaarPatternSum<2>(c_NX, 4, grad_wav_size, y, x);
                    Y = c_aptW[tid] * icvCalcHaarPatternSum<2>(c_NY, 4, grad_wav_size, y, x);

                    angle = atan2f(Y, X);
                    if (angle < 0)
                        angle += 2.0f * CV_PI_F;
                    angle *= 180.0f / CV_PI_F;
                }
            }
            s_X[tid] = X;
            s_Y[tid] = Y;
            s_angle[tid] = angle;
            __syncthreads();

            float bestx = 0, besty = 0, best_mod = 0;

            #pragma unroll
            for (int i = 0; i < 18; ++i)
            {
                const int dir = (i * 4 + threadIdx.y) * ORI_SEARCH_INC;

                float sumx = 0.0f, sumy = 0.0f;
                int d = ::abs(__float2int_rn(s_angle[threadIdx.x]) - dir);
                if (d < ORI_WIN / 2 || d > 360 - ORI_WIN / 2)
                {
                    sumx = s_X[threadIdx.x];
                    sumy = s_Y[threadIdx.x];
                }
                d = ::abs(__float2int_rn(s_angle[threadIdx.x + 32]) - dir);
                if (d < ORI_WIN / 2 || d > 360 - ORI_WIN / 2)
                {
                    sumx += s_X[threadIdx.x + 32];
                    sumy += s_Y[threadIdx.x + 32];
                }
                d = ::abs(__float2int_rn(s_angle[threadIdx.x + 64]) - dir);
                if (d < ORI_WIN / 2 || d > 360 - ORI_WIN / 2)
                {
                    sumx += s_X[threadIdx.x + 64];
                    sumy += s_Y[threadIdx.x + 64];
                }
                d = ::abs(__float2int_rn(s_angle[threadIdx.x + 96]) - dir);
                if (d < ORI_WIN / 2 || d > 360 - ORI_WIN / 2)
                {
                    sumx += s_X[threadIdx.x + 96];
                    sumy += s_Y[threadIdx.x + 96];
                }

                device::reduce<32>(s_sumx + threadIdx.y * 32, sumx, threadIdx.x, plus<volatile float>());
                device::reduce<32>(s_sumy + threadIdx.y * 32, sumy, threadIdx.x, plus<volatile float>());

                const float temp_mod = sumx * sumx + sumy * sumy;
                if (temp_mod > best_mod)
                {
                    best_mod = temp_mod;
                    bestx = sumx;
                    besty = sumy;
                }

                __syncthreads();
            }

            if (threadIdx.x == 0)
            {
                s_X[threadIdx.y] = bestx;
                s_Y[threadIdx.y] = besty;
                s_angle[threadIdx.y] = best_mod;
            }
            __syncthreads();

            if (threadIdx.x == 0 && threadIdx.y == 0)
            {
                int bestIdx = 0;

                if (s_angle[1] > s_angle[bestIdx])
                    bestIdx = 1;
                if (s_angle[2] > s_angle[bestIdx])
                    bestIdx = 2;
                if (s_angle[3] > s_angle[bestIdx])
                    bestIdx = 3;

                float kp_dir = atan2f(s_Y[bestIdx], s_X[bestIdx]);
                if (kp_dir < 0)
                    kp_dir += 2.0f * CV_PI_F;
                kp_dir *= 180.0f / CV_PI_F;

                featureDir[blockIdx.x] = kp_dir;
            }
        }

        #undef ORI_SEARCH_INC
        #undef ORI_WIN
        #undef ORI_SAMPLES

        void icvCalcOrientation_gpu(const float* featureX, const float* featureY, const float* featureSize, float* featureDir, int nFeatures)
        {
            dim3 threads;
            threads.x = 32;
            threads.y = 4;

            dim3 grid;
            grid.x = nFeatures;

            icvCalcOrientation<<<grid, threads>>>(featureX, featureY, featureSize, featureDir);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////
        // Descriptors

        #define PATCH_SZ 20

        __constant__ float c_DW[PATCH_SZ * PATCH_SZ] =
        {
            3.695352233989979e-006f, 8.444558261544444e-006f, 1.760426494001877e-005f, 3.34794785885606e-005f, 5.808438800158911e-005f, 9.193058212986216e-005f, 0.0001327334757661447f, 0.0001748319627949968f, 0.0002100782439811155f, 0.0002302826324012131f, 0.0002302826324012131f, 0.0002100782439811155f, 0.0001748319627949968f, 0.0001327334757661447f, 9.193058212986216e-005f, 5.808438800158911e-005f, 3.34794785885606e-005f, 1.760426494001877e-005f, 8.444558261544444e-006f, 3.695352233989979e-006f,
            8.444558261544444e-006f, 1.929736572492402e-005f, 4.022897701361217e-005f, 7.650675252079964e-005f, 0.0001327334903180599f, 0.0002100782585330308f, 0.0003033203829545528f, 0.0003995231236331165f, 0.0004800673632416874f, 0.0005262381164357066f, 0.0005262381164357066f, 0.0004800673632416874f, 0.0003995231236331165f, 0.0003033203829545528f, 0.0002100782585330308f, 0.0001327334903180599f, 7.650675252079964e-005f, 4.022897701361217e-005f, 1.929736572492402e-005f, 8.444558261544444e-006f,
            1.760426494001877e-005f, 4.022897701361217e-005f, 8.386484114453197e-005f, 0.0001594926579855382f, 0.0002767078403849155f, 0.0004379475140012801f, 0.0006323281559161842f, 0.0008328808471560478f, 0.001000790391117334f, 0.001097041997127235f, 0.001097041997127235f, 0.001000790391117334f, 0.0008328808471560478f, 0.0006323281559161842f, 0.0004379475140012801f, 0.0002767078403849155f, 0.0001594926579855382f, 8.386484114453197e-005f, 4.022897701361217e-005f, 1.760426494001877e-005f,
            3.34794785885606e-005f, 7.650675252079964e-005f, 0.0001594926579855382f, 0.0003033203247468919f, 0.0005262380582280457f, 0.0008328807889483869f, 0.001202550483867526f, 0.001583957928232849f, 0.001903285388834775f, 0.002086334861814976f, 0.002086334861814976f, 0.001903285388834775f, 0.001583957928232849f, 0.001202550483867526f, 0.0008328807889483869f, 0.0005262380582280457f, 0.0003033203247468919f, 0.0001594926579855382f, 7.650675252079964e-005f, 3.34794785885606e-005f,
            5.808438800158911e-005f, 0.0001327334903180599f, 0.0002767078403849155f, 0.0005262380582280457f, 0.0009129836107604206f, 0.001444985857233405f, 0.002086335094645619f, 0.002748048631474376f, 0.00330205773934722f, 0.003619635012000799f, 0.003619635012000799f, 0.00330205773934722f, 0.002748048631474376f, 0.002086335094645619f, 0.001444985857233405f, 0.0009129836107604206f, 0.0005262380582280457f, 0.0002767078403849155f, 0.0001327334903180599f, 5.808438800158911e-005f,
            9.193058212986216e-005f, 0.0002100782585330308f, 0.0004379475140012801f, 0.0008328807889483869f, 0.001444985857233405f, 0.002286989474669099f, 0.00330205773934722f, 0.004349356517195702f, 0.00522619066759944f, 0.005728822201490402f, 0.005728822201490402f, 0.00522619066759944f, 0.004349356517195702f, 0.00330205773934722f, 0.002286989474669099f, 0.001444985857233405f, 0.0008328807889483869f, 0.0004379475140012801f, 0.0002100782585330308f, 9.193058212986216e-005f,
            0.0001327334757661447f, 0.0003033203829545528f, 0.0006323281559161842f, 0.001202550483867526f, 0.002086335094645619f, 0.00330205773934722f, 0.004767658654600382f, 0.006279794964939356f, 0.007545807864516974f, 0.008271530270576477f, 0.008271530270576477f, 0.007545807864516974f, 0.006279794964939356f, 0.004767658654600382f, 0.00330205773934722f, 0.002086335094645619f, 0.001202550483867526f, 0.0006323281559161842f, 0.0003033203829545528f, 0.0001327334757661447f,
            0.0001748319627949968f, 0.0003995231236331165f, 0.0008328808471560478f, 0.001583957928232849f, 0.002748048631474376f, 0.004349356517195702f, 0.006279794964939356f, 0.008271529339253902f, 0.009939077310264111f, 0.01089497376233339f, 0.01089497376233339f, 0.009939077310264111f, 0.008271529339253902f, 0.006279794964939356f, 0.004349356517195702f, 0.002748048631474376f, 0.001583957928232849f, 0.0008328808471560478f, 0.0003995231236331165f, 0.0001748319627949968f,
            0.0002100782439811155f, 0.0004800673632416874f, 0.001000790391117334f, 0.001903285388834775f, 0.00330205773934722f, 0.00522619066759944f, 0.007545807864516974f, 0.009939077310264111f, 0.01194280479103327f, 0.01309141051024199f, 0.01309141051024199f, 0.01194280479103327f, 0.009939077310264111f, 0.007545807864516974f, 0.00522619066759944f, 0.00330205773934722f, 0.001903285388834775f, 0.001000790391117334f, 0.0004800673632416874f, 0.0002100782439811155f,
            0.0002302826324012131f, 0.0005262381164357066f, 0.001097041997127235f, 0.002086334861814976f, 0.003619635012000799f, 0.005728822201490402f, 0.008271530270576477f, 0.01089497376233339f, 0.01309141051024199f, 0.01435048412531614f, 0.01435048412531614f, 0.01309141051024199f, 0.01089497376233339f, 0.008271530270576477f, 0.005728822201490402f, 0.003619635012000799f, 0.002086334861814976f, 0.001097041997127235f, 0.0005262381164357066f, 0.0002302826324012131f,
            0.0002302826324012131f, 0.0005262381164357066f, 0.001097041997127235f, 0.002086334861814976f, 0.003619635012000799f, 0.005728822201490402f, 0.008271530270576477f, 0.01089497376233339f, 0.01309141051024199f, 0.01435048412531614f, 0.01435048412531614f, 0.01309141051024199f, 0.01089497376233339f, 0.008271530270576477f, 0.005728822201490402f, 0.003619635012000799f, 0.002086334861814976f, 0.001097041997127235f, 0.0005262381164357066f, 0.0002302826324012131f,
            0.0002100782439811155f, 0.0004800673632416874f, 0.001000790391117334f, 0.001903285388834775f, 0.00330205773934722f, 0.00522619066759944f, 0.007545807864516974f, 0.009939077310264111f, 0.01194280479103327f, 0.01309141051024199f, 0.01309141051024199f, 0.01194280479103327f, 0.009939077310264111f, 0.007545807864516974f, 0.00522619066759944f, 0.00330205773934722f, 0.001903285388834775f, 0.001000790391117334f, 0.0004800673632416874f, 0.0002100782439811155f,
            0.0001748319627949968f, 0.0003995231236331165f, 0.0008328808471560478f, 0.001583957928232849f, 0.002748048631474376f, 0.004349356517195702f, 0.006279794964939356f, 0.008271529339253902f, 0.009939077310264111f, 0.01089497376233339f, 0.01089497376233339f, 0.009939077310264111f, 0.008271529339253902f, 0.006279794964939356f, 0.004349356517195702f, 0.002748048631474376f, 0.001583957928232849f, 0.0008328808471560478f, 0.0003995231236331165f, 0.0001748319627949968f,
            0.0001327334757661447f, 0.0003033203829545528f, 0.0006323281559161842f, 0.001202550483867526f, 0.002086335094645619f, 0.00330205773934722f, 0.004767658654600382f, 0.006279794964939356f, 0.007545807864516974f, 0.008271530270576477f, 0.008271530270576477f, 0.007545807864516974f, 0.006279794964939356f, 0.004767658654600382f, 0.00330205773934722f, 0.002086335094645619f, 0.001202550483867526f, 0.0006323281559161842f, 0.0003033203829545528f, 0.0001327334757661447f,
            9.193058212986216e-005f, 0.0002100782585330308f, 0.0004379475140012801f, 0.0008328807889483869f, 0.001444985857233405f, 0.002286989474669099f, 0.00330205773934722f, 0.004349356517195702f, 0.00522619066759944f, 0.005728822201490402f, 0.005728822201490402f, 0.00522619066759944f, 0.004349356517195702f, 0.00330205773934722f, 0.002286989474669099f, 0.001444985857233405f, 0.0008328807889483869f, 0.0004379475140012801f, 0.0002100782585330308f, 9.193058212986216e-005f,
            5.808438800158911e-005f, 0.0001327334903180599f, 0.0002767078403849155f, 0.0005262380582280457f, 0.0009129836107604206f, 0.001444985857233405f, 0.002086335094645619f, 0.002748048631474376f, 0.00330205773934722f, 0.003619635012000799f, 0.003619635012000799f, 0.00330205773934722f, 0.002748048631474376f, 0.002086335094645619f, 0.001444985857233405f, 0.0009129836107604206f, 0.0005262380582280457f, 0.0002767078403849155f, 0.0001327334903180599f, 5.808438800158911e-005f,
            3.34794785885606e-005f, 7.650675252079964e-005f, 0.0001594926579855382f, 0.0003033203247468919f, 0.0005262380582280457f, 0.0008328807889483869f, 0.001202550483867526f, 0.001583957928232849f, 0.001903285388834775f, 0.002086334861814976f, 0.002086334861814976f, 0.001903285388834775f, 0.001583957928232849f, 0.001202550483867526f, 0.0008328807889483869f, 0.0005262380582280457f, 0.0003033203247468919f, 0.0001594926579855382f, 7.650675252079964e-005f, 3.34794785885606e-005f,
            1.760426494001877e-005f, 4.022897701361217e-005f, 8.386484114453197e-005f, 0.0001594926579855382f, 0.0002767078403849155f, 0.0004379475140012801f, 0.0006323281559161842f, 0.0008328808471560478f, 0.001000790391117334f, 0.001097041997127235f, 0.001097041997127235f, 0.001000790391117334f, 0.0008328808471560478f, 0.0006323281559161842f, 0.0004379475140012801f, 0.0002767078403849155f, 0.0001594926579855382f, 8.386484114453197e-005f, 4.022897701361217e-005f, 1.760426494001877e-005f,
            8.444558261544444e-006f, 1.929736572492402e-005f, 4.022897701361217e-005f, 7.650675252079964e-005f, 0.0001327334903180599f, 0.0002100782585330308f, 0.0003033203829545528f, 0.0003995231236331165f, 0.0004800673632416874f, 0.0005262381164357066f, 0.0005262381164357066f, 0.0004800673632416874f, 0.0003995231236331165f, 0.0003033203829545528f, 0.0002100782585330308f, 0.0001327334903180599f, 7.650675252079964e-005f, 4.022897701361217e-005f, 1.929736572492402e-005f, 8.444558261544444e-006f,
            3.695352233989979e-006f, 8.444558261544444e-006f, 1.760426494001877e-005f, 3.34794785885606e-005f, 5.808438800158911e-005f, 9.193058212986216e-005f, 0.0001327334757661447f, 0.0001748319627949968f, 0.0002100782439811155f, 0.0002302826324012131f, 0.0002302826324012131f, 0.0002100782439811155f, 0.0001748319627949968f, 0.0001327334757661447f, 9.193058212986216e-005f, 5.808438800158911e-005f, 3.34794785885606e-005f, 1.760426494001877e-005f, 8.444558261544444e-006f, 3.695352233989979e-006f
        };

        struct WinReader
        {
            typedef uchar elem_type;

            __device__ __forceinline__ WinReader(float centerX_, float centerY_, float win_offset_, float cos_dir_, float sin_dir_) :
                centerX(centerX_), centerY(centerY_), win_offset(win_offset_), cos_dir(cos_dir_), sin_dir(sin_dir_)
            {
            }

            __device__ __forceinline__ uchar operator ()(int i, int j) const
            {
                float pixel_x = centerX + (win_offset + j) * cos_dir + (win_offset + i) * sin_dir;
                float pixel_y = centerY - (win_offset + j) * sin_dir + (win_offset + i) * cos_dir;

                return tex2D(imgTex, pixel_x, pixel_y);
            }

            float centerX;
            float centerY;
            float win_offset;
            float cos_dir;
            float sin_dir;
        };

        __device__ void calc_dx_dy(float s_dx_bin[25], float s_dy_bin[25],
            const float* featureX, const float* featureY, const float* featureSize, const float* featureDir)
        {
            __shared__ float s_PATCH[6][6];

            const float centerX = featureX[blockIdx.x];
            const float centerY = featureY[blockIdx.x];
            const float size = featureSize[blockIdx.x];
            const float descriptor_dir = featureDir[blockIdx.x] * (float)(CV_PI_F / 180.0f);

            /* The sampling intervals and wavelet sized for selecting an orientation
             and building the keypoint descriptor are defined relative to 's' */
            const float s = size * 1.2f / 9.0f;

            /* Extract a window of pixels around the keypoint of size 20s */
            const int win_size = (int)((PATCH_SZ + 1) * s);

            float sin_dir;
            float cos_dir;
            sincosf(descriptor_dir, &sin_dir, &cos_dir);

            /* Nearest neighbour version (faster) */
            const float win_offset = -(float)(win_size - 1) / 2;

            // Compute sampling points
            // since grids are 2D, need to compute xBlock and yBlock indices
            const int xBlock = (blockIdx.y & 3);  // blockIdx.y % 4
            const int yBlock = (blockIdx.y >> 2); // floor(blockIdx.y/4)
            const int xIndex = xBlock * 5 + threadIdx.x;
            const int yIndex = yBlock * 5 + threadIdx.y;

            const float icoo = ((float)yIndex / (PATCH_SZ + 1)) * win_size;
            const float jcoo = ((float)xIndex / (PATCH_SZ + 1)) * win_size;

            LinearFilter<WinReader> filter(WinReader(centerX, centerY, win_offset, cos_dir, sin_dir));

            s_PATCH[threadIdx.y][threadIdx.x] = filter(icoo, jcoo);

            __syncthreads();

            if (threadIdx.x < 5 && threadIdx.y < 5)
            {
                const int tid = threadIdx.y * 5 + threadIdx.x;

                const float dw = c_DW[yIndex * PATCH_SZ + xIndex];

                const float vx = (s_PATCH[threadIdx.y    ][threadIdx.x + 1] - s_PATCH[threadIdx.y][threadIdx.x] + s_PATCH[threadIdx.y + 1][threadIdx.x + 1] - s_PATCH[threadIdx.y + 1][threadIdx.x    ]) * dw;
                const float vy = (s_PATCH[threadIdx.y + 1][threadIdx.x    ] - s_PATCH[threadIdx.y][threadIdx.x] + s_PATCH[threadIdx.y + 1][threadIdx.x + 1] - s_PATCH[threadIdx.y    ][threadIdx.x + 1]) * dw;

                s_dx_bin[tid] = vx;
                s_dy_bin[tid] = vy;
            }
        }

        __device__ void reduce_sum25(volatile float* sdata1, volatile float* sdata2, volatile float* sdata3, volatile float* sdata4, int tid)
        {
            // first step is to reduce from 25 to 16
            if (tid < 9) // use 9 threads
            {
                sdata1[tid] += sdata1[tid + 16];
                sdata2[tid] += sdata2[tid + 16];
                sdata3[tid] += sdata3[tid + 16];
                sdata4[tid] += sdata4[tid + 16];
            }

            // sum (reduce) from 16 to 1 (unrolled - aligned to a half-warp)
            if (tid < 8)
            {
                sdata1[tid] += sdata1[tid + 8];
                sdata1[tid] += sdata1[tid + 4];
                sdata1[tid] += sdata1[tid + 2];
                sdata1[tid] += sdata1[tid + 1];

                sdata2[tid] += sdata2[tid + 8];
                sdata2[tid] += sdata2[tid + 4];
                sdata2[tid] += sdata2[tid + 2];
                sdata2[tid] += sdata2[tid + 1];

                sdata3[tid] += sdata3[tid + 8];
                sdata3[tid] += sdata3[tid + 4];
                sdata3[tid] += sdata3[tid + 2];
                sdata3[tid] += sdata3[tid + 1];

                sdata4[tid] += sdata4[tid + 8];
                sdata4[tid] += sdata4[tid + 4];
                sdata4[tid] += sdata4[tid + 2];
                sdata4[tid] += sdata4[tid + 1];
            }
        }

        __global__ void compute_descriptors64(PtrStepf descriptors, const float* featureX, const float* featureY, const float* featureSize, const float* featureDir)
        {
            // 2 floats (dx,dy) for each thread (5x5 sample points in each sub-region)
            __shared__ float sdx[25];
            __shared__ float sdy[25];
            __shared__ float sdxabs[25];
            __shared__ float sdyabs[25];

            calc_dx_dy(sdx, sdy, featureX, featureY, featureSize, featureDir);
            __syncthreads();


            const int tid = threadIdx.y * blockDim.x + threadIdx.x;

            if (tid < 25)
            {
                sdxabs[tid] = ::fabs(sdx[tid]); // |dx| array
                sdyabs[tid] = ::fabs(sdy[tid]); // |dy| array
                __syncthreads();

                reduce_sum25(sdx, sdy, sdxabs, sdyabs, tid);
                __syncthreads();

                float* descriptors_block = descriptors.ptr(blockIdx.x) + (blockIdx.y << 2);

                // write dx, dy, |dx|, |dy|
                if (tid == 0)
                {
                    descriptors_block[0] = sdx[0];
                    descriptors_block[1] = sdy[0];
                    descriptors_block[2] = sdxabs[0];
                    descriptors_block[3] = sdyabs[0];
                }
            }
        }

        __global__ void compute_descriptors128(PtrStepf descriptors, const float* featureX, const float* featureY, const float* featureSize, const float* featureDir)
        {
            // 2 floats (dx,dy) for each thread (5x5 sample points in each sub-region)
            __shared__ float sdx[25];
            __shared__ float sdy[25];

            // sum (reduce) 5x5 area response
            __shared__ float sd1[25];
            __shared__ float sd2[25];
            __shared__ float sdabs1[25];
            __shared__ float sdabs2[25];

            calc_dx_dy(sdx, sdy, featureX, featureY, featureSize, featureDir);
            __syncthreads();

            const int tid = threadIdx.y * blockDim.x + threadIdx.x;

            if (tid < 25)
            {
                if (sdy[tid] >= 0)
                {
                    sd1[tid] = sdx[tid];
                    sdabs1[tid] = ::fabs(sdx[tid]);
                    sd2[tid] = 0;
                    sdabs2[tid] = 0;
                }
                else
                {
                    sd1[tid] = 0;
                    sdabs1[tid] = 0;
                    sd2[tid] = sdx[tid];
                    sdabs2[tid] = ::fabs(sdx[tid]);
                }
                __syncthreads();

                reduce_sum25(sd1, sd2, sdabs1, sdabs2, tid);
                __syncthreads();

                float* descriptors_block = descriptors.ptr(blockIdx.x) + (blockIdx.y << 3);

                // write dx (dy >= 0), |dx| (dy >= 0), dx (dy < 0), |dx| (dy < 0)
                if (tid == 0)
                {
                    descriptors_block[0] = sd1[0];
                    descriptors_block[1] = sdabs1[0];
                    descriptors_block[2] = sd2[0];
                    descriptors_block[3] = sdabs2[0];
                }
                __syncthreads();

                if (sdx[tid] >= 0)
                {
                    sd1[tid] = sdy[tid];
                    sdabs1[tid] = ::fabs(sdy[tid]);
                    sd2[tid] = 0;
                    sdabs2[tid] = 0;
                }
                else
                {
                    sd1[tid] = 0;
                    sdabs1[tid] = 0;
                    sd2[tid] = sdy[tid];
                    sdabs2[tid] = ::fabs(sdy[tid]);
                }
                __syncthreads();

                reduce_sum25(sd1, sd2, sdabs1, sdabs2, tid);
                __syncthreads();

                // write dy (dx >= 0), |dy| (dx >= 0), dy (dx < 0), |dy| (dx < 0)
                if (tid == 0)
                {
                    descriptors_block[4] = sd1[0];
                    descriptors_block[5] = sdabs1[0];
                    descriptors_block[6] = sd2[0];
                    descriptors_block[7] = sdabs2[0];
                }
            }
        }

        template <int BLOCK_DIM_X> __global__ void normalize_descriptors(PtrStepf descriptors)
        {
            // no need for thread ID
            float* descriptor_base = descriptors.ptr(blockIdx.x);

            // read in the unnormalized descriptor values (squared)
            __shared__ float sqDesc[BLOCK_DIM_X];
            const float lookup = descriptor_base[threadIdx.x];
            sqDesc[threadIdx.x] = lookup * lookup;
            __syncthreads();

            if (BLOCK_DIM_X >= 128)
            {
                if (threadIdx.x < 64)
                    sqDesc[threadIdx.x] += sqDesc[threadIdx.x + 64];
                __syncthreads();
            }

            // reduction to get total
            if (threadIdx.x < 32)
            {
                volatile float* smem = sqDesc;

                smem[threadIdx.x] += smem[threadIdx.x + 32];
                smem[threadIdx.x] += smem[threadIdx.x + 16];
                smem[threadIdx.x] += smem[threadIdx.x + 8];
                smem[threadIdx.x] += smem[threadIdx.x + 4];
                smem[threadIdx.x] += smem[threadIdx.x + 2];
                smem[threadIdx.x] += smem[threadIdx.x + 1];
            }

            // compute length (square root)
            __shared__ float len;
            if (threadIdx.x == 0)
            {
                len = sqrtf(sqDesc[0]);
            }
            __syncthreads();

            // normalize and store in output
            descriptor_base[threadIdx.x] = lookup / len;
        }

        void compute_descriptors_gpu(const DevMem2Df& descriptors,
            const float* featureX, const float* featureY, const float* featureSize, const float* featureDir, int nFeatures)
        {
            // compute unnormalized descriptors, then normalize them - odd indexing since grid must be 2D

            if (descriptors.cols == 64)
            {
                compute_descriptors64<<<dim3(nFeatures, 16, 1), dim3(6, 6, 1)>>>(descriptors, featureX, featureY, featureSize, featureDir);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                normalize_descriptors<64><<<dim3(nFeatures, 1, 1), dim3(64, 1, 1)>>>(descriptors);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );
            }
            else
            {
                compute_descriptors128<<<dim3(nFeatures, 16, 1), dim3(6, 6, 1)>>>(descriptors, featureX, featureY, featureSize, featureDir);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                normalize_descriptors<128><<<dim3(nFeatures, 1, 1), dim3(128, 1, 1)>>>(descriptors);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );
            }
        }
    } // namespace surf
}}} // namespace cv { namespace gpu { namespace device
