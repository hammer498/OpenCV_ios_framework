#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/vec_distance.hpp"
#include "opencv2/gpu/device/datamov_utils.hpp"

namespace cv { namespace gpu { namespace device 
{
    namespace bf_radius_match 
    {
        ///////////////////////////////////////////////////////////////////////////////
        // Match Unrolled

        template <int BLOCK_SIZE, int MAX_DESC_LEN, bool SAVE_IMG_IDX, typename Dist, typename T, typename Mask>
        __global__ void matchUnrolled(const DevMem2D_<T> query, int imgIdx, const DevMem2D_<T> train, float maxDistance, const Mask mask,
            PtrStepi bestTrainIdx, PtrStepi bestImgIdx, PtrStepf bestDistance, unsigned int* nMatches, int maxCount)
        {
            #if __CUDA_ARCH__ >= 110

            extern __shared__ int smem[];

            const int queryIdx = blockIdx.y * BLOCK_SIZE + threadIdx.y;
            const int trainIdx = blockIdx.x * BLOCK_SIZE + threadIdx.x;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            Dist dist;

            #pragma unroll
            for (int i = 0; i < MAX_DESC_LEN / BLOCK_SIZE; ++i)
            {
                const int loadX = threadIdx.x + i * BLOCK_SIZE;

                s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0;
                s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = 0;

                if (loadX < query.cols)
                {
                    T val;

                    ForceGlob<T>::Load(query.ptr(::min(queryIdx, query.rows - 1)), loadX, val);
                    s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = val;

                    ForceGlob<T>::Load(train.ptr(::min(blockIdx.x * BLOCK_SIZE + threadIdx.y, train.rows - 1)), loadX, val);
                    s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = val;
                }

                __syncthreads();

                #pragma unroll
                for (int j = 0; j < BLOCK_SIZE; ++j)
                    dist.reduceIter(s_query[threadIdx.y * BLOCK_SIZE + j], s_train[j * BLOCK_SIZE + threadIdx.x]);

                __syncthreads();
            }

            float distVal = (typename Dist::result_type)dist;

            if (queryIdx < query.rows && trainIdx < train.rows && mask(queryIdx, trainIdx) && distVal < maxDistance)
            {
                unsigned int ind = atomicInc(nMatches + queryIdx, (unsigned int) -1);
                if (ind < maxCount)
                {
                    bestTrainIdx.ptr(queryIdx)[ind] = trainIdx;
                    if (SAVE_IMG_IDX) bestImgIdx.ptr(queryIdx)[ind] = imgIdx;
                    bestDistance.ptr(queryIdx)[ind] = distVal;
                }
            }

            #endif
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask> 
        void matchUnrolled(const DevMem2D_<T>& query, const DevMem2D_<T>& train, float maxDistance, const Mask& mask, 
            const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(train.rows, BLOCK_SIZE), divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (2 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            matchUnrolled<BLOCK_SIZE, MAX_DESC_LEN, false, Dist><<<grid, block, smemSize, stream>>>(query, 0, train, maxDistance, mask, 
                trainIdx, PtrStepi(), distance, nMatches.data, trainIdx.cols);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }   

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T> 
        void matchUnrolled(const DevMem2D_<T>& query, const DevMem2D_<T>* trains, int n, float maxDistance, const DevMem2Db* masks, 
            const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, 
            hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

            const size_t smemSize = (2 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            for (int i = 0; i < n; ++i)
            {
                const DevMem2D_<T> train = trains[i];

                const dim3 grid(divUp(train.rows, BLOCK_SIZE), divUp(query.rows, BLOCK_SIZE));

                if (masks != 0 && masks[i].data)
                {
                    matchUnrolled<BLOCK_SIZE, MAX_DESC_LEN, true, Dist><<<grid, block, smemSize, stream>>>(query, i, train, maxDistance, SingleMask(masks[i]), 
                        trainIdx, imgIdx, distance, nMatches.data, trainIdx.cols);
                }
                else
                {
                    matchUnrolled<BLOCK_SIZE, MAX_DESC_LEN, true, Dist><<<grid, block, smemSize, stream>>>(query, i, train, maxDistance, WithOutMask(), 
                        trainIdx, imgIdx, distance, nMatches.data, trainIdx.cols);
                }
                cudaSafeCall( hipGetLastError() );
            }

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ///////////////////////////////////////////////////////////////////////////////
        // Match

        template <int BLOCK_SIZE, bool SAVE_IMG_IDX, typename Dist, typename T, typename Mask>
        __global__ void match(const DevMem2D_<T> query, int imgIdx, const DevMem2D_<T> train, float maxDistance, const Mask mask,
            PtrStepi bestTrainIdx, PtrStepi bestImgIdx, PtrStepf bestDistance, unsigned int* nMatches, int maxCount)
        {
            #if __CUDA_ARCH__ >= 110

            extern __shared__ int smem[];

            const int queryIdx = blockIdx.y * BLOCK_SIZE + threadIdx.y;
            const int trainIdx = blockIdx.x * BLOCK_SIZE + threadIdx.x;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            Dist dist;

            for (int i = 0, endi = (query.cols + BLOCK_SIZE - 1) / BLOCK_SIZE; i < endi; ++i)
            {
                const int loadX = threadIdx.x + i * BLOCK_SIZE;

                s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0;
                s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = 0;

                if (loadX < query.cols)
                {
                    T val;

                    ForceGlob<T>::Load(query.ptr(::min(queryIdx, query.rows - 1)), loadX, val);
                    s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = val;

                    ForceGlob<T>::Load(train.ptr(::min(blockIdx.x * BLOCK_SIZE + threadIdx.y, train.rows - 1)), loadX, val);
                    s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = val;
                }

                __syncthreads();

                #pragma unroll
                for (int j = 0; j < BLOCK_SIZE; ++j)
                    dist.reduceIter(s_query[threadIdx.y * BLOCK_SIZE + j], s_train[j * BLOCK_SIZE + threadIdx.x]);

                __syncthreads();
            }

            float distVal = (typename Dist::result_type)dist;

            if (queryIdx < query.rows && trainIdx < train.rows && mask(queryIdx, trainIdx) && distVal < maxDistance)
            {
                unsigned int ind = atomicInc(nMatches + queryIdx, (unsigned int) -1);
                if (ind < maxCount)
                {
                    bestTrainIdx.ptr(queryIdx)[ind] = trainIdx;
                    if (SAVE_IMG_IDX) bestImgIdx.ptr(queryIdx)[ind] = imgIdx;
                    bestDistance.ptr(queryIdx)[ind] = distVal;
                }
            }

            #endif
        }

        template <int BLOCK_SIZE, typename Dist, typename T, typename Mask> 
        void match(const DevMem2D_<T>& query, const DevMem2D_<T>& train, float maxDistance, const Mask& mask, 
            const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, 
            hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(train.rows, BLOCK_SIZE), divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (2 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            match<BLOCK_SIZE, false, Dist><<<grid, block, smemSize, stream>>>(query, 0, train, maxDistance, mask, 
                trainIdx, PtrStepi(), distance, nMatches.data, trainIdx.cols);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <int BLOCK_SIZE, typename Dist, typename T> 
        void match(const DevMem2D_<T>& query, const DevMem2D_<T>* trains, int n, float maxDistance, const DevMem2Db* masks, 
            const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, 
            hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

            const size_t smemSize = (2 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            for (int i = 0; i < n; ++i)
            {
                const DevMem2D_<T> train = trains[i];

                const dim3 grid(divUp(train.rows, BLOCK_SIZE), divUp(query.rows, BLOCK_SIZE));

                if (masks != 0 && masks[i].data)
                {
                    match<BLOCK_SIZE, true, Dist><<<grid, block, smemSize, stream>>>(query, i, train, maxDistance, SingleMask(masks[i]), 
                        trainIdx, imgIdx, distance, nMatches.data, trainIdx.cols);
                }
                else
                {
                    match<BLOCK_SIZE, true, Dist><<<grid, block, smemSize, stream>>>(query, i, train, maxDistance, WithOutMask(), 
                        trainIdx, imgIdx, distance, nMatches.data, trainIdx.cols);
                }
                cudaSafeCall( hipGetLastError() );
            }

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ///////////////////////////////////////////////////////////////////////////////
        // Match dispatcher

        template <typename Dist, typename T, typename Mask> 
        void matchDispatcher(const DevMem2D_<T>& query, const DevMem2D_<T>& train, float maxDistance, const Mask& mask, 
                             const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, 
                             int cc, hipStream_t stream)
        {
            if (query.cols <= 64)
            {
                matchUnrolled<16, 64, Dist>(query, train, maxDistance, mask, trainIdx, distance, nMatches, stream);
            }
            else if (query.cols <= 128)
            {
                matchUnrolled<16, 128, Dist>(query, train, maxDistance, mask, trainIdx, distance, nMatches, stream);
            }
            /*else if (query.cols <= 256)
            {
                matchUnrolled<16, 256, Dist>(query, train, maxDistance, mask, trainIdx, distance, nMatches, stream);
            }
            else if (query.cols <= 512)
            {            
                matchUnrolled<16, 512, Dist>(query, train, maxDistance, mask, trainIdx, distance, nMatches, stream);
            }
            else if (query.cols <= 1024)
            {            
                matchUnrolled<16, 1024, Dist>(query, train, maxDistance, mask, trainIdx, distance, nMatches, stream);
            }*/
            else
            {
                match<16, Dist>(query, train, maxDistance, mask, trainIdx, distance, nMatches, stream);
            }
        }

        template <typename Dist, typename T> 
        void matchDispatcher(const DevMem2D_<T>& query, const DevMem2D_<T>* trains, int n, float maxDistance, const DevMem2Db* masks, 
                             const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, 
                             int cc, hipStream_t stream)
        {
            if (query.cols <= 64)
            {
                matchUnrolled<16, 64, Dist>(query, trains, n, maxDistance, masks, trainIdx, imgIdx, distance, nMatches, stream);
            }
            else if (query.cols <= 128)
            {
                matchUnrolled<16, 128, Dist>(query, trains, n, maxDistance, masks, trainIdx, imgIdx, distance, nMatches, stream);
            }
            /*else if (query.cols <= 256)
            {
                matchUnrolled<16, 256, Dist>(query, trains, n, maxDistance, masks, trainIdx, imgIdx, distance, nMatches, stream);
            }
            else if (query.cols <= 512)
            {            
                matchUnrolled<16, 512, Dist>(query, trains, n, maxDistance, masks, trainIdx, imgIdx, distance, nMatches, stream);
            }
            else if (query.cols <= 1024)
            {            
                matchUnrolled<16, 1024, Dist>(query, trains, n, maxDistance, masks, trainIdx, imgIdx, distance, nMatches, stream);
            }*/
            else
            {
                match<16, Dist>(query, trains, n, maxDistance, masks, trainIdx, imgIdx, distance, nMatches, stream);
            }
        } 

        ///////////////////////////////////////////////////////////////////////////////
        // Radius Match caller

        template <typename T> void matchL1_gpu(const DevMem2Db& query, const DevMem2Db& train, float maxDistance, const DevMem2Db& mask, 
            const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, 
            int cc, hipStream_t stream)
        {
            if (mask.data)
            {
                matchDispatcher< L1Dist<T> >(static_cast< DevMem2D_<T> >(query), static_cast< DevMem2D_<T> >(train), maxDistance, SingleMask(mask), 
                    trainIdx, distance, nMatches, 
                    cc, stream);
            }
            else
            {
                matchDispatcher< L1Dist<T> >(static_cast< DevMem2D_<T> >(query), static_cast< DevMem2D_<T> >(train), maxDistance, WithOutMask(), 
                    trainIdx, distance, nMatches, 
                    cc, stream);
            }
        }

        template void matchL1_gpu<uchar >(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        //template void matchL1_gpu<schar >(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        template void matchL1_gpu<ushort>(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        template void matchL1_gpu<short >(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        template void matchL1_gpu<int   >(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        template void matchL1_gpu<float >(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);

        template <typename T> void matchL2_gpu(const DevMem2Db& query, const DevMem2Db& train, float maxDistance, const DevMem2Db& mask, 
            const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, 
            int cc, hipStream_t stream)
        {
            if (mask.data)
            {
                matchDispatcher<L2Dist>(static_cast< DevMem2D_<T> >(query), static_cast< DevMem2D_<T> >(train), maxDistance, SingleMask(mask), 
                    trainIdx, distance, nMatches, 
                    cc, stream);
            }
            else
            {
                matchDispatcher<L2Dist>(static_cast< DevMem2D_<T> >(query), static_cast< DevMem2D_<T> >(train), maxDistance, WithOutMask(), 
                    trainIdx, distance, nMatches, 
                    cc, stream);
            }
        }

        //template void matchL2_gpu<uchar >(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        //template void matchL2_gpu<schar >(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        //template void matchL2_gpu<ushort>(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        //template void matchL2_gpu<short >(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        //template void matchL2_gpu<int   >(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        template void matchL2_gpu<float >(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);

        template <typename T> void matchHamming_gpu(const DevMem2Db& query, const DevMem2Db& train, float maxDistance, const DevMem2Db& mask, 
            const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, 
            int cc, hipStream_t stream)
        {
            if (mask.data)
            {
                matchDispatcher<HammingDist>(static_cast< DevMem2D_<T> >(query), static_cast< DevMem2D_<T> >(train), maxDistance, SingleMask(mask), 
                    trainIdx, distance, nMatches, 
                    cc, stream);
            }
            else
            {
                matchDispatcher<HammingDist>(static_cast< DevMem2D_<T> >(query), static_cast< DevMem2D_<T> >(train), maxDistance, WithOutMask(), 
                    trainIdx, distance, nMatches, 
                    cc, stream);
            }
        }

        template void matchHamming_gpu<uchar >(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        //template void matchHamming_gpu<schar >(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        template void matchHamming_gpu<ushort>(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        //template void matchHamming_gpu<short >(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        template void matchHamming_gpu<int   >(const DevMem2Db& queryDescs, const DevMem2Db& trainDescs, float maxDistance, const DevMem2Db& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);

        template <typename T> void matchL1_gpu(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, 
            const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, 
            int cc, hipStream_t stream)
        {
            matchDispatcher< L1Dist<T> >(static_cast< DevMem2D_<T> >(query), (const DevMem2D_<T>*)trains, n, maxDistance, masks, 
                trainIdx, imgIdx, distance, nMatches, 
                cc, stream);
        }

        template void matchL1_gpu<uchar >(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        //template void matchL1_gpu<schar >(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        template void matchL1_gpu<ushort>(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        template void matchL1_gpu<short >(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        template void matchL1_gpu<int   >(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        template void matchL1_gpu<float >(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);

        template <typename T> void matchL2_gpu(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, 
            const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, 
            int cc, hipStream_t stream)
        {
            matchDispatcher<L2Dist>(static_cast< DevMem2D_<T> >(query), (const DevMem2D_<T>*)trains, n, maxDistance, masks, 
                trainIdx, imgIdx, distance, nMatches, 
                cc, stream);
        }

        //template void matchL2_gpu<uchar >(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        //template void matchL2_gpu<schar >(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        //template void matchL2_gpu<ushort>(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        //template void matchL2_gpu<short >(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        //template void matchL2_gpu<int   >(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        template void matchL2_gpu<float >(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);

        template <typename T> void matchHamming_gpu(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, 
            const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, 
            int cc, hipStream_t stream)
        {
            matchDispatcher<HammingDist>(static_cast< DevMem2D_<T> >(query), (const DevMem2D_<T>*)trains, n, maxDistance, masks, 
                trainIdx, imgIdx, distance, nMatches, 
                cc, stream);
        }

        template void matchHamming_gpu<uchar >(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        //template void matchHamming_gpu<schar >(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        template void matchHamming_gpu<ushort>(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        //template void matchHamming_gpu<short >(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
        template void matchHamming_gpu<int   >(const DevMem2Db& query, const DevMem2Db* trains, int n, float maxDistance, const DevMem2Db* masks, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, const DevMem2D_<unsigned int>& nMatches, int cc, hipStream_t stream);
    } // namespace bf_radius_match
}}} // namespace cv { namespace gpu { namespace device
