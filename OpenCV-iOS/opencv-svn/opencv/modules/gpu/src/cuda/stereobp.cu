#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"
#include "opencv2/gpu/device/limits.hpp"

namespace cv { namespace gpu { namespace device 
{
    namespace stereobp 
    {
        ///////////////////////////////////////////////////////////////
        /////////////////////// load constants ////////////////////////
        ///////////////////////////////////////////////////////////////

        __constant__ int   cndisp;
        __constant__ float cmax_data_term;
        __constant__ float cdata_weight;
        __constant__ float cmax_disc_term;
        __constant__ float cdisc_single_jump;

        void load_constants(int ndisp, float max_data_term, float data_weight, float max_disc_term, float disc_single_jump)
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cndisp),            &ndisp,            sizeof(int  )) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cmax_data_term),    &max_data_term,    sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdata_weight),      &data_weight,      sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cmax_disc_term),    &max_disc_term,    sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdisc_single_jump), &disc_single_jump, sizeof(float)) );
        }

        ///////////////////////////////////////////////////////////////
        ////////////////////////// comp data //////////////////////////
        ///////////////////////////////////////////////////////////////

        template <int cn> struct PixDiff;
        template <> struct PixDiff<1>
        {
            __device__ __forceinline__ PixDiff(const uchar* ls)
            {
                l = *ls;
            }
            __device__ __forceinline__ float operator()(const uchar* rs) const
            {
                return ::abs((int)l - *rs);
            }
            uchar l;
        };
        template <> struct PixDiff<3>
        {
            __device__ __forceinline__ PixDiff(const uchar* ls)
            {
                l = *((uchar3*)ls);
            }
            __device__ __forceinline__ float operator()(const uchar* rs) const
            {
                const float tr = 0.299f;
                const float tg = 0.587f;
                const float tb = 0.114f;

                float val  = tb * ::abs((int)l.x - rs[0]);
                      val += tg * ::abs((int)l.y - rs[1]);
                      val += tr * ::abs((int)l.z - rs[2]);

                return val;
            }
            uchar3 l;
        };
        template <> struct PixDiff<4>
        {
            __device__ __forceinline__ PixDiff(const uchar* ls)
            {
                l = *((uchar4*)ls);
            }
            __device__ __forceinline__ float operator()(const uchar* rs) const
            {
                const float tr = 0.299f;
                const float tg = 0.587f;
                const float tb = 0.114f;

                uchar4 r = *((uchar4*)rs);

                float val  = tb * ::abs((int)l.x - r.x);
                      val += tg * ::abs((int)l.y - r.y);
                      val += tr * ::abs((int)l.z - r.z);

                return val;
            }
            uchar4 l;
        };

        template <int cn, typename D>
        __global__ void comp_data(const DevMem2Db left, const PtrStepb right, PtrElemStep_<D> data)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y > 0 && y < left.rows - 1 && x > 0 && x < left.cols - 1)
            {
                const uchar* ls = left.ptr(y) + x * cn;
                const PixDiff<cn> pixDiff(ls);
                const uchar* rs = right.ptr(y) + x * cn;

                D* ds = data.ptr(y) + x;
                const size_t disp_step = data.step * left.rows;

                for (int disp = 0; disp < cndisp; disp++)
                {
                    if (x - disp >= 1)
                    {
                        float val = pixDiff(rs - disp * cn);

                        ds[disp * disp_step] = saturate_cast<D>(fmin(cdata_weight * val, cdata_weight * cmax_data_term));
                    }
                    else
                    {
                        ds[disp * disp_step] = saturate_cast<D>(cdata_weight * cmax_data_term);
                    }
                }
            }
        }

        template<typename T, typename D>
        void comp_data_gpu(const DevMem2Db& left, const DevMem2Db& right, const DevMem2Db& data, hipStream_t stream);

        template <> void comp_data_gpu<uchar, short>(const DevMem2Db& left, const DevMem2Db& right, const DevMem2Db& data, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(left.cols, threads.x);
            grid.y = divUp(left.rows, threads.y);

            comp_data<1, short><<<grid, threads, 0, stream>>>(left, right, (DevMem2D_<short>)data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
        template <> void comp_data_gpu<uchar, float>(const DevMem2Db& left, const DevMem2Db& right, const DevMem2Db& data, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(left.cols, threads.x);
            grid.y = divUp(left.rows, threads.y);

            comp_data<1, float><<<grid, threads, 0, stream>>>(left, right, (DevMem2D_<float>)data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <> void comp_data_gpu<uchar3, short>(const DevMem2Db& left, const DevMem2Db& right, const DevMem2Db& data, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(left.cols, threads.x);
            grid.y = divUp(left.rows, threads.y);

            comp_data<3, short><<<grid, threads, 0, stream>>>(left, right, (DevMem2D_<short>)data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
        template <> void comp_data_gpu<uchar3, float>(const DevMem2Db& left, const DevMem2Db& right, const DevMem2Db& data, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(left.cols, threads.x);
            grid.y = divUp(left.rows, threads.y);

            comp_data<3, float><<<grid, threads, 0, stream>>>(left, right, (DevMem2D_<float>)data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <> void comp_data_gpu<uchar4, short>(const DevMem2Db& left, const DevMem2Db& right, const DevMem2Db& data, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(left.cols, threads.x);
            grid.y = divUp(left.rows, threads.y);

            comp_data<4, short><<<grid, threads, 0, stream>>>(left, right, (DevMem2D_<short>)data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
        template <> void comp_data_gpu<uchar4, float>(const DevMem2Db& left, const DevMem2Db& right, const DevMem2Db& data, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(left.cols, threads.x);
            grid.y = divUp(left.rows, threads.y);

            comp_data<4, float><<<grid, threads, 0, stream>>>(left, right, (DevMem2D_<float>)data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ///////////////////////////////////////////////////////////////
        //////////////////////// data step down ///////////////////////
        ///////////////////////////////////////////////////////////////

        template <typename T>
        __global__ void data_step_down(int dst_cols, int dst_rows, int src_rows, const PtrStep<T> src, PtrStep<T> dst)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < dst_cols && y < dst_rows)
            {
                for (int d = 0; d < cndisp; ++d)
                {
                    float dst_reg  = src.ptr(d * src_rows + (2*y+0))[(2*x+0)];
                          dst_reg += src.ptr(d * src_rows + (2*y+1))[(2*x+0)];
                          dst_reg += src.ptr(d * src_rows + (2*y+0))[(2*x+1)];
                          dst_reg += src.ptr(d * src_rows + (2*y+1))[(2*x+1)];

                    dst.ptr(d * dst_rows + y)[x] = saturate_cast<T>(dst_reg);
                }
            }
        }

        template<typename T>
        void data_step_down_gpu(int dst_cols, int dst_rows, int src_rows, const DevMem2Db& src, const DevMem2Db& dst, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(dst_cols, threads.x);
            grid.y = divUp(dst_rows, threads.y);

            data_step_down<T><<<grid, threads, 0, stream>>>(dst_cols, dst_rows, src_rows, (DevMem2D_<T>)src, (DevMem2D_<T>)dst);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void data_step_down_gpu<short>(int dst_cols, int dst_rows, int src_rows, const DevMem2Db& src, const DevMem2Db& dst, hipStream_t stream);
        template void data_step_down_gpu<float>(int dst_cols, int dst_rows, int src_rows, const DevMem2Db& src, const DevMem2Db& dst, hipStream_t stream);

        ///////////////////////////////////////////////////////////////
        /////////////////// level up messages  ////////////////////////
        ///////////////////////////////////////////////////////////////

        template <typename T>
        __global__ void level_up_message(int dst_cols, int dst_rows, int src_rows, const PtrElemStep_<T> src, PtrElemStep_<T> dst)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < dst_cols && y < dst_rows)
            {
                const size_t dst_disp_step = dst.step * dst_rows;
                const size_t src_disp_step = src.step * src_rows;

                T*       dstr = dst.ptr(y  ) + x;
                const T* srcr = src.ptr(y/2) + x/2;

                for (int d = 0; d < cndisp; ++d)
                    dstr[d * dst_disp_step] = srcr[d * src_disp_step];
            }
        }

        template <typename T>
        void level_up_messages_gpu(int dst_idx, int dst_cols, int dst_rows, int src_rows, DevMem2Db* mus, DevMem2Db* mds, DevMem2Db* mls, DevMem2Db* mrs, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(dst_cols, threads.x);
            grid.y = divUp(dst_rows, threads.y);

            int src_idx = (dst_idx + 1) & 1;

            level_up_message<T><<<grid, threads, 0, stream>>>(dst_cols, dst_rows, src_rows, (DevMem2D_<T>)mus[src_idx], (DevMem2D_<T>)mus[dst_idx]);
            cudaSafeCall( hipGetLastError() );

            level_up_message<T><<<grid, threads, 0, stream>>>(dst_cols, dst_rows, src_rows, (DevMem2D_<T>)mds[src_idx], (DevMem2D_<T>)mds[dst_idx]);
            cudaSafeCall( hipGetLastError() );

            level_up_message<T><<<grid, threads, 0, stream>>>(dst_cols, dst_rows, src_rows, (DevMem2D_<T>)mls[src_idx], (DevMem2D_<T>)mls[dst_idx]);
            cudaSafeCall( hipGetLastError() );

            level_up_message<T><<<grid, threads, 0, stream>>>(dst_cols, dst_rows, src_rows, (DevMem2D_<T>)mrs[src_idx], (DevMem2D_<T>)mrs[dst_idx]);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void level_up_messages_gpu<short>(int dst_idx, int dst_cols, int dst_rows, int src_rows, DevMem2Db* mus, DevMem2Db* mds, DevMem2Db* mls, DevMem2Db* mrs, hipStream_t stream);
        template void level_up_messages_gpu<float>(int dst_idx, int dst_cols, int dst_rows, int src_rows, DevMem2Db* mus, DevMem2Db* mds, DevMem2Db* mls, DevMem2Db* mrs, hipStream_t stream);

        ///////////////////////////////////////////////////////////////
        ////////////////////  calc all iterations /////////////////////
        ///////////////////////////////////////////////////////////////

        template <typename T>
        __device__ void calc_min_linear_penalty(T* dst, size_t step)
        {
            float prev = dst[0];
            float cur;
            for (int disp = 1; disp < cndisp; ++disp)
            {
                prev += cdisc_single_jump;
                cur = dst[step * disp];
                if (prev < cur)
                {
                    cur = prev;
                    dst[step * disp] = saturate_cast<T>(prev);
                }
                prev = cur;
            }

            prev = dst[(cndisp - 1) * step];
            for (int disp = cndisp - 2; disp >= 0; disp--)
            {
                prev += cdisc_single_jump;
                cur = dst[step * disp];
                if (prev < cur)
                {
                    cur = prev;
                    dst[step * disp] = saturate_cast<T>(prev);
                }
                prev = cur;
            }
        }

        template <typename T>
        __device__ void message(const T* msg1, const T* msg2, const T* msg3, const T* data, T* dst, size_t msg_disp_step, size_t data_disp_step)
        {
            float minimum = device::numeric_limits<float>::max();

            for(int i = 0; i < cndisp; ++i)
            {
                float dst_reg  = msg1[msg_disp_step * i];
                      dst_reg += msg2[msg_disp_step * i];
                      dst_reg += msg3[msg_disp_step * i];
                      dst_reg += data[data_disp_step * i];

                if (dst_reg < minimum)
                    minimum = dst_reg;

                dst[msg_disp_step * i] = saturate_cast<T>(dst_reg);
            }

            calc_min_linear_penalty(dst, msg_disp_step);

            minimum += cmax_disc_term;

            float sum = 0;
            for(int i = 0; i < cndisp; ++i)
            {
                float dst_reg = dst[msg_disp_step * i];
                if (dst_reg > minimum)
                {
                    dst_reg = minimum;
                    dst[msg_disp_step * i] = saturate_cast<T>(minimum);
                }
                sum += dst_reg;
            }
            sum /= cndisp;

            for(int i = 0; i < cndisp; ++i)
                dst[msg_disp_step * i] -= sum;
        }

        template <typename T>
        __global__ void one_iteration(int t, PtrElemStep_<T> u, T* d, T* l, T* r, const PtrElemStep_<T> data, int cols, int rows)
        {
            const int y = blockIdx.y * blockDim.y + threadIdx.y;
            const int x = ((blockIdx.x * blockDim.x + threadIdx.x) << 1) + ((y + t) & 1);

            if ((y > 0) && (y < rows - 1) && (x > 0) && (x < cols - 1))
            {
                T* us = u.ptr(y) + x;
                T* ds = d + y * u.step + x;
                T* ls = l + y * u.step + x;
                T* rs = r + y * u.step + x;
                const T* dt = data.ptr(y) + x;

                size_t msg_disp_step = u.step * rows;
                size_t data_disp_step = data.step * rows;

                message(us + u.step, ls      + 1, rs - 1, dt, us, msg_disp_step, data_disp_step);
                message(ds - u.step, ls      + 1, rs - 1, dt, ds, msg_disp_step, data_disp_step);
                message(us + u.step, ds - u.step, rs - 1, dt, rs, msg_disp_step, data_disp_step);
                message(us + u.step, ds - u.step, ls + 1, dt, ls, msg_disp_step, data_disp_step);
            }
        }

        template <typename T>
        void calc_all_iterations_gpu(int cols, int rows, int iters, const DevMem2Db& u, const DevMem2Db& d,
            const DevMem2Db& l, const DevMem2Db& r, const DevMem2Db& data, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(cols, threads.x << 1);
            grid.y = divUp(rows, threads.y);

            for(int t = 0; t < iters; ++t)
            {
                one_iteration<T><<<grid, threads, 0, stream>>>(t, (DevMem2D_<T>)u, (T*)d.data, (T*)l.data, (T*)r.data, (DevMem2D_<T>)data, cols, rows);
                cudaSafeCall( hipGetLastError() );

                if (stream == 0)
                    cudaSafeCall( hipDeviceSynchronize() );
            }
        }

        template void calc_all_iterations_gpu<short>(int cols, int rows, int iters, const DevMem2Db& u, const DevMem2Db& d, const DevMem2Db& l, const DevMem2Db& r, const DevMem2Db& data, hipStream_t stream);
        template void calc_all_iterations_gpu<float>(int cols, int rows, int iters, const DevMem2Db& u, const DevMem2Db& d, const DevMem2Db& l, const DevMem2Db& r, const DevMem2Db& data, hipStream_t stream);

        ///////////////////////////////////////////////////////////////
        /////////////////////////// output ////////////////////////////
        ///////////////////////////////////////////////////////////////

        template <typename T>
        __global__ void output(const PtrElemStep_<T> u, const T* d, const T* l, const T* r, const T* data,
            DevMem2D_<short> disp)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y > 0 && y < disp.rows - 1 && x > 0 && x < disp.cols - 1)
            {
                const T* us = u.ptr(y + 1) + x;
                const T* ds = d + (y - 1) * u.step + x;
                const T* ls = l + y * u.step + (x + 1);
                const T* rs = r + y * u.step + (x - 1);
                const T* dt = data + y * u.step + x;

                size_t disp_step = disp.rows * u.step;

                int best = 0;
                float best_val = numeric_limits<float>::max();
                for (int d = 0; d < cndisp; ++d)
                {
                    float val  = us[d * disp_step];
                          val += ds[d * disp_step];
                          val += ls[d * disp_step];
                          val += rs[d * disp_step];
                          val += dt[d * disp_step];

                    if (val < best_val)
                    {
                        best_val = val;
                        best = d;
                    }
                }

                disp.ptr(y)[x] = saturate_cast<short>(best);
            }
        }

        template <typename T>
        void output_gpu(const DevMem2Db& u, const DevMem2Db& d, const DevMem2Db& l, const DevMem2Db& r, const DevMem2Db& data,
            const DevMem2D_<short>& disp, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(disp.cols, threads.x);
            grid.y = divUp(disp.rows, threads.y);

            output<T><<<grid, threads, 0, stream>>>((DevMem2D_<T>)u, (const T*)d.data, (const T*)l.data, (const T*)r.data, (const T*)data.data, disp);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void output_gpu<short>(const DevMem2Db& u, const DevMem2Db& d, const DevMem2Db& l, const DevMem2Db& r, const DevMem2Db& data, const DevMem2D_<short>& disp, hipStream_t stream);
        template void output_gpu<float>(const DevMem2Db& u, const DevMem2Db& d, const DevMem2Db& l, const DevMem2Db& r, const DevMem2Db& data, const DevMem2D_<short>& disp, hipStream_t stream);
    } // namespace stereobp
}}} // namespace cv { namespace gpu { namespace device
