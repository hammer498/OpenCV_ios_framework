#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Copyright (C) 1993-2011, NVIDIA Corporation, all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/utility.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"

namespace cv { namespace gpu { namespace device 
{
    #define UINT_BITS 32U

    //Warps == subhistograms per threadblock
    #define WARP_COUNT 6

    //Threadblock size
    #define HISTOGRAM256_THREADBLOCK_SIZE (WARP_COUNT * OPENCV_GPU_WARP_SIZE)
    #define HISTOGRAM256_BIN_COUNT 256

    //Shared memory per threadblock
    #define HISTOGRAM256_THREADBLOCK_MEMORY (WARP_COUNT * HISTOGRAM256_BIN_COUNT)

    #define PARTIAL_HISTOGRAM256_COUNT 240

    #define MERGE_THREADBLOCK_SIZE 256

    #define USE_SMEM_ATOMICS (__CUDA_ARCH__ >= 120)

    namespace hist 
    {
        #if (!USE_SMEM_ATOMICS)

            #define TAG_MASK ( (1U << (UINT_BITS - OPENCV_GPU_LOG_WARP_SIZE)) - 1U )

            __forceinline__ __device__ void addByte(volatile uint* s_WarpHist, uint data, uint threadTag)
            {
                uint count;
                do
                {
                    count = s_WarpHist[data] & TAG_MASK;
                    count = threadTag | (count + 1);
                    s_WarpHist[data] = count;
                } while (s_WarpHist[data] != count);
            }

        #else

            #define TAG_MASK 0xFFFFFFFFU

            __forceinline__ __device__ void addByte(uint* s_WarpHist, uint data, uint threadTag)
            {
                atomicAdd(s_WarpHist + data, 1);
            }

        #endif

        __forceinline__ __device__ void addWord(uint* s_WarpHist, uint data, uint tag, uint pos_x, uint cols)
        {
            uint x = pos_x << 2;

            if (x + 0 < cols) addByte(s_WarpHist, (data >>  0) & 0xFFU, tag);
            if (x + 1 < cols) addByte(s_WarpHist, (data >>  8) & 0xFFU, tag);
            if (x + 2 < cols) addByte(s_WarpHist, (data >> 16) & 0xFFU, tag);
            if (x + 3 < cols) addByte(s_WarpHist, (data >> 24) & 0xFFU, tag);
        }

        __global__ void histogram256(const PtrStep<uint> d_Data, uint* d_PartialHistograms, uint dataCount, uint cols)
        {
            //Per-warp subhistogram storage
            __shared__ uint s_Hist[HISTOGRAM256_THREADBLOCK_MEMORY];
            uint* s_WarpHist= s_Hist + (threadIdx.x >> OPENCV_GPU_LOG_WARP_SIZE) * HISTOGRAM256_BIN_COUNT;

            //Clear shared memory storage for current threadblock before processing
            #pragma unroll
            for (uint i = 0; i < (HISTOGRAM256_THREADBLOCK_MEMORY / HISTOGRAM256_THREADBLOCK_SIZE); i++)
               s_Hist[threadIdx.x + i * HISTOGRAM256_THREADBLOCK_SIZE] = 0;

            //Cycle through the entire data set, update subhistograms for each warp
            const uint tag = threadIdx.x << (UINT_BITS - OPENCV_GPU_LOG_WARP_SIZE);

            __syncthreads();
            const uint colsui = d_Data.step / sizeof(uint);
            for(uint pos = blockIdx.x * blockDim.x + threadIdx.x; pos < dataCount; pos += blockDim.x * gridDim.x)
            {
                uint pos_y = pos / colsui;
                uint pos_x = pos % colsui;
                uint data = d_Data.ptr(pos_y)[pos_x];
                addWord(s_WarpHist, data, tag, pos_x, cols);
            }

            //Merge per-warp histograms into per-block and write to global memory
            __syncthreads();
            for(uint bin = threadIdx.x; bin < HISTOGRAM256_BIN_COUNT; bin += HISTOGRAM256_THREADBLOCK_SIZE)
            {
                uint sum = 0;

                for (uint i = 0; i < WARP_COUNT; i++)
                    sum += s_Hist[bin + i * HISTOGRAM256_BIN_COUNT] & TAG_MASK;

                d_PartialHistograms[blockIdx.x * HISTOGRAM256_BIN_COUNT + bin] = sum;
            }
        }

        ////////////////////////////////////////////////////////////////////////////////
        // Merge histogram256() output
        // Run one threadblock per bin; each threadblock adds up the same bin counter
        // from every partial histogram. Reads are uncoalesced, but mergeHistogram256
        // takes only a fraction of total processing time
        ////////////////////////////////////////////////////////////////////////////////

        __global__ void mergeHistogram256(const uint* d_PartialHistograms, int* d_Histogram)
        {
            uint sum = 0;

            #pragma unroll
            for (uint i = threadIdx.x; i < PARTIAL_HISTOGRAM256_COUNT; i += MERGE_THREADBLOCK_SIZE)
                sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM256_BIN_COUNT];

            __shared__ uint data[MERGE_THREADBLOCK_SIZE];
            data[threadIdx.x] = sum;

            for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
            {
                __syncthreads();
                if(threadIdx.x < stride)
                    data[threadIdx.x] += data[threadIdx.x + stride];
            }

            if(threadIdx.x == 0)
                d_Histogram[blockIdx.x] = saturate_cast<int>(data[0]);
        }

        void histogram256_gpu(DevMem2Db src, int* hist, uint* buf, hipStream_t stream)
        {
            histogram256<<<PARTIAL_HISTOGRAM256_COUNT, HISTOGRAM256_THREADBLOCK_SIZE, 0, stream>>>(
                DevMem2D_<uint>(src),
                buf, 
                static_cast<uint>(src.rows * src.step / sizeof(uint)),
                src.cols);

            cudaSafeCall( hipGetLastError() );

            mergeHistogram256<<<HISTOGRAM256_BIN_COUNT, MERGE_THREADBLOCK_SIZE, 0, stream>>>(buf, hist);

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        __constant__ int c_lut[256];

        __global__ void equalizeHist(const DevMem2Db src, PtrStepb dst)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < src.cols && y < src.rows)
            {
                const uchar val = src.ptr(y)[x];
                const int lut = c_lut[val];
                dst.ptr(y)[x] = __float2int_rn(255.0f / (src.cols * src.rows) * lut);
            }
        }

        void equalizeHist_gpu(DevMem2Db src, DevMem2Db dst, const int* lut, hipStream_t stream)
        {
            dim3 block(16, 16);
            dim3 grid(divUp(src.cols, block.x), divUp(src.rows, block.y));

            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_lut), lut, 256 * sizeof(int), 0, hipMemcpyDeviceToDevice) );

            equalizeHist<<<grid, block, 0, stream>>>(src, dst);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
    } // namespace hist
}}} // namespace cv { namespace gpu { namespace device
