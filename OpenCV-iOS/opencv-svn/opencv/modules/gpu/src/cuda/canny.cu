#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <utility>
#include <algorithm>
#include "internal_shared.hpp"

namespace cv { namespace gpu { namespace device 
{
    namespace canny 
    {
        __global__ void calcSobelRowPass(const PtrStepb src, PtrStepi dx_buf, PtrStepi dy_buf, int rows, int cols)
        {
            __shared__ int smem[16][18];

            const int j = blockIdx.x * blockDim.x + threadIdx.x;
            const int i = blockIdx.y * blockDim.y + threadIdx.y;

            if (i < rows)
            {
                smem[threadIdx.y][threadIdx.x + 1] = src.ptr(i)[j];
                if (threadIdx.x == 0)
                {
                    smem[threadIdx.y][0] = src.ptr(i)[::max(j - 1, 0)];
                    smem[threadIdx.y][17] = src.ptr(i)[::min(j + 16, cols - 1)];
                }
                __syncthreads();

                if (j < cols)
                {
                    dx_buf.ptr(i)[j] = -smem[threadIdx.y][threadIdx.x] + smem[threadIdx.y][threadIdx.x + 2];
                    dy_buf.ptr(i)[j] = smem[threadIdx.y][threadIdx.x] + 2 * smem[threadIdx.y][threadIdx.x + 1] + smem[threadIdx.y][threadIdx.x + 2];
                }
            }
        }

        void calcSobelRowPass_gpu(PtrStepb src, PtrStepi dx_buf, PtrStepi dy_buf, int rows, int cols)
        {
            dim3 block(16, 16, 1);
            dim3 grid(divUp(cols, block.x), divUp(rows, block.y), 1);

            calcSobelRowPass<<<grid, block>>>(src, dx_buf, dy_buf, rows, cols);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        struct L1
        {
            static __device__ __forceinline__ float calc(int x, int y)
            {
                return ::abs(x) + ::abs(y);
            }
        };
        struct L2
        {
            static __device__ __forceinline__ float calc(int x, int y)
            {
                return ::sqrtf(x * x + y * y);
            }
        };

        template <typename Norm> __global__ void calcMagnitude(const PtrStepi dx_buf, const PtrStepi dy_buf, 
            PtrStepi dx, PtrStepi dy, PtrStepf mag, int rows, int cols)
        {
            __shared__ int sdx[18][16];
            __shared__ int sdy[18][16];

            const int j = blockIdx.x * blockDim.x + threadIdx.x;
            const int i = blockIdx.y * blockDim.y + threadIdx.y;

            if (j < cols)
            {
                sdx[threadIdx.y + 1][threadIdx.x] = dx_buf.ptr(i)[j];
                sdy[threadIdx.y + 1][threadIdx.x] = dy_buf.ptr(i)[j];
                if (threadIdx.y == 0)
                {
                    sdx[0][threadIdx.x] = dx_buf.ptr(::max(i - 1, 0))[j];
                    sdx[17][threadIdx.x] = dx_buf.ptr(::min(i + 16, rows - 1))[j];

                    sdy[0][threadIdx.x] = dy_buf.ptr(::max(i - 1, 0))[j];
                    sdy[17][threadIdx.x] = dy_buf.ptr(::min(i + 16, rows - 1))[j];
                }
                __syncthreads();

                if (i < rows)
                {
                    int x = sdx[threadIdx.y][threadIdx.x] + 2 * sdx[threadIdx.y + 1][threadIdx.x] + sdx[threadIdx.y + 2][threadIdx.x];
                    int y = -sdy[threadIdx.y][threadIdx.x] + sdy[threadIdx.y + 2][threadIdx.x];

                    dx.ptr(i)[j] = x;
                    dy.ptr(i)[j] = y;

                    mag.ptr(i + 1)[j + 1] = Norm::calc(x, y);
                }
            }
        }

        void calcMagnitude_gpu(PtrStepi dx_buf, PtrStepi dy_buf, PtrStepi dx, PtrStepi dy, PtrStepf mag, int rows, int cols, bool L2Grad)
        {
            dim3 block(16, 16, 1);
            dim3 grid(divUp(cols, block.x), divUp(rows, block.y), 1);

            if (L2Grad)
                calcMagnitude<L2><<<grid, block>>>(dx_buf, dy_buf, dx, dy, mag, rows, cols);
            else
                calcMagnitude<L1><<<grid, block>>>(dx_buf, dy_buf, dx, dy, mag, rows, cols);

            cudaSafeCall( hipGetLastError() );

            cudaSafeCall(hipDeviceSynchronize());
        }

        template <typename Norm> __global__ void calcMagnitude(PtrStepi dx, PtrStepi dy, PtrStepf mag, int rows, int cols)
        {
            const int j = blockIdx.x * blockDim.x + threadIdx.x;
            const int i = blockIdx.y * blockDim.y + threadIdx.y;

            if (i < rows && j < cols)
                mag.ptr(i + 1)[j + 1] = Norm::calc(dx.ptr(i)[j], dy.ptr(i)[j]);
        }

        void calcMagnitude_gpu(PtrStepi dx, PtrStepi dy, PtrStepf mag, int rows, int cols, bool L2Grad)
        {
            dim3 block(16, 16, 1);
            dim3 grid(divUp(cols, block.x), divUp(rows, block.y), 1);

            if (L2Grad)
                calcMagnitude<L2><<<grid, block>>>(dx, dy, mag, rows, cols);
            else
                calcMagnitude<L1><<<grid, block>>>(dx, dy, mag, rows, cols);

            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        //////////////////////////////////////////////////////////////////////////////////////////
            
        #define CANNY_SHIFT 15
        #define TG22        (int)(0.4142135623730950488016887242097*(1<<CANNY_SHIFT) + 0.5)

        __global__ void calcMap(const PtrStepi dx, const PtrStepi dy, const PtrStepf mag, PtrStepi map, int rows, int cols, float low_thresh, float high_thresh)
        {
            __shared__ float smem[18][18];

            const int j = blockIdx.x * 16 + threadIdx.x;
            const int i = blockIdx.y * 16 + threadIdx.y;

            const int tid = threadIdx.y * 16 + threadIdx.x;
            const int lx = tid % 18;
            const int ly = tid / 18;

            if (ly < 14)
                smem[ly][lx] = mag.ptr(blockIdx.y * 16 + ly)[blockIdx.x * 16 + lx];

            if (ly < 4 && blockIdx.y * 16 + ly + 14 <= rows && blockIdx.x * 16 + lx <= cols)
                smem[ly + 14][lx] = mag.ptr(blockIdx.y * 16 + ly + 14)[blockIdx.x * 16 + lx];

            __syncthreads();

            if (i < rows && j < cols)
            {
                int x = dx.ptr(i)[j];
                int y = dy.ptr(i)[j];
                const int s = (x ^ y) < 0 ? -1 : 1;
                const float m = smem[threadIdx.y + 1][threadIdx.x + 1];

                x = ::abs(x);
                y = ::abs(y);

                // 0 - the pixel can not belong to an edge
                // 1 - the pixel might belong to an edge
                // 2 - the pixel does belong to an edge
                int edge_type = 0;

                if (m > low_thresh)
                {
                    const int tg22x = x * TG22;
                    const int tg67x = tg22x + ((x + x) << CANNY_SHIFT);

                    y <<= CANNY_SHIFT;

                    if (y < tg22x)
                    {
                        if (m > smem[threadIdx.y + 1][threadIdx.x] && m >= smem[threadIdx.y + 1][threadIdx.x + 2])
                            edge_type = 1 + (int)(m > high_thresh);
                    }
                    else if( y > tg67x )
                    {
                        if (m > smem[threadIdx.y][threadIdx.x + 1] && m >= smem[threadIdx.y + 2][threadIdx.x + 1])
                            edge_type = 1 + (int)(m > high_thresh);
                    }
                    else
                    {
                        if (m > smem[threadIdx.y][threadIdx.x + 1 - s] && m > smem[threadIdx.y + 2][threadIdx.x + 1 + s])
                            edge_type = 1 + (int)(m > high_thresh);
                    }
                }
                
                map.ptr(i + 1)[j + 1] = edge_type;
            }
        }

        #undef CANNY_SHIFT
        #undef TG22

        void calcMap_gpu(PtrStepi dx, PtrStepi dy, PtrStepf mag, PtrStepi map, int rows, int cols, float low_thresh, float high_thresh)
        {
            dim3 block(16, 16, 1);
            dim3 grid(divUp(cols, block.x), divUp(rows, block.y), 1);

            calcMap<<<grid, block>>>(dx, dy, mag, map, rows, cols, low_thresh, high_thresh);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        //////////////////////////////////////////////////////////////////////////////////////////

        __device__ unsigned int counter = 0;

        __global__ void edgesHysteresisLocal(PtrStepi map, ushort2* st, int rows, int cols)
        {
            #if __CUDA_ARCH__ >= 120

            __shared__ int smem[18][18];

            const int j = blockIdx.x * 16 + threadIdx.x;
            const int i = blockIdx.y * 16 + threadIdx.y;

            const int tid = threadIdx.y * 16 + threadIdx.x;
            const int lx = tid % 18;
            const int ly = tid / 18; 

            if (ly < 14)
                smem[ly][lx] = map.ptr(blockIdx.y * 16 + ly)[blockIdx.x * 16 + lx];

            if (ly < 4 && blockIdx.y * 16 + ly + 14 <= rows && blockIdx.x * 16 + lx <= cols)
                smem[ly + 14][lx] = map.ptr(blockIdx.y * 16 + ly + 14)[blockIdx.x * 16 + lx];

            __syncthreads();

            if (i < rows && j < cols)
            {
                int n;

                #pragma unroll
                for (int k = 0; k < 16; ++k)
                {
                    n = 0;

                    if (smem[threadIdx.y + 1][threadIdx.x + 1] == 1)
                    {
                        n += smem[threadIdx.y    ][threadIdx.x    ] == 2;
                        n += smem[threadIdx.y    ][threadIdx.x + 1] == 2;
                        n += smem[threadIdx.y    ][threadIdx.x + 2] == 2;
                        
                        n += smem[threadIdx.y + 1][threadIdx.x    ] == 2;
                        n += smem[threadIdx.y + 1][threadIdx.x + 2] == 2;
                        
                        n += smem[threadIdx.y + 2][threadIdx.x    ] == 2;
                        n += smem[threadIdx.y + 2][threadIdx.x + 1] == 2;
                        n += smem[threadIdx.y + 2][threadIdx.x + 2] == 2;
                    }

                    if (n > 0)
                        smem[threadIdx.y + 1][threadIdx.x + 1] = 2;
                }

                const int e = smem[threadIdx.y + 1][threadIdx.x + 1];

                map.ptr(i + 1)[j + 1] = e;

                n = 0;

                if (e == 2)
                {
                    n += smem[threadIdx.y    ][threadIdx.x    ] == 1;
                    n += smem[threadIdx.y    ][threadIdx.x + 1] == 1;
                    n += smem[threadIdx.y    ][threadIdx.x + 2] == 1;
                    
                    n += smem[threadIdx.y + 1][threadIdx.x    ] == 1;
                    n += smem[threadIdx.y + 1][threadIdx.x + 2] == 1;
                    
                    n += smem[threadIdx.y + 2][threadIdx.x    ] == 1;
                    n += smem[threadIdx.y + 2][threadIdx.x + 1] == 1;
                    n += smem[threadIdx.y + 2][threadIdx.x + 2] == 1;
                }

                if (n > 0)
                {
                    const unsigned int ind = atomicInc(&counter, (unsigned int)(-1));
                    st[ind] = make_ushort2(j + 1, i + 1);
                }
            }

            #endif
        }

        void edgesHysteresisLocal_gpu(PtrStepi map, ushort2* st1, int rows, int cols)
        {
            void* counter_ptr;
            cudaSafeCall( hipGetSymbolAddress(&counter_ptr, HIP_SYMBOL(counter)) );

            cudaSafeCall( hipMemset(counter_ptr, 0, sizeof(unsigned int)) );

            dim3 block(16, 16, 1);
            dim3 grid(divUp(cols, block.x), divUp(rows, block.y), 1);

            edgesHysteresisLocal<<<grid, block>>>(map, st1, rows, cols);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        __constant__ int c_dx[8] = {-1,  0,  1, -1, 1, -1, 0, 1};
        __constant__ int c_dy[8] = {-1, -1, -1,  0, 0,  1, 1, 1};

        __global__ void edgesHysteresisGlobal(PtrStepi map, ushort2* st1, ushort2* st2, int rows, int cols, int count)
        {
            #if __CUDA_ARCH__ >= 120

            const int stack_size = 512;
            
            __shared__ unsigned int s_counter;
            __shared__ unsigned int s_ind;
            __shared__ ushort2 s_st[stack_size];

            if (threadIdx.x == 0)
                s_counter = 0;
            __syncthreads();

            int ind = blockIdx.y * gridDim.x + blockIdx.x;

            if (ind < count)
            {
                ushort2 pos = st1[ind];

                if (pos.x > 0 && pos.x <= cols && pos.y > 0 && pos.y <= rows)
                {
                    if (threadIdx.x < 8)
                    {
                        pos.x += c_dx[threadIdx.x];
                        pos.y += c_dy[threadIdx.x];

                        if (map.ptr(pos.y)[pos.x] == 1)
                        {
                            map.ptr(pos.y)[pos.x] = 2;

                            ind = atomicInc(&s_counter, (unsigned int)(-1));

                            s_st[ind] = pos;
                        }
                    }
                    __syncthreads();

                    while (s_counter > 0 && s_counter <= stack_size - blockDim.x)
                    {
                        const int subTaskIdx = threadIdx.x >> 3;
                        const int portion = ::min(s_counter, blockDim.x >> 3);

                        pos.x = pos.y = 0;

                        if (subTaskIdx < portion)
                            pos = s_st[s_counter - 1 - subTaskIdx];
                        __syncthreads();
                            
                        if (threadIdx.x == 0)
                            s_counter -= portion;
                        __syncthreads();
                         
                        if (pos.x > 0 && pos.x <= cols && pos.y > 0 && pos.y <= rows)
                        {
                            pos.x += c_dx[threadIdx.x & 7];
                            pos.y += c_dy[threadIdx.x & 7];

                            if (map.ptr(pos.y)[pos.x] == 1)
                            {
                                map.ptr(pos.y)[pos.x] = 2;

                                ind = atomicInc(&s_counter, (unsigned int)(-1));

                                s_st[ind] = pos;
                            }
                        }
                        __syncthreads();
                    }

                    if (s_counter > 0)
                    {
                        if (threadIdx.x == 0)
                        {
                            ind = atomicAdd(&counter, s_counter);
                            s_ind = ind - s_counter;
                        }
                        __syncthreads();

                        ind = s_ind;

                        for (int i = threadIdx.x; i < s_counter; i += blockDim.x)
                        {
                            st2[ind + i] = s_st[i];
                        }
                    }
                }
            }

            #endif
        }

        void edgesHysteresisGlobal_gpu(PtrStepi map, ushort2* st1, ushort2* st2, int rows, int cols)
        {
            void* counter_ptr;
            cudaSafeCall( hipGetSymbolAddress(&counter_ptr, HIP_SYMBOL(counter)) );
            
            unsigned int count;
            cudaSafeCall( hipMemcpy(&count, counter_ptr, sizeof(unsigned int), hipMemcpyDeviceToHost) );

            while (count > 0)
            {
                cudaSafeCall( hipMemset(counter_ptr, 0, sizeof(unsigned int)) );

                dim3 block(128, 1, 1);
                dim3 grid(std::min(count, 65535u), divUp(count, 65535), 1);
                edgesHysteresisGlobal<<<grid, block>>>(map, st1, st2, rows, cols, count);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                cudaSafeCall( hipMemcpy(&count, counter_ptr, sizeof(unsigned int), hipMemcpyDeviceToHost) );

                std::swap(st1, st2);
            }
        }

        __global__ void getEdges(PtrStepi map, PtrStepb dst, int rows, int cols)
        {
            const int j = blockIdx.x * 16 + threadIdx.x;
            const int i = blockIdx.y * 16 + threadIdx.y;

            if (i < rows && j < cols)
                dst.ptr(i)[j] = (uchar)(-(map.ptr(i + 1)[j + 1] >> 1));
        }

        void getEdges_gpu(PtrStepi map, PtrStepb dst, int rows, int cols)
        {
            dim3 block(16, 16, 1);
            dim3 grid(divUp(cols, block.x), divUp(rows, block.y), 1);

            getEdges<<<grid, block>>>(map, dst, rows, cols);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }
    } // namespace canny
}}} // namespace cv { namespace gpu { namespace device
