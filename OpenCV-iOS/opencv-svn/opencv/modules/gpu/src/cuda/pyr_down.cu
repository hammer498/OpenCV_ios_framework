#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/border_interpolate.hpp"
#include "opencv2/gpu/device/vec_traits.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace imgproc
    {
        template <typename T, typename B> __global__ void pyrDown(const PtrStep<T> src, PtrStep<T> dst, const B b, int dst_cols)
        {
            typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type value_type;

            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y;

            __shared__ value_type smem[256 + 4];

            value_type sum;

            const int src_y = 2*y;

            sum = VecTraits<value_type>::all(0);

            sum = sum + 0.0625f * b.at(src_y - 2, x, src.data, src.step);
            sum = sum + 0.25f   * b.at(src_y - 1, x, src.data, src.step);
            sum = sum + 0.375f  * b.at(src_y    , x, src.data, src.step);
            sum = sum + 0.25f   * b.at(src_y + 1, x, src.data, src.step);
            sum = sum + 0.0625f * b.at(src_y + 2, x, src.data, src.step);

            smem[2 + threadIdx.x] = sum;

            if (threadIdx.x < 2)
            {
                const int left_x = x - 2;

                sum = VecTraits<value_type>::all(0);

                sum = sum + 0.0625f * b.at(src_y - 2, left_x, src.data, src.step);
                sum = sum + 0.25f   * b.at(src_y - 1, left_x, src.data, src.step);
                sum = sum + 0.375f  * b.at(src_y    , left_x, src.data, src.step);
                sum = sum + 0.25f   * b.at(src_y + 1, left_x, src.data, src.step);
                sum = sum + 0.0625f * b.at(src_y + 2, left_x, src.data, src.step);

                smem[threadIdx.x] = sum;
            }

            if (threadIdx.x > 253)
            {
                const int right_x = x + 2;

                sum = VecTraits<value_type>::all(0);

                sum = sum + 0.0625f * b.at(src_y - 2, right_x, src.data, src.step);
                sum = sum + 0.25f   * b.at(src_y - 1, right_x, src.data, src.step);
                sum = sum + 0.375f  * b.at(src_y    , right_x, src.data, src.step);
                sum = sum + 0.25f   * b.at(src_y + 1, right_x, src.data, src.step);
                sum = sum + 0.0625f * b.at(src_y + 2, right_x, src.data, src.step);

                smem[4 + threadIdx.x] = sum;
            }

            __syncthreads();

            if (threadIdx.x < 128)
            {
                const int tid2 = threadIdx.x * 2;

                sum = VecTraits<value_type>::all(0);

                sum = sum + 0.0625f * smem[2 + tid2 - 2];
                sum = sum + 0.25f   * smem[2 + tid2 - 1];
                sum = sum + 0.375f  * smem[2 + tid2    ];
                sum = sum + 0.25f   * smem[2 + tid2 + 1];
                sum = sum + 0.0625f * smem[2 + tid2 + 2];

                const int dst_x = (blockIdx.x * blockDim.x + tid2) / 2;

                if (dst_x < dst_cols)
                    dst.ptr(y)[dst_x] = saturate_cast<T>(sum);
            }
        }

        template <typename T, template <typename> class B> void pyrDown_caller(DevMem2D_<T> src, DevMem2D_<T> dst, hipStream_t stream)
        {
            const dim3 block(256);
            const dim3 grid(divUp(src.cols, block.x), dst.rows);

            B<T> b(src.rows, src.cols);

            pyrDown<T><<<grid, block, 0, stream>>>(src, dst, b, dst.cols);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <typename T> void pyrDown_gpu(DevMem2Db src, DevMem2Db dst, hipStream_t stream)
        {
            pyrDown_caller<T, BrdReflect101>(static_cast< DevMem2D_<T> >(src), static_cast< DevMem2D_<T> >(dst), stream);
        }

        template void pyrDown_gpu<uchar>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        //template void pyrDown_gpu<uchar2>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void pyrDown_gpu<uchar3>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void pyrDown_gpu<uchar4>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);

        //template void pyrDown_gpu<schar>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        //template void pyrDown_gpu<char2>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        //template void pyrDown_gpu<char3>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        //template void pyrDown_gpu<char4>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);

        template void pyrDown_gpu<ushort>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        //template void pyrDown_gpu<ushort2>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void pyrDown_gpu<ushort3>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void pyrDown_gpu<ushort4>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);

        template void pyrDown_gpu<short>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        //template void pyrDown_gpu<short2>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void pyrDown_gpu<short3>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void pyrDown_gpu<short4>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);

        //template void pyrDown_gpu<int>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        //template void pyrDown_gpu<int2>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        //template void pyrDown_gpu<int3>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        //template void pyrDown_gpu<int4>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);

        template void pyrDown_gpu<float>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        //template void pyrDown_gpu<float2>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void pyrDown_gpu<float3>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void pyrDown_gpu<float4>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
    } // namespace imgproc
}}} // namespace cv { namespace gpu { namespace device
