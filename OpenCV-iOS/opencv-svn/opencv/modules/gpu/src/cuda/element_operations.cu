#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/functional.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/transform.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"

namespace cv { namespace gpu { namespace device
{
    //////////////////////////////////////////////////////////////////////////
    // add

    template <typename T, typename D> struct Add : binary_function<T, T, D>
    {
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return saturate_cast<D>(a + b);
        }
    };

    template <> struct TransformFunctorTraits< Add<ushort, ushort> > : DefaultTransformFunctorTraits< Add<ushort, ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Add<short, short> > : DefaultTransformFunctorTraits< Add<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Add<int, int> > : DefaultTransformFunctorTraits< Add<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Add<float, float> > : DefaultTransformFunctorTraits< Add<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> void add_gpu(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream)
    {
        if (mask.data)
            cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<T>)src2, (DevMem2D_<D>)dst, Add<T, D>(), SingleMask(mask), stream);
        else
            cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<T>)src2, (DevMem2D_<D>)dst, Add<T, D>(), WithOutMask(), stream);
    }

    template void add_gpu<uchar, uchar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<uchar, schar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, short>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, int>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, float>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<schar, uchar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, schar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, short>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, int>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, float>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<ushort, uchar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<ushort, schar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<ushort, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<ushort, short>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<ushort, int>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<ushort, float>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<ushort, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<short, uchar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<short, schar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<short, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<short, short>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<short, int>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<short, float>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<short, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<int, uchar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<int, schar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<int, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<int, short>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<int, int>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<int, float>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<int, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<float, uchar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<float, schar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<float, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<float, short>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<float, int>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<float, float>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<float, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<double, uchar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, schar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, short>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, int>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, float>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<double, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    template <typename T, typename D> struct AddScalar : unary_function<T, D>
    {
        AddScalar(double val_) : val(val_) {}
        __device__ __forceinline__ D operator ()(T a) const
        {
            return saturate_cast<D>(a + val);
        }
        const double val;
    };

    template <> struct TransformFunctorTraits< AddScalar<ushort, ushort> > : DefaultTransformFunctorTraits< AddScalar<ushort, ushort>  >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddScalar<short, short> > : DefaultTransformFunctorTraits< AddScalar<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddScalar<int, int> > : DefaultTransformFunctorTraits< AddScalar<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddScalar<float, float> > : DefaultTransformFunctorTraits< AddScalar<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> void add_gpu(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&val) );
        AddScalar<T, D> op(val);
        if (mask.data)
            cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<D>)dst, op, SingleMask(mask), stream);
        else
            cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<D>)dst, op, WithOutMask(), stream);
    }

    template void add_gpu<uchar, uchar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<uchar, schar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, short >(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, int   >(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, float >(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<schar, uchar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, schar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, short>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, int>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, float>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<ushort, uchar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<ushort, schar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<ushort, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<ushort, short>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<ushort, int>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<ushort, float>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<ushort, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<short, uchar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<short, schar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<short, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<short, short>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<short, int>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<short, float>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<short, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<int, uchar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<int, schar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<int, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<int, short>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<int, int>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<int, float>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<int, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<float, uchar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<float, schar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<float, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<float, short>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<float, int>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<float, float>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<float, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<double, uchar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, schar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, short>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, int>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, float>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<double, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // subtract

    template <typename T, typename D> struct Subtract : binary_function<T, T, D>
    {
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return saturate_cast<D>(a - b);
        }
    };

    template <> struct TransformFunctorTraits< Subtract<ushort, ushort> > : DefaultTransformFunctorTraits< Subtract<ushort, ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Subtract<short, short> > : DefaultTransformFunctorTraits< Subtract<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Subtract<int, int> > : DefaultTransformFunctorTraits< Subtract<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Subtract<float, float> > : DefaultTransformFunctorTraits< Subtract<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> void subtract_gpu(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream)
    {
        if (mask.data)
            cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<T>)src2, (DevMem2D_<D>)dst, Subtract<T, D>(), SingleMask(mask), stream);
        else
            cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<T>)src2, (DevMem2D_<D>)dst, Subtract<T, D>(), WithOutMask(), stream);
    }

    template void subtract_gpu<uchar, uchar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<uchar, schar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, short>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, int>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, float>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<schar, uchar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, schar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, short>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, int>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, float>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<ushort, uchar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<ushort, schar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<ushort, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<ushort, short>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<ushort, int>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<ushort, float>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<ushort, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<short, uchar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<short, schar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<short, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<short, short>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<short, int>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<short, float>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<short, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<int, uchar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<int, schar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<int, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<int, short>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<int, int>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<int, float>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<int, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<float, uchar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<float, schar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<float, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<float, short>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<float, int>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<float, float>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<float, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<double, uchar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, schar>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, short>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, int>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, float>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<double, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    template <typename T, typename D> struct SubtractScalar : unary_function<T, D>
    {
        SubtractScalar(double val_) : val(val_) {}
        __device__ __forceinline__ D operator ()(T a) const
        {
            return saturate_cast<D>(a - val);
        }
        const double val;
    };

    template <> struct TransformFunctorTraits< SubtractScalar<ushort, ushort> > : DefaultTransformFunctorTraits< SubtractScalar<ushort, ushort>  >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< SubtractScalar<short, short> > : DefaultTransformFunctorTraits< SubtractScalar<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< SubtractScalar<int, int> > : DefaultTransformFunctorTraits< SubtractScalar<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< SubtractScalar<float, float> > : DefaultTransformFunctorTraits< SubtractScalar<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> void subtract_gpu(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&val) );
        SubtractScalar<T, D> op(val);
        if (mask.data)
            cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<D>)dst, op, SingleMask(mask), stream);
        else
            cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<D>)dst, op, WithOutMask(), stream);
    }

    template void subtract_gpu<uchar, uchar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<uchar, schar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, short >(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, int   >(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, float >(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<schar, uchar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, schar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, short>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, int>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, float>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<ushort, uchar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<ushort, schar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<ushort, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<ushort, short>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<ushort, int>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<ushort, float>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<ushort, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<short, uchar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<short, schar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<short, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<short, short>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<short, int>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<short, float>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<short, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<int, uchar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<int, schar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<int, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<int, short>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<int, int>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<int, float>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<int, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<float, uchar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<float, schar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<float, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<float, short>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<float, int>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<float, float>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<float, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<double, uchar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, schar>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, short>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, int>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, float>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<double, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, const PtrStepb& mask, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // multiply

    struct multiply_8uc4_32f : binary_function<uint, float, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, float b) const
        {
            uint res = 0;

            res |= (saturate_cast<uchar>((0xffu & (a      )) * b)      );
            res |= (saturate_cast<uchar>((0xffu & (a >>  8)) * b) <<  8);
            res |= (saturate_cast<uchar>((0xffu & (a >> 16)) * b) << 16);
            res |= (saturate_cast<uchar>((0xffu & (a >> 24)) * b) << 24);

            return res;
        }
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(multiply_8uc4_32f)
    {
        enum { smart_block_dim_x = 8 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 8 };
    };

    void multiply_gpu(const DevMem2D_<uchar4>& src1, const DevMem2Df& src2, const DevMem2D_<uchar4>& dst, hipStream_t stream)
    {
        cv::gpu::device::transform(static_cast< DevMem2D_<uint> >(src1), src2, static_cast< DevMem2D_<uint> >(dst), multiply_8uc4_32f(), WithOutMask(), stream);
    }

    struct multiply_16sc4_32f : binary_function<short4, float, short4>
    {
        __device__ __forceinline__ short4 operator ()(short4 a, float b) const
        {
            return make_short4(saturate_cast<short>(a.x * b), saturate_cast<short>(a.y * b),
                               saturate_cast<short>(a.z * b), saturate_cast<short>(a.w * b));
        }
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(multiply_16sc4_32f)
    {
        enum { smart_block_dim_x = 8 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 8 };
    };

    void multiply_gpu(const DevMem2D_<short4>& src1, const DevMem2Df& src2, const DevMem2D_<short4>& dst, hipStream_t stream)
    {
        cv::gpu::device::transform(static_cast< DevMem2D_<short4> >(src1), src2, static_cast< DevMem2D_<short4> >(dst), multiply_16sc4_32f(), WithOutMask(), stream);
    }

    template <typename T, typename D> struct Multiply : binary_function<T, T, D>
    {
        Multiply(float scale_) : scale(scale_) {}
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return saturate_cast<D>(scale * a * b);
        }
        const float scale;
    };
    template <typename T> struct Multiply<T, double> : binary_function<T, T, double>
    {
        Multiply(double scale_) : scale(scale_) {}
        __device__ __forceinline__ double operator ()(T a, T b) const
        {
            return scale * a * b;
        }
        const double scale;
    };
    template <> struct Multiply<int, int> : binary_function<int, int, int>
    {
        Multiply(double scale_) : scale(scale_) {}
        __device__ __forceinline__ int operator ()(int a, int b) const
        {
            return saturate_cast<int>(scale * a * b);
        }
        const double scale;
    };

    template <> struct TransformFunctorTraits< Multiply<ushort, ushort> > : DefaultTransformFunctorTraits< Multiply<ushort, ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Multiply<short, short> > : DefaultTransformFunctorTraits< Multiply<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Multiply<int, int> > : DefaultTransformFunctorTraits< Multiply<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Multiply<float, float> > : DefaultTransformFunctorTraits< Multiply<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> struct MultiplyCaller
    {
        static void call(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream)
        {
            Multiply<T, D> op(static_cast<float>(scale));
            cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<T>)src2, (DevMem2D_<D>)dst, op, WithOutMask(), stream);
        }
    };
    template <typename T> struct MultiplyCaller<T, double>
    {
        static void call(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream)
        {
            cudaSafeCall( cudaSetDoubleForDevice(&scale) );
            Multiply<T, double> op(scale);
            cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<T>)src2, (DevMem2D_<double>)dst, op, WithOutMask(), stream);
        }
    };
    template <> struct MultiplyCaller<int, int>
    {
        static void call(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream)
        {
            cudaSafeCall( cudaSetDoubleForDevice(&scale) );
            Multiply<int, int> op(scale);
            cv::gpu::device::transform((DevMem2D_<int>)src1, (DevMem2D_<int>)src2, (DevMem2D_<int>)dst, op, WithOutMask(), stream);
        }
    };

    template <typename T, typename D> void multiply_gpu(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream)
    {
        MultiplyCaller<T, D>::call(src1, src2, dst, scale, stream);
    }

    template void multiply_gpu<uchar, uchar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<uchar, schar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, short >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, int   >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, float >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<schar, uchar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, schar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, short >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, int   >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, float >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<ushort, uchar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<ushort, schar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<ushort, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<ushort, short >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<ushort, int   >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<ushort, float >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<ushort, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<short, uchar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<short, schar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<short, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<short, short >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<short, int   >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<short, float >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<short, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<int, uchar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<int, schar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<int, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<int, short >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<int, int   >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<int, float >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<int, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<float, uchar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<float, schar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<float, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<float, short >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<float, int   >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<float, float >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<float, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<double, uchar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, schar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, short >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, int   >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, float >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<double, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);

    template <typename T, typename D> struct MultiplyScalar : unary_function<T, D>
    {
        MultiplyScalar(double val_, double scale_) : val(val_), scale(scale_) {}
        __device__ __forceinline__ D operator ()(T a) const
        {
            return saturate_cast<D>(scale * a * val);
        }
        const double val;
        const double scale;
    };

    template <> struct TransformFunctorTraits< MultiplyScalar<ushort, ushort> > : DefaultTransformFunctorTraits< MultiplyScalar<ushort, ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< MultiplyScalar<short, short> > : DefaultTransformFunctorTraits< MultiplyScalar<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< MultiplyScalar<int, int> > : DefaultTransformFunctorTraits< MultiplyScalar<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< MultiplyScalar<float, float> > : DefaultTransformFunctorTraits< MultiplyScalar<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> void multiply_gpu(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&val) );
        cudaSafeCall( cudaSetDoubleForDevice(&scale) );
        MultiplyScalar<T, D> op(val, scale);
        cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<D>)dst, op, WithOutMask(), stream);
    }

    template void multiply_gpu<uchar, uchar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<uchar, schar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, short >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, int   >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, float >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<schar, uchar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, schar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, short >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, int   >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, float >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<ushort, uchar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<ushort, schar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<ushort, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<ushort, short >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<ushort, int   >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<ushort, float >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<ushort, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<short, uchar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<short, schar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<short, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<short, short >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<short, int   >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<short, float >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<short, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<int, uchar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<int, schar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<int, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<int, short >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<int, int   >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<int, float >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<int, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<float, uchar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<float, schar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<float, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<float, short >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<float, int   >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<float, float >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<float, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<double, uchar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, schar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, short >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, int   >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, float >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void multiply_gpu<double, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // divide

    struct divide_8uc4_32f : binary_function<uchar4, float, uchar4>
    {
        __device__ __forceinline__ uchar4 operator ()(uchar4 a, float b) const
        {
            return b != 0 ? make_uchar4(saturate_cast<uchar>(a.x / b), saturate_cast<uchar>(a.y / b),
                                        saturate_cast<uchar>(a.z / b), saturate_cast<uchar>(a.w / b))
                          : make_uchar4(0,0,0,0);
        }
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(divide_8uc4_32f)
    {
        enum { smart_block_dim_x = 8 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 8 };
    };

    void divide_gpu(const DevMem2D_<uchar4>& src1, const DevMem2Df& src2, const DevMem2D_<uchar4>& dst, hipStream_t stream)
    {
        cv::gpu::device::transform(static_cast< DevMem2D_<uchar4> >(src1), src2, static_cast< DevMem2D_<uchar4> >(dst), divide_8uc4_32f(), WithOutMask(), stream);
    }


    struct divide_16sc4_32f : binary_function<short4, float, short4>
    {
        __device__ __forceinline__ short4 operator ()(short4 a, float b) const
        {
            return b != 0 ? make_short4(saturate_cast<short>(a.x / b), saturate_cast<short>(a.y / b),
                                        saturate_cast<short>(a.z / b), saturate_cast<short>(a.w / b))
                          : make_short4(0,0,0,0);
        }
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(divide_16sc4_32f)
    {
        enum { smart_block_dim_x = 8 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 8 };
    };

    void divide_gpu(const DevMem2D_<short4>& src1, const DevMem2Df& src2, const DevMem2D_<short4>& dst, hipStream_t stream)
    {
        cv::gpu::device::transform(static_cast< DevMem2D_<short4> >(src1), src2, static_cast< DevMem2D_<short4> >(dst), divide_16sc4_32f(), WithOutMask(), stream);
    }

    template <typename T, typename D> struct Divide : binary_function<T, T, D>
    {
        Divide(double scale_) : scale(scale_) {}
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return b != 0 ? saturate_cast<D>(a * scale / b) : 0;
        }
        const double scale;
    };

    template <> struct TransformFunctorTraits< Divide<ushort, ushort> > : DefaultTransformFunctorTraits< Divide<ushort, ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Divide<short, short> > : DefaultTransformFunctorTraits< Divide<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Divide<int, int> > : DefaultTransformFunctorTraits< Divide<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Divide<float, float> > : DefaultTransformFunctorTraits< Divide<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> void divide_gpu(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&scale) );
        Divide<T, D> op(scale);
        cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<T>)src2, (DevMem2D_<D>)dst, op, WithOutMask(), stream);
    }

    template void divide_gpu<uchar, uchar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<uchar, schar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, short >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, int   >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, float >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void divide_gpu<schar, uchar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, schar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, short >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, int   >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, float >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void divide_gpu<ushort, uchar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<ushort, schar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<ushort, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<ushort, short >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<ushort, int   >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<ushort, float >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<ushort, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void divide_gpu<short, uchar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<short, schar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<short, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<short, short >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<short, int   >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<short, float >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<short, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void divide_gpu<int, uchar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<int, schar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<int, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<int, short >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<int, int   >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<int, float >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<int, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void divide_gpu<float, uchar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<float, schar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<float, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<float, short >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<float, int   >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<float, float >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<float, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void divide_gpu<double, uchar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, schar >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, ushort>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, short >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, int   >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, float >(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<double, double>(const DevMem2Db& src1, const DevMem2Db& src2, const DevMem2Db& dst, double scale, hipStream_t stream);

    template <typename T, typename D> struct DivideScalar : unary_function<T, D>
    {
        DivideScalar(double val_, double scale_) : val(val_), scale(scale_) {}
        __device__ __forceinline__ D operator ()(T a) const
        {
            return saturate_cast<D>(scale * a / val);
        }
        const double val;
        const double scale;
    };

    template <> struct TransformFunctorTraits< DivideScalar<ushort, ushort> > : DefaultTransformFunctorTraits< DivideScalar<ushort, ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< DivideScalar<short, short> > : DefaultTransformFunctorTraits< DivideScalar<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< DivideScalar<int, int> > : DefaultTransformFunctorTraits< DivideScalar<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< DivideScalar<float, float> > : DefaultTransformFunctorTraits< DivideScalar<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> void divide_gpu(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&val) );
        cudaSafeCall( cudaSetDoubleForDevice(&scale) );
        DivideScalar<T, D> op(val, scale);
        cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<D>)dst, op, WithOutMask(), stream);
    }

    template void divide_gpu<uchar, uchar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<uchar, schar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, short >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, int   >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, float >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void divide_gpu<schar, uchar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, schar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, short >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, int   >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, float >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void divide_gpu<ushort, uchar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<ushort, schar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<ushort, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<ushort, short >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<ushort, int   >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<ushort, float >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<ushort, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void divide_gpu<short, uchar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<short, schar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<short, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<short, short >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<short, int   >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<short, float >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<short, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void divide_gpu<int, uchar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<int, schar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<int, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<int, short >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<int, int   >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<int, float >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<int, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void divide_gpu<float, uchar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<float, schar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<float, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<float, short >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<float, int   >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<float, float >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<float, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);

    //template void divide_gpu<double, uchar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, schar >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, ushort>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, short >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, int   >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, float >(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);
    template void divide_gpu<double, double>(const DevMem2Db& src1, double val, const DevMem2Db& dst, double scale, hipStream_t stream);

    template <typename T, typename D> struct Reciprocal : unary_function<T, D>
    {
        Reciprocal(double scale_) : scale(scale_) {}
        __device__ __forceinline__ D operator ()(T a) const
        {
            return a != 0 ? saturate_cast<D>(scale / a) : 0;
        }
        const double scale;
    };

    template <> struct TransformFunctorTraits< Reciprocal<ushort, ushort> > : DefaultTransformFunctorTraits< Reciprocal<ushort, ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Reciprocal<short, short> > : DefaultTransformFunctorTraits< Reciprocal<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Reciprocal<int, int> > : DefaultTransformFunctorTraits< Reciprocal<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Reciprocal<float, float> > : DefaultTransformFunctorTraits< Reciprocal<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> void divide_gpu(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&scalar) );
        Reciprocal<T, D> op(scalar);
        cv::gpu::device::transform((DevMem2D_<T>)src2, (DevMem2D_<D>)dst, op, WithOutMask(), stream);
    }

    template void divide_gpu<uchar, uchar >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<uchar, schar >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<uchar, ushort>(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<uchar, short >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<uchar, int   >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<uchar, float >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<uchar, double>(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);

    //template void divide_gpu<schar, uchar >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<schar, schar >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<schar, ushort>(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<schar, short >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<schar, int   >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<schar, float >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<schar, double>(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);

    //template void divide_gpu<ushort, uchar >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<ushort, schar >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<ushort, ushort>(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<ushort, short >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<ushort, int   >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<ushort, float >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<ushort, double>(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);

    //template void divide_gpu<short, uchar >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<short, schar >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<short, ushort>(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<short, short >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<short, int   >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<short, float >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<short, double>(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);

    //template void divide_gpu<int, uchar >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<int, schar >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<int, ushort>(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<int, short >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<int, int   >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<int, float >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<int, double>(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);

    //template void divide_gpu<float, uchar >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<float, schar >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<float, ushort>(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<float, short >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<float, int   >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<float, float >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<float, double>(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);

    //template void divide_gpu<double, uchar >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<double, schar >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<double, ushort>(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<double, short >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<double, int   >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    //template void divide_gpu<double, float >(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);
    template void divide_gpu<double, double>(double scalar, const DevMem2Db& src2, const DevMem2Db& dst, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // absdiff

    template <typename T> struct Absdiff : binary_function<T, T, T>
    {
        static __device__ __forceinline__ int abs(int a)
        {
            return ::abs(a);
        }
        static __device__ __forceinline__ float abs(float a)
        {
            return ::fabsf(a);
        }
        static __device__ __forceinline__ double abs(double a)
        {
            return ::fabs(a);
        }

        __device__ __forceinline__ T operator ()(T a, T b) const
        {
            return saturate_cast<T>(::abs(a - b));
        }
    };

    template <> struct TransformFunctorTraits< Absdiff<ushort> > : DefaultTransformFunctorTraits< Absdiff<ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Absdiff<short> > : DefaultTransformFunctorTraits< Absdiff<short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Absdiff<int> > : DefaultTransformFunctorTraits< Absdiff<int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Absdiff<float> > : DefaultTransformFunctorTraits< Absdiff<float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T> void absdiff_gpu(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream)
    {
        cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<T>)src2, (DevMem2D_<T>)dst, Absdiff<T>(), WithOutMask(), stream);
    }

    //template void absdiff_gpu<uchar >(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void absdiff_gpu<schar >(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    //template void absdiff_gpu<ushort>(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void absdiff_gpu<short >(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void absdiff_gpu<int   >(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    //template void absdiff_gpu<float >(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void absdiff_gpu<double>(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);

    template <typename T> struct AbsdiffScalar : unary_function<T, T>
    {
        AbsdiffScalar(double val_) : val(val_) {}
        __device__ __forceinline__ T operator ()(T a) const
        {
            return saturate_cast<T>(::fabs(a - val));
        }
        double val;
    };

    template <> struct TransformFunctorTraits< AbsdiffScalar<ushort> > : DefaultTransformFunctorTraits< AbsdiffScalar<ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AbsdiffScalar<short> > : DefaultTransformFunctorTraits< AbsdiffScalar<short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AbsdiffScalar<int> > : DefaultTransformFunctorTraits< AbsdiffScalar<int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AbsdiffScalar<float> > : DefaultTransformFunctorTraits< AbsdiffScalar<float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T> void absdiff_gpu(const DevMem2Db src1, double val, DevMem2Db dst, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&val) );
        AbsdiffScalar<T> op(val);
        cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<T>)dst, op, WithOutMask(), stream);
    }

    //template void absdiff_gpu<uchar >(const DevMem2Db src1, double src2, DevMem2Db dst, hipStream_t stream);
    template void absdiff_gpu<schar >(const DevMem2Db src1, double src2, DevMem2Db dst, hipStream_t stream);
    //template void absdiff_gpu<ushort>(const DevMem2Db src1, double src2, DevMem2Db dst, hipStream_t stream);
    template void absdiff_gpu<short >(const DevMem2Db src1, double src2, DevMem2Db dst, hipStream_t stream);
    template void absdiff_gpu<int   >(const DevMem2Db src1, double src2, DevMem2Db dst, hipStream_t stream);
    //template void absdiff_gpu<float >(const DevMem2Db src1, double src2, DevMem2Db dst, hipStream_t stream);
    template void absdiff_gpu<double>(const DevMem2Db src1, double src2, DevMem2Db dst, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////////////////
    // Compare

#define TYPE_VEC(type, cn) typename TypeVec<type, cn>::vec_type

    template <template <typename> class Op, typename T, int cn> struct Compare;
    template <template <typename> class Op, typename T>
    struct Compare<Op, T, 1>: binary_function<T, T, uchar>
    {
        __device__ __forceinline__ uchar operator()(T src1, T src2) const
        {
            Op<T> op;
            return static_cast<uchar>(static_cast<int>(op(src1, src2)) * 255);
        }
    };
    template <template <typename> class Op, typename T>
    struct Compare<Op, T, 2>: binary_function<TYPE_VEC(T, 2), TYPE_VEC(T, 2), TYPE_VEC(uchar, 2)>
    {
        __device__ __forceinline__ TYPE_VEC(uchar, 2) operator()(const TYPE_VEC(T, 2) & src1, const TYPE_VEC(T, 2) & src2) const
        {
            Op<T> op;
            return VecTraits<TYPE_VEC(uchar, 2)>::make(
                        static_cast<uchar>(static_cast<int>(op(src1.x, src2.x)) * 255),
                        static_cast<uchar>(static_cast<int>(op(src1.y, src2.y)) * 255));
        }
    };
    template <template <typename> class Op, typename T>
    struct Compare<Op, T, 3>: binary_function<TYPE_VEC(T, 3), TYPE_VEC(T, 3), TYPE_VEC(uchar, 3)>
    {
        __device__ __forceinline__ TYPE_VEC(uchar, 3) operator()(const TYPE_VEC(T, 3) & src1, const TYPE_VEC(T, 3) & src2) const
        {
            Op<T> op;
            return VecTraits<TYPE_VEC(uchar, 3)>::make(
                        static_cast<uchar>(static_cast<int>(op(src1.x, src2.x)) * 255),
                        static_cast<uchar>(static_cast<int>(op(src1.y, src2.y)) * 255),
                        static_cast<uchar>(static_cast<int>(op(src1.z, src2.z)) * 255));
        }
    };
    template <template <typename> class Op, typename T>
    struct Compare<Op, T, 4>: binary_function<TYPE_VEC(T, 4), TYPE_VEC(T, 4), TYPE_VEC(uchar, 4)>
    {
        __device__ __forceinline__ TYPE_VEC(uchar, 4) operator()(const TYPE_VEC(T, 4) & src1, const TYPE_VEC(T, 4) & src2) const
        {
            Op<T> op;
            return VecTraits<TYPE_VEC(uchar, 4)>::make(
                        static_cast<uchar>(static_cast<int>(op(src1.x, src2.x)) * 255),
                        static_cast<uchar>(static_cast<int>(op(src1.y, src2.y)) * 255),
                        static_cast<uchar>(static_cast<int>(op(src1.z, src2.z)) * 255),
                        static_cast<uchar>(static_cast<int>(op(src1.w, src2.w)) * 255));
        }
    };

#undef TYPE_VEC

#define IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(op, type, block_dim_y, shift) \
    template <> struct TransformFunctorTraits< Compare<op, type, 1> > : DefaultTransformFunctorTraits< Compare<op, type, 1> > \
    { \
        enum { smart_block_dim_y = block_dim_y }; \
        enum { smart_shift = shift }; \
    };

    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(equal_to, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(equal_to, float, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(not_equal_to, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(not_equal_to, float, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(greater, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(greater, float, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(less, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(less, float, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(greater_equal, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(greater_equal, float, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(less_equal, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(less_equal, float, 8, 4)

#undef IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS

    template <template <typename> class Op, typename T> void compare(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream)
    {
        Compare<Op, T, 1> op;
        cv::gpu::device::transform(static_cast< DevMem2D_<T> >(src1), static_cast< DevMem2D_<T> >(src2), dst, op, WithOutMask(), stream);
    }

    template <typename T> void compare_eq(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream)
    {
        compare<equal_to, T>(src1, src2, dst, stream);
    }
    template <typename T> void compare_ne(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream)
    {
        compare<not_equal_to, T>(src1, src2, dst, stream);
    }
    template <typename T> void compare_lt(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream)
    {
        compare<less, T>(src1, src2, dst, stream);
    }
    template <typename T> void compare_le(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream)
    {
        compare<less_equal, T>(src1, src2, dst, stream);
    }

    template void compare_eq<uchar >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_eq<schar >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_eq<ushort>(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_eq<short >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_eq<int   >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_eq<float >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_eq<double>(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);

    template void compare_ne<uchar >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_ne<schar >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_ne<ushort>(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_ne<short >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_ne<int   >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_ne<float >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_ne<double>(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);

    template void compare_lt<uchar >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_lt<schar >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_lt<ushort>(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_lt<short >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_lt<int   >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_lt<float >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_lt<double>(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);

    template void compare_le<uchar >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_le<schar >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_le<ushort>(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_le<short >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_le<int   >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_le<float >(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void compare_le<double>(DevMem2Db src1, DevMem2Db src2, DevMem2Db dst, hipStream_t stream);

    template <template <typename> class Op, typename T, int cn> void compare(DevMem2Db src, double val[4], DevMem2Db dst, hipStream_t stream)
    {
        typedef typename TypeVec<T, cn>::vec_type src_t;
        typedef typename TypeVec<uchar, cn>::vec_type dst_t;

        T sval[] = {static_cast<T>(val[0]), static_cast<T>(val[1]), static_cast<T>(val[2]), static_cast<T>(val[3])};
        src_t val1 = VecTraits<src_t>::make(sval);

        Compare<Op, T, cn> op;

        cv::gpu::device::transform(static_cast< DevMem2D_<src_t> >(src), static_cast< DevMem2D_<dst_t> >(dst), cv::gpu::device::bind2nd(op, val1), WithOutMask(), stream);
    }

    template <typename T> void compare_eq(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream)
    {
        typedef void (*func_t)(DevMem2Db src, double val[4], DevMem2Db dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            compare<equal_to, T, 1>,
            compare<equal_to, T, 2>,
            compare<equal_to, T, 3>,
            compare<equal_to, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void compare_ne(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream)
    {
        typedef void (*func_t)(DevMem2Db src, double val[4], DevMem2Db dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            compare<not_equal_to, T, 1>,
            compare<not_equal_to, T, 2>,
            compare<not_equal_to, T, 3>,
            compare<not_equal_to, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void compare_lt(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream)
    {
        typedef void (*func_t)(DevMem2Db src, double val[4], DevMem2Db dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            compare<less, T, 1>,
            compare<less, T, 2>,
            compare<less, T, 3>,
            compare<less, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void compare_le(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream)
    {
        typedef void (*func_t)(DevMem2Db src, double val[4], DevMem2Db dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            compare<less_equal, T, 1>,
            compare<less_equal, T, 2>,
            compare<less_equal, T, 3>,
            compare<less_equal, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void compare_gt(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream)
    {
        typedef void (*func_t)(DevMem2Db src, double val[4], DevMem2Db dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            compare<greater, T, 1>,
            compare<greater, T, 2>,
            compare<greater, T, 3>,
            compare<greater, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void compare_ge(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream)
    {
        typedef void (*func_t)(DevMem2Db src, double val[4], DevMem2Db dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            compare<greater_equal, T, 1>,
            compare<greater_equal, T, 2>,
            compare<greater_equal, T, 3>,
            compare<greater_equal, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }

    template void compare_eq<uchar >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_eq<schar >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_eq<ushort>(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_eq<short >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_eq<int   >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_eq<float >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_eq<double>(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);

    template void compare_ne<uchar >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_ne<schar >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_ne<ushort>(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_ne<short >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_ne<int   >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_ne<float >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_ne<double>(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);

    template void compare_lt<uchar >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_lt<schar >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_lt<ushort>(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_lt<short >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_lt<int   >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_lt<float >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_lt<double>(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);

    template void compare_le<uchar >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_le<schar >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_le<ushort>(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_le<short >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_le<int   >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_le<float >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_le<double>(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);

    template void compare_gt<uchar >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_gt<schar >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_gt<ushort>(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_gt<short >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_gt<int   >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_gt<float >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_gt<double>(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);

    template void compare_ge<uchar >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_ge<schar >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_ge<ushort>(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_ge<short >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_ge<int   >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_ge<float >(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);
    template void compare_ge<double>(DevMem2Db src, int cn, double val[4], DevMem2Db dst, hipStream_t stream);


    //////////////////////////////////////////////////////////////////////////
    // Unary bitwise logical matrix operations

    enum { UN_OP_NOT };

    template <typename T, int opid>
    struct UnOp;

    template <typename T>
    struct UnOp<T, UN_OP_NOT>
    {
        static __device__ __forceinline__ T call(T v) { return ~v; }
    };


    template <int opid>
    __global__ void bitwiseUnOpKernel(int rows, int width, const PtrStepb src, PtrStepb dst)
    {
        const int x = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (y < rows)
        {
            uchar* dst_ptr = dst.ptr(y) + x;
            const uchar* src_ptr = src.ptr(y) + x;
            if (x + sizeof(uint) - 1 < width)
            {
                *(uint*)dst_ptr = UnOp<uint, opid>::call(*(uint*)src_ptr);
            }
            else
            {
                const uchar* src_end = src.ptr(y) + width;
                while (src_ptr < src_end)
                {
                    *dst_ptr++ = UnOp<uchar, opid>::call(*src_ptr++);
                }
            }
        }
    }


    template <int opid>
    void bitwiseUnOp(int rows, int width, const PtrStepb src, PtrStepb dst,
                     hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(width, threads.x * sizeof(uint)),
                  divUp(rows, threads.y));

        bitwiseUnOpKernel<opid><<<grid, threads>>>(rows, width, src, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }


    template <typename T, int opid>
    __global__ void bitwiseUnOpKernel(int rows, int cols, int cn, const PtrStepb src,
                                      const PtrStepb mask, PtrStepb dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask.ptr(y)[x / cn])
        {
            T* dst_row = (T*)dst.ptr(y);
            const T* src_row = (const T*)src.ptr(y);

            dst_row[x] = UnOp<T, opid>::call(src_row[x]);
        }
    }


    template <typename T, int opid>
    void bitwiseUnOp(int rows, int cols, int cn, const PtrStepb src,
                     const PtrStepb mask, PtrStepb dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        bitwiseUnOpKernel<T, opid><<<grid, threads>>>(rows, cols, cn, src, mask, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }


    void bitwiseNotCaller(int rows, int cols, size_t elem_size1, int cn,
                          const PtrStepb src, PtrStepb dst, hipStream_t stream)
    {
        bitwiseUnOp<UN_OP_NOT>(rows, static_cast<int>(cols * elem_size1 * cn), src, dst, stream);
    }


    template <typename T>
    void bitwiseMaskNotCaller(int rows, int cols, int cn, const PtrStepb src,
                              const PtrStepb mask, PtrStepb dst, hipStream_t stream)
    {
        bitwiseUnOp<T, UN_OP_NOT>(rows, cols * cn, cn, src, mask, dst, stream);
    }

    template void bitwiseMaskNotCaller<uchar>(int, int, int, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);
    template void bitwiseMaskNotCaller<ushort>(int, int, int, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);
    template void bitwiseMaskNotCaller<uint>(int, int, int, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);


    //////////////////////////////////////////////////////////////////////////
    // Binary bitwise logical matrix operations

    enum { BIN_OP_OR, BIN_OP_AND, BIN_OP_XOR };

    template <typename T, int opid>
    struct BinOp;

    template <typename T>
    struct BinOp<T, BIN_OP_OR>
    {
        static __device__ __forceinline__ T call(T a, T b) { return a | b; }
    };


    template <typename T>
    struct BinOp<T, BIN_OP_AND>
    {
        static __device__ __forceinline__ T call(T a, T b) { return a & b; }
    };

    template <typename T>
    struct BinOp<T, BIN_OP_XOR>
    {
        static __device__ __forceinline__ T call(T a, T b) { return a ^ b; }
    };


    template <int opid>
    __global__ void bitwiseBinOpKernel(int rows, int width, const PtrStepb src1,
                                       const PtrStepb src2, PtrStepb dst)
    {
        const int x = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (y < rows)
        {
            uchar* dst_ptr = dst.ptr(y) + x;
            const uchar* src1_ptr = src1.ptr(y) + x;
            const uchar* src2_ptr = src2.ptr(y) + x;

            if (x + sizeof(uint) - 1 < width)
            {
                *(uint*)dst_ptr = BinOp<uint, opid>::call(*(uint*)src1_ptr, *(uint*)src2_ptr);
            }
            else
            {
                const uchar* src1_end = src1.ptr(y) + width;
                while (src1_ptr < src1_end)
                {
                    *dst_ptr++ = BinOp<uchar, opid>::call(*src1_ptr++, *src2_ptr++);
                }
            }
        }
    }


    template <int opid>
    void bitwiseBinOp(int rows, int width, const PtrStepb src1, const PtrStepb src2,
                      PtrStepb dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(width, threads.x * sizeof(uint)), divUp(rows, threads.y));

        bitwiseBinOpKernel<opid><<<grid, threads>>>(rows, width, src1, src2, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }


    template <typename T, int opid>
    __global__ void bitwiseBinOpKernel(
            int rows, int cols, int cn, const PtrStepb src1, const PtrStepb src2,
            const PtrStepb mask, PtrStepb dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask.ptr(y)[x / cn])
        {
            T* dst_row = (T*)dst.ptr(y);
            const T* src1_row = (const T*)src1.ptr(y);
            const T* src2_row = (const T*)src2.ptr(y);

            dst_row[x] = BinOp<T, opid>::call(src1_row[x], src2_row[x]);
        }
    }


    template <typename T, int opid>
    void bitwiseBinOp(int rows, int cols, int cn, const PtrStepb src1, const PtrStepb src2,
                        const PtrStepb mask, PtrStepb dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        bitwiseBinOpKernel<T, opid><<<grid, threads>>>(rows, cols, cn, src1, src2, mask, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }


    void bitwiseOrCaller(int rows, int cols, size_t elem_size1, int cn, const PtrStepb src1,
                         const PtrStepb src2, PtrStepb dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_OR>(rows, static_cast<int>(cols * elem_size1 * cn), src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskOrCaller(int rows, int cols, int cn, const PtrStepb src1, const PtrStepb src2,
                             const PtrStepb mask, PtrStepb dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_OR>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskOrCaller<uchar>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);
    template void bitwiseMaskOrCaller<ushort>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);
    template void bitwiseMaskOrCaller<uint>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);


    void bitwiseAndCaller(int rows, int cols, size_t elem_size1, int cn, const PtrStepb src1,
                          const PtrStepb src2, PtrStepb dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_AND>(rows, static_cast<int>(cols * elem_size1 * cn), src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskAndCaller(int rows, int cols, int cn, const PtrStepb src1, const PtrStepb src2,
                              const PtrStepb mask, PtrStepb dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_AND>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskAndCaller<uchar>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);
    template void bitwiseMaskAndCaller<ushort>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);
    template void bitwiseMaskAndCaller<uint>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);


    void bitwiseXorCaller(int rows, int cols, size_t elem_size1, int cn, const PtrStepb src1,
                          const PtrStepb src2, PtrStepb dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_XOR>(rows, static_cast<int>(cols * elem_size1 * cn), src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskXorCaller(int rows, int cols, int cn, const PtrStepb src1, const PtrStepb src2,
                              const PtrStepb mask, PtrStepb dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_XOR>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskXorCaller<uchar>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);
    template void bitwiseMaskXorCaller<ushort>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);
    template void bitwiseMaskXorCaller<uint>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);

    //////////////////////////////////////////////////////////////////////////
    // min/max

    namespace detail
    {
        template <size_t size, typename F> struct MinMaxTraits : DefaultTransformFunctorTraits<F>
        {
        };
        template <typename F> struct MinMaxTraits<2, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_shift = 4 };
        };
        template <typename F> struct MinMaxTraits<4, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_block_dim_y = 4 };
            enum { smart_shift = 4 };
        };
    }

    template <typename T> struct TransformFunctorTraits< minimum<T> > : detail::MinMaxTraits< sizeof(T), minimum<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< maximum<T> > : detail::MinMaxTraits< sizeof(T), maximum<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< binder2nd< minimum<T> > > : detail::MinMaxTraits< sizeof(T), binder2nd< minimum<T> > >
    {
    };
    template <typename T> struct TransformFunctorTraits< binder2nd< maximum<T> > > : detail::MinMaxTraits< sizeof(T), binder2nd< maximum<T> > >
    {
    };

    template <typename T>
    void min_gpu(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream)
    {
        cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<T>)src2, (DevMem2D_<T>)dst, minimum<T>(), WithOutMask(), stream);
    }

    template void min_gpu<uchar >(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void min_gpu<schar >(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void min_gpu<ushort>(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void min_gpu<short >(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void min_gpu<int   >(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void min_gpu<float >(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void min_gpu<double>(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);

    template <typename T>
    void max_gpu(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream)
    {
        cv::gpu::device::transform((DevMem2D_<T>)src1, (DevMem2D_<T>)src2, (DevMem2D_<T>)dst, maximum<T>(), WithOutMask(), stream);
    }

    template void max_gpu<uchar >(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void max_gpu<schar >(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void max_gpu<ushort>(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void max_gpu<short >(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void max_gpu<int   >(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void max_gpu<float >(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);
    template void max_gpu<double>(const DevMem2Db src1, const DevMem2Db src2, DevMem2Db dst, hipStream_t stream);

    template <typename T>
    void min_gpu(const DevMem2Db src, T val, DevMem2Db dst, hipStream_t stream)
    {
        cv::gpu::device::transform((DevMem2D_<T>)src, (DevMem2D_<T>)dst, device::bind2nd(minimum<T>(), val), WithOutMask(), stream);
    }

    template void min_gpu<uchar >(const DevMem2Db src, uchar  val, DevMem2Db dst, hipStream_t stream);
    template void min_gpu<schar >(const DevMem2Db src, schar  val, DevMem2Db dst, hipStream_t stream);
    template void min_gpu<ushort>(const DevMem2Db src, ushort val, DevMem2Db dst, hipStream_t stream);
    template void min_gpu<short >(const DevMem2Db src, short  val, DevMem2Db dst, hipStream_t stream);
    template void min_gpu<int   >(const DevMem2Db src, int    val, DevMem2Db dst, hipStream_t stream);
    template void min_gpu<float >(const DevMem2Db src, float  val, DevMem2Db dst, hipStream_t stream);
    template void min_gpu<double>(const DevMem2Db src, double val, DevMem2Db dst, hipStream_t stream);

    template <typename T>
    void max_gpu(const DevMem2Db src, T val, DevMem2Db dst, hipStream_t stream)
    {
        cv::gpu::device::transform((DevMem2D_<T>)src, (DevMem2D_<T>)dst, device::bind2nd(maximum<T>(), val), WithOutMask(), stream);
    }

    template void max_gpu<uchar >(const DevMem2Db src, uchar  val, DevMem2Db dst, hipStream_t stream);
    template void max_gpu<schar >(const DevMem2Db src, schar  val, DevMem2Db dst, hipStream_t stream);
    template void max_gpu<ushort>(const DevMem2Db src, ushort val, DevMem2Db dst, hipStream_t stream);
    template void max_gpu<short >(const DevMem2Db src, short  val, DevMem2Db dst, hipStream_t stream);
    template void max_gpu<int   >(const DevMem2Db src, int    val, DevMem2Db dst, hipStream_t stream);
    template void max_gpu<float >(const DevMem2Db src, float  val, DevMem2Db dst, hipStream_t stream);
    template void max_gpu<double>(const DevMem2Db src, double val, DevMem2Db dst, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // threshold

    namespace detail
    {
        template <size_t size, typename F> struct ThresholdTraits : DefaultTransformFunctorTraits<F>
        {
        };
        template <typename F> struct ThresholdTraits<2, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_shift = 4 };
        };
        template <typename F> struct ThresholdTraits<4, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_block_dim_y = 4 };
            enum { smart_shift = 4 };
        };
    }

    template <typename T> struct TransformFunctorTraits< thresh_binary_func<T> > : detail::ThresholdTraits< sizeof(T), thresh_binary_func<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< thresh_binary_inv_func<T> > : detail::ThresholdTraits< sizeof(T), thresh_binary_inv_func<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< thresh_trunc_func<T> > : detail::ThresholdTraits< sizeof(T), thresh_trunc_func<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< thresh_to_zero_func<T> > : detail::ThresholdTraits< sizeof(T), thresh_to_zero_func<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< thresh_to_zero_inv_func<T> > : detail::ThresholdTraits< sizeof(T), thresh_to_zero_inv_func<T> >
    {
    };

    template <template <typename> class Op, typename T>
    void threshold_caller(const DevMem2D_<T>& src, const DevMem2D_<T>& dst, T thresh, T maxVal, hipStream_t stream)
    {
        Op<T> op(thresh, maxVal);
        cv::gpu::device::transform(src, dst, op, WithOutMask(), stream);
    }

    template <typename T>
    void threshold_gpu(const DevMem2Db& src, const DevMem2Db& dst, T thresh, T maxVal, int type,
        hipStream_t stream)
    {
        typedef void (*caller_t)(const DevMem2D_<T>& src, const DevMem2D_<T>& dst, T thresh, T maxVal, hipStream_t stream);

        static const caller_t callers[] =
        {
            threshold_caller<thresh_binary_func, T>,
            threshold_caller<thresh_binary_inv_func, T>,
            threshold_caller<thresh_trunc_func, T>,
            threshold_caller<thresh_to_zero_func, T>,
            threshold_caller<thresh_to_zero_inv_func, T>
        };

        callers[type]((DevMem2D_<T>)src, (DevMem2D_<T>)dst, thresh, maxVal, stream);
    }

    template void threshold_gpu<uchar>(const DevMem2Db& src, const DevMem2Db& dst, uchar thresh, uchar maxVal, int type, hipStream_t stream);
    template void threshold_gpu<schar>(const DevMem2Db& src, const DevMem2Db& dst, schar thresh, schar maxVal, int type, hipStream_t stream);
    template void threshold_gpu<ushort>(const DevMem2Db& src, const DevMem2Db& dst, ushort thresh, ushort maxVal, int type, hipStream_t stream);
    template void threshold_gpu<short>(const DevMem2Db& src, const DevMem2Db& dst, short thresh, short maxVal, int type, hipStream_t stream);
    template void threshold_gpu<int>(const DevMem2Db& src, const DevMem2Db& dst, int thresh, int maxVal, int type, hipStream_t stream);
    template void threshold_gpu<float>(const DevMem2Db& src, const DevMem2Db& dst, float thresh, float maxVal, int type, hipStream_t stream);
    template void threshold_gpu<double>(const DevMem2Db& src, const DevMem2Db& dst, double thresh, double maxVal, int type, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // pow

    template<typename T, bool Signed = device::numeric_limits<T>::is_signed> struct PowOp : unary_function<T, T>
    {
        const float power;

        PowOp(double power_) : power(static_cast<float>(power_)) {}

        __device__ __forceinline__ T operator()(T e) const
        {
            return saturate_cast<T>(__powf((float)e, power));
        }
    };
    template<typename T> struct PowOp<T, true> : unary_function<T, T>
    {
        const float power;

        PowOp(double power_) : power(static_cast<float>(power_)) {}

        __device__ __forceinline__ T operator()(T e) const
        {
            T res = saturate_cast<T>(__powf((float)e, power));

            if ((e < 0) && (1 & static_cast<int>(power)))
                res *= -1;

            return res;
        }
    };
    template<> struct PowOp<float> : unary_function<float, float>
    {
        const float power;

        PowOp(double power_) : power(static_cast<float>(power_)) {}

        __device__ __forceinline__ float operator()(float e) const
        {
            return __powf(::fabs(e), power);
        }
    };
    template<> struct PowOp<double> : unary_function<double, double>
    {
        const double power;

        PowOp(double power_) : power(power_) {}

        __device__ __forceinline__ double operator()(double e) const
        {
            return ::pow(::fabs(e), power);
        }
    };

    namespace detail
    {
        template <size_t size, typename T> struct PowOpTraits : DefaultTransformFunctorTraits< PowOp<T> >
        {
        };
        template <typename T> struct PowOpTraits<1, T> : DefaultTransformFunctorTraits< PowOp<T> >
        {
            enum { smart_block_dim_y = 8 };
            enum { smart_shift = 8 };
        };
        template <typename T> struct PowOpTraits<2, T> : DefaultTransformFunctorTraits< PowOp<T> >
        {
            enum { smart_shift = 4 };
        };
        template <typename T> struct PowOpTraits<4, T> : DefaultTransformFunctorTraits< PowOp<T> >
        {
            enum { smart_block_dim_y = 4 };
            enum { smart_shift = 4 };
        };
    }

    template <typename T> struct TransformFunctorTraits< PowOp<T> > : detail::PowOpTraits<sizeof(T), T>
    {
    };

    template<typename T>
    void pow_caller(DevMem2Db src, double power, DevMem2Db dst, hipStream_t stream)
    {
        cv::gpu::device::transform((DevMem2D_<T>)src, (DevMem2D_<T>)dst, PowOp<T>(power), WithOutMask(), stream);
    }

    template void pow_caller<uchar>(DevMem2Db src, double power, DevMem2Db dst, hipStream_t stream);
    template void pow_caller<schar>(DevMem2Db src, double power, DevMem2Db dst, hipStream_t stream);
    template void pow_caller<short>(DevMem2Db src, double power, DevMem2Db dst, hipStream_t stream);
    template void pow_caller<ushort>(DevMem2Db src, double power, DevMem2Db dst, hipStream_t stream);
    template void pow_caller<int>(DevMem2Db src, double power, DevMem2Db dst, hipStream_t stream);
    template void pow_caller<float>(DevMem2Db src, double power, DevMem2Db dst, hipStream_t stream);
    template void pow_caller<double>(DevMem2Db src, double power, DevMem2Db dst, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // addWeighted

    namespace detail
    {
        template <typename T> struct UseDouble
        {
            enum {value = 0};
        };
        template <> struct UseDouble<int>
        {
            enum {value = 1};
        };
        template <> struct UseDouble<float>
        {
            enum {value = 1};
        };
        template <> struct UseDouble<double>
        {
            enum {value = 1};
        };
    }
    template <typename T1, typename T2, typename D> struct UseDouble
    {
        enum {value = (detail::UseDouble<T1>::value || detail::UseDouble<T2>::value || detail::UseDouble<D>::value)};
    };

    namespace detail
    {
        template <typename T1, typename T2, typename D, bool useDouble> struct AddWeighted;
        template <typename T1, typename T2, typename D> struct AddWeighted<T1, T2, D, false> : binary_function<T1, T2, D>
        {
            AddWeighted(double alpha_, double beta_, double gamma_) : alpha(static_cast<float>(alpha_)), beta(static_cast<float>(beta_)), gamma(static_cast<float>(gamma_)) {}

            __device__ __forceinline__ D operator ()(T1 a, T2 b) const
            {
                return saturate_cast<D>(a * alpha + b * beta + gamma);
            }

            const float alpha;
            const float beta;
            const float gamma;
        };
        template <typename T1, typename T2, typename D> struct AddWeighted<T1, T2, D, true> : binary_function<T1, T2, D>
        {
            AddWeighted(double alpha_, double beta_, double gamma_) : alpha(alpha_), beta(beta_), gamma(gamma_) {}

            __device__ __forceinline__ D operator ()(T1 a, T2 b) const
            {
                return saturate_cast<D>(a * alpha + b * beta + gamma);
            }

            const double alpha;
            const double beta;
            const double gamma;
        };
    }
    template <typename T1, typename T2, typename D> struct AddWeighted : detail::AddWeighted<T1, T2, D, UseDouble<T1, T2, D>::value>
    {
        AddWeighted(double alpha_, double beta_, double gamma_) : detail::AddWeighted<T1, T2, D, UseDouble<T1, T2, D>::value>(alpha_, beta_, gamma_) {}
    };

    template <> struct TransformFunctorTraits< AddWeighted<ushort, ushort, ushort> > : DefaultTransformFunctorTraits< AddWeighted<ushort, ushort, ushort> >
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<ushort, ushort, short> > : DefaultTransformFunctorTraits< AddWeighted<ushort, ushort, short> >
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<ushort, short, ushort> > : DefaultTransformFunctorTraits< AddWeighted<ushort, short, ushort> >
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<ushort, short, short> > : DefaultTransformFunctorTraits< AddWeighted<ushort, short, short> >
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<short, short, ushort> > : DefaultTransformFunctorTraits< AddWeighted<short, short, ushort> >
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<short, short, short> > : DefaultTransformFunctorTraits< AddWeighted<short, short, short> >
    {
        enum { smart_shift = 4 };
    };

    template <> struct TransformFunctorTraits< AddWeighted<int, int, int> > : DefaultTransformFunctorTraits< AddWeighted<int, int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<int, int, float> > : DefaultTransformFunctorTraits< AddWeighted<int, int, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<int, float, int> > : DefaultTransformFunctorTraits< AddWeighted<int, float, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<int, float, float> > : DefaultTransformFunctorTraits< AddWeighted<int, float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<float, float, int> > : DefaultTransformFunctorTraits< AddWeighted<float, float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<float, float, float> > : DefaultTransformFunctorTraits< AddWeighted<float, float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T1, typename T2, typename D>
    void addWeighted_gpu(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream)
    {
        if (UseDouble<T1, T2, D>::value)
        {
            cudaSafeCall( cudaSetDoubleForDevice(&alpha) );
            cudaSafeCall( cudaSetDoubleForDevice(&beta) );
            cudaSafeCall( cudaSetDoubleForDevice(&gamma) );
        }

        AddWeighted<T1, T2, D> op(alpha, beta, gamma);

        cv::gpu::device::transform(static_cast< DevMem2D_<T1> >(src1), static_cast< DevMem2D_<T2> >(src2), static_cast< DevMem2D_<D> >(dst), op, WithOutMask(), stream);
    }

    template void addWeighted_gpu<uchar, uchar, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, schar, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, ushort, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, short, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, int, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, float, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, double, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);



    template void addWeighted_gpu<schar, schar, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<schar, ushort, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<schar, short, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<schar, int, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<schar, float, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<schar, double, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);



    template void addWeighted_gpu<ushort, ushort, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<ushort, short, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<ushort, int, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<ushort, float, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<ushort, double, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);



    template void addWeighted_gpu<short, short, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<short, int, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<short, float, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<short, double, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);



    template void addWeighted_gpu<int, int, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<int, float, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<int, double, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);



    template void addWeighted_gpu<float, float, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);

    template void addWeighted_gpu<float, double, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);



    template void addWeighted_gpu<double, double, uchar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, schar>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, ushort>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, short>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, int>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, float>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, double>(const DevMem2Db& src1, double alpha, const DevMem2Db& src2, double beta, double gamma, const DevMem2Db& dst, hipStream_t stream);
}}} // namespace cv { namespace gpu { namespace device
