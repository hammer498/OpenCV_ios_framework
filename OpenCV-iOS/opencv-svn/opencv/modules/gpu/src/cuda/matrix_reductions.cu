#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"
#include "opencv2/gpu/device/vec_math.hpp"

namespace cv { namespace gpu { namespace device 
{
    namespace matrix_reductions 
    {
        // Performs reduction in shared memory
        template <int size, typename T>
        __device__ void sumInSmem(volatile T* data, const uint tid)
        {
            T sum = data[tid];

            if (size >= 512) { if (tid < 256) { data[tid] = sum = sum + data[tid + 256]; } __syncthreads(); }
            if (size >= 256) { if (tid < 128) { data[tid] = sum = sum + data[tid + 128]; } __syncthreads(); }
            if (size >= 128) { if (tid < 64) { data[tid] = sum = sum + data[tid + 64]; } __syncthreads(); }

            if (tid < 32)
            {
                if (size >= 64) data[tid] = sum = sum + data[tid + 32];
                if (size >= 32) data[tid] = sum = sum + data[tid + 16];
                if (size >= 16) data[tid] = sum = sum + data[tid + 8];
                if (size >= 8) data[tid] = sum = sum + data[tid + 4];
                if (size >= 4) data[tid] = sum = sum + data[tid + 2];
                if (size >= 2) data[tid] = sum = sum + data[tid + 1];
            }
        }

        struct Mask8U
        {
            explicit Mask8U(PtrStepb mask): mask(mask) {}

            __device__ __forceinline__ bool operator()(int y, int x) const 
            { 
                return mask.ptr(y)[x]; 
            }

            PtrStepb mask;
        };

        struct MaskTrue 
        { 
            __device__ __forceinline__ bool operator()(int y, int x) const 
            { 
                return true; 
            } 
        };

        //////////////////////////////////////////////////////////////////////////////
        // Min max

        // To avoid shared bank conflicts we convert each value into value of 
        // appropriate type (32 bits minimum)
        template <typename T> struct MinMaxTypeTraits {};
        template <> struct MinMaxTypeTraits<uchar> { typedef int best_type; };
        template <> struct MinMaxTypeTraits<char> { typedef int best_type; };
        template <> struct MinMaxTypeTraits<ushort> { typedef int best_type; };
        template <> struct MinMaxTypeTraits<short> { typedef int best_type; };
        template <> struct MinMaxTypeTraits<int> { typedef int best_type; };
        template <> struct MinMaxTypeTraits<float> { typedef float best_type; };
        template <> struct MinMaxTypeTraits<double> { typedef double best_type; };

        namespace minmax 
        {
            __constant__ int ctwidth;
            __constant__ int ctheight;

            // Global counter of blocks finished its work
            __device__ uint blocks_finished = 0;


            // Estimates good thread configuration
            //  - threads variable satisfies to threads.x * threads.y == 256
            void estimateThreadCfg(int cols, int rows, dim3& threads, dim3& grid)
            {
                threads = dim3(32, 8);
                grid = dim3(divUp(cols, threads.x * 8), divUp(rows, threads.y * 32));
                grid.x = std::min(grid.x, threads.x);
                grid.y = std::min(grid.y, threads.y);
            }


            // Returns required buffer sizes
            void getBufSizeRequired(int cols, int rows, int elem_size, int& bufcols, int& bufrows)
            {
                dim3 threads, grid;
                estimateThreadCfg(cols, rows, threads, grid);
                bufcols = grid.x * grid.y * elem_size; 
                bufrows = 2;
            }


            // Estimates device constants which are used in the kernels using specified thread configuration
            void setKernelConsts(int cols, int rows, const dim3& threads, const dim3& grid)
            {        
                int twidth = divUp(divUp(cols, grid.x), threads.x);
                int theight = divUp(divUp(rows, grid.y), threads.y);
                cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctwidth), &twidth, sizeof(ctwidth))); 
                cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctheight), &theight, sizeof(ctheight))); 
            }  


            // Does min and max in shared memory
            template <typename T>
            __device__ __forceinline__ void merge(uint tid, uint offset, volatile T* minval, volatile T* maxval)
            {
                minval[tid] = ::min(minval[tid], minval[tid + offset]);
                maxval[tid] = ::max(maxval[tid], maxval[tid + offset]);
            }


            template <int size, typename T>
            __device__ void findMinMaxInSmem(volatile T* minval, volatile T* maxval, const uint tid)
            {
                if (size >= 512) { if (tid < 256) { merge(tid, 256, minval, maxval); } __syncthreads(); }
                if (size >= 256) { if (tid < 128) { merge(tid, 128, minval, maxval); }  __syncthreads(); }
                if (size >= 128) { if (tid < 64) { merge(tid, 64, minval, maxval); } __syncthreads(); }

                if (tid < 32)
                {
                    if (size >= 64) merge(tid, 32, minval, maxval);
                    if (size >= 32) merge(tid, 16, minval, maxval);
                    if (size >= 16) merge(tid, 8, minval, maxval);
                    if (size >= 8) merge(tid, 4, minval, maxval);
                    if (size >= 4) merge(tid, 2, minval, maxval);
                    if (size >= 2) merge(tid, 1, minval, maxval);
                }
            }


            template <int nthreads, typename T, typename Mask>
            __global__ void minMaxKernel(const DevMem2Db src, Mask mask, T* minval, T* maxval)
            {
                typedef typename MinMaxTypeTraits<T>::best_type best_type;
                __shared__ best_type sminval[nthreads];
                __shared__ best_type smaxval[nthreads];

                uint x0 = blockIdx.x * blockDim.x * ctwidth + threadIdx.x;
                uint y0 = blockIdx.y * blockDim.y * ctheight + threadIdx.y;
                uint tid = threadIdx.y * blockDim.x + threadIdx.x;

                T mymin = numeric_limits<T>::max();
                T mymax = numeric_limits<T>::is_signed ? -numeric_limits<T>::max() : numeric_limits<T>::min();
                uint y_end = ::min(y0 + (ctheight - 1) * blockDim.y + 1, src.rows);
                uint x_end = ::min(x0 + (ctwidth - 1) * blockDim.x + 1, src.cols);
                for (uint y = y0; y < y_end; y += blockDim.y)
                {
                    const T* src_row = (const T*)src.ptr(y);
                    for (uint x = x0; x < x_end; x += blockDim.x)
                    {
                        T val = src_row[x];
                        if (mask(y, x)) 
                        { 
                            mymin = ::min(mymin, val); 
                            mymax = ::max(mymax, val); 
                        }
                    }
                }

                sminval[tid] = mymin;
                smaxval[tid] = mymax;
                __syncthreads();

                findMinMaxInSmem<nthreads, best_type>(sminval, smaxval, tid);

                if (tid == 0) 
                {
                    minval[blockIdx.y * gridDim.x + blockIdx.x] = (T)sminval[0];
                    maxval[blockIdx.y * gridDim.x + blockIdx.x] = (T)smaxval[0];
                }

            #if __CUDA_ARCH__ >= 110
		        __shared__ bool is_last;

		        if (tid == 0)
		        {
			        minval[blockIdx.y * gridDim.x + blockIdx.x] = (T)sminval[0];
                    maxval[blockIdx.y * gridDim.x + blockIdx.x] = (T)smaxval[0];
			        __threadfence();

			        uint ticket = atomicInc(&blocks_finished, gridDim.x * gridDim.y);
			        is_last = ticket == gridDim.x * gridDim.y - 1;
		        }

		        __syncthreads();

		        if (is_last)
		        {
                    uint idx = ::min(tid, gridDim.x * gridDim.y - 1);

                    sminval[tid] = minval[idx];
                    smaxval[tid] = maxval[idx];
                    __syncthreads();

			        findMinMaxInSmem<nthreads, best_type>(sminval, smaxval, tid);

                    if (tid == 0) 
                    {
                        minval[0] = (T)sminval[0];
                        maxval[0] = (T)smaxval[0];
                        blocks_finished = 0;
                    }
		        }
            #else
                if (tid == 0) 
                {
                    minval[blockIdx.y * gridDim.x + blockIdx.x] = (T)sminval[0];
                    maxval[blockIdx.y * gridDim.x + blockIdx.x] = (T)smaxval[0];
                }
            #endif
            }

   
            template <typename T>
            void minMaxMaskCaller(const DevMem2Db src, const PtrStepb mask, double* minval, double* maxval, PtrStepb buf)
            {
                dim3 threads, grid;
                estimateThreadCfg(src.cols, src.rows, threads, grid);
                setKernelConsts(src.cols, src.rows, threads, grid);

                T* minval_buf = (T*)buf.ptr(0);
                T* maxval_buf = (T*)buf.ptr(1);

                minMaxKernel<256, T, Mask8U><<<grid, threads>>>(src, Mask8U(mask), minval_buf, maxval_buf);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                T minval_, maxval_;
                cudaSafeCall( hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost) );
                cudaSafeCall( hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost) );
                *minval = minval_;
                *maxval = maxval_;
            }  

            template void minMaxMaskCaller<uchar>(const DevMem2Db, const PtrStepb, double*, double*, PtrStepb);
            template void minMaxMaskCaller<char>(const DevMem2Db, const PtrStepb, double*, double*, PtrStepb);
            template void minMaxMaskCaller<ushort>(const DevMem2Db, const PtrStepb, double*, double*, PtrStepb);
            template void minMaxMaskCaller<short>(const DevMem2Db, const PtrStepb, double*, double*, PtrStepb);
            template void minMaxMaskCaller<int>(const DevMem2Db, const PtrStepb, double*, double*, PtrStepb);
            template void minMaxMaskCaller<float>(const DevMem2Db, const PtrStepb, double*, double*, PtrStepb);
            template void minMaxMaskCaller<double>(const DevMem2Db, const PtrStepb, double*, double*, PtrStepb);


            template <typename T>
            void minMaxCaller(const DevMem2Db src, double* minval, double* maxval, PtrStepb buf)
            {
                dim3 threads, grid;
                estimateThreadCfg(src.cols, src.rows, threads, grid);
                setKernelConsts(src.cols, src.rows, threads, grid);

                T* minval_buf = (T*)buf.ptr(0);
                T* maxval_buf = (T*)buf.ptr(1);

                minMaxKernel<256, T, MaskTrue><<<grid, threads>>>(src, MaskTrue(), minval_buf, maxval_buf);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                T minval_, maxval_;
                cudaSafeCall( hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost) );
                cudaSafeCall( hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost) );
                *minval = minval_;
                *maxval = maxval_;
            }  

            template void minMaxCaller<uchar>(const DevMem2Db, double*, double*, PtrStepb);
            template void minMaxCaller<char>(const DevMem2Db, double*, double*, PtrStepb);
            template void minMaxCaller<ushort>(const DevMem2Db, double*, double*, PtrStepb);
            template void minMaxCaller<short>(const DevMem2Db, double*, double*, PtrStepb);
            template void minMaxCaller<int>(const DevMem2Db, double*, double*, PtrStepb);
            template void minMaxCaller<float>(const DevMem2Db, double*,double*, PtrStepb);
            template void minMaxCaller<double>(const DevMem2Db, double*, double*, PtrStepb);


            template <int nthreads, typename T>
            __global__ void minMaxPass2Kernel(T* minval, T* maxval, int size)
            {
                typedef typename MinMaxTypeTraits<T>::best_type best_type;
                __shared__ best_type sminval[nthreads];
                __shared__ best_type smaxval[nthreads];
                
                uint tid = threadIdx.y * blockDim.x + threadIdx.x;
                uint idx = ::min(tid, size - 1);

                sminval[tid] = minval[idx];
                smaxval[tid] = maxval[idx];
                __syncthreads();

                findMinMaxInSmem<nthreads, best_type>(sminval, smaxval, tid);

                if (tid == 0) 
                {
                    minval[0] = (T)sminval[0];
                    maxval[0] = (T)smaxval[0];
                }
            }


            template <typename T>
            void minMaxMaskMultipassCaller(const DevMem2Db src, const PtrStepb mask, double* minval, double* maxval, PtrStepb buf)
            {
                dim3 threads, grid;
                estimateThreadCfg(src.cols, src.rows, threads, grid);
                setKernelConsts(src.cols, src.rows, threads, grid);

                T* minval_buf = (T*)buf.ptr(0);
                T* maxval_buf = (T*)buf.ptr(1);

                minMaxKernel<256, T, Mask8U><<<grid, threads>>>(src, Mask8U(mask), minval_buf, maxval_buf);
                cudaSafeCall( hipGetLastError() );
                minMaxPass2Kernel<256, T><<<1, 256>>>(minval_buf, maxval_buf, grid.x * grid.y);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall(hipDeviceSynchronize());

                T minval_, maxval_;
                cudaSafeCall( hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost) );
                cudaSafeCall( hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost) );
                *minval = minval_;
                *maxval = maxval_;
            }

            template void minMaxMaskMultipassCaller<uchar>(const DevMem2Db, const PtrStepb, double*, double*, PtrStepb);
            template void minMaxMaskMultipassCaller<char>(const DevMem2Db, const PtrStepb, double*, double*, PtrStepb);
            template void minMaxMaskMultipassCaller<ushort>(const DevMem2Db, const PtrStepb, double*, double*, PtrStepb);
            template void minMaxMaskMultipassCaller<short>(const DevMem2Db, const PtrStepb, double*, double*, PtrStepb);
            template void minMaxMaskMultipassCaller<int>(const DevMem2Db, const PtrStepb, double*, double*, PtrStepb);
            template void minMaxMaskMultipassCaller<float>(const DevMem2Db, const PtrStepb, double*, double*, PtrStepb);


            template <typename T>
            void minMaxMultipassCaller(const DevMem2Db src, double* minval, double* maxval, PtrStepb buf)
            {
                dim3 threads, grid;
                estimateThreadCfg(src.cols, src.rows, threads, grid);
                setKernelConsts(src.cols, src.rows, threads, grid);

                T* minval_buf = (T*)buf.ptr(0);
                T* maxval_buf = (T*)buf.ptr(1);

                minMaxKernel<256, T, MaskTrue><<<grid, threads>>>(src, MaskTrue(), minval_buf, maxval_buf);
                cudaSafeCall( hipGetLastError() );
                minMaxPass2Kernel<256, T><<<1, 256>>>(minval_buf, maxval_buf, grid.x * grid.y);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                T minval_, maxval_;
                cudaSafeCall( hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost) );
                cudaSafeCall( hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost) );
                *minval = minval_;
                *maxval = maxval_;
            }

            template void minMaxMultipassCaller<uchar>(const DevMem2Db, double*, double*, PtrStepb);
            template void minMaxMultipassCaller<char>(const DevMem2Db, double*, double*, PtrStepb);
            template void minMaxMultipassCaller<ushort>(const DevMem2Db, double*, double*, PtrStepb);
            template void minMaxMultipassCaller<short>(const DevMem2Db, double*, double*, PtrStepb);
            template void minMaxMultipassCaller<int>(const DevMem2Db, double*, double*, PtrStepb);
            template void minMaxMultipassCaller<float>(const DevMem2Db, double*, double*, PtrStepb);
        } // namespace minmax

        ///////////////////////////////////////////////////////////////////////////////
        // minMaxLoc

        namespace minmaxloc 
        {
            __constant__ int ctwidth;
            __constant__ int ctheight;

            // Global counter of blocks finished its work
            __device__ uint blocks_finished = 0;


            // Estimates good thread configuration
            //  - threads variable satisfies to threads.x * threads.y == 256
            void estimateThreadCfg(int cols, int rows, dim3& threads, dim3& grid)
            {
                threads = dim3(32, 8);
                grid = dim3(divUp(cols, threads.x * 8), divUp(rows, threads.y * 32));
                grid.x = std::min(grid.x, threads.x);
                grid.y = std::min(grid.y, threads.y);
            }


            // Returns required buffer sizes
            void getBufSizeRequired(int cols, int rows, int elem_size, int& b1cols, 
                                    int& b1rows, int& b2cols, int& b2rows)
            {
                dim3 threads, grid;
                estimateThreadCfg(cols, rows, threads, grid);
                b1cols = grid.x * grid.y * elem_size; // For values
                b1rows = 2;
                b2cols = grid.x * grid.y * sizeof(int); // For locations
                b2rows = 2;
            }


            // Estimates device constants which are used in the kernels using specified thread configuration
            void setKernelConsts(int cols, int rows, const dim3& threads, const dim3& grid)
            {        
                int twidth = divUp(divUp(cols, grid.x), threads.x);
                int theight = divUp(divUp(rows, grid.y), threads.y);
                cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctwidth), &twidth, sizeof(ctwidth))); 
                cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctheight), &theight, sizeof(ctheight))); 
            }  


            template <typename T>
            __device__ void merge(uint tid, uint offset, volatile T* minval, volatile T* maxval, 
                                  volatile uint* minloc, volatile uint* maxloc)
            {
                T val = minval[tid + offset];
                if (val < minval[tid])
                {
                    minval[tid] = val;
                    minloc[tid] = minloc[tid + offset];
                }
                val = maxval[tid + offset];
                if (val > maxval[tid])
                {
                    maxval[tid] = val;
                    maxloc[tid] = maxloc[tid + offset];
                }
            }


            template <int size, typename T>
            __device__ void findMinMaxLocInSmem(volatile T* minval, volatile T* maxval, volatile uint* minloc, 
                                                volatile uint* maxloc, const uint tid)
            {
                if (size >= 512) { if (tid < 256) { merge(tid, 256, minval, maxval, minloc, maxloc); } __syncthreads(); }
                if (size >= 256) { if (tid < 128) { merge(tid, 128, minval, maxval, minloc, maxloc); }  __syncthreads(); }
                if (size >= 128) { if (tid < 64) { merge(tid, 64, minval, maxval, minloc, maxloc); } __syncthreads(); }

                if (tid < 32)
                {
                    if (size >= 64) merge(tid, 32, minval, maxval, minloc, maxloc);
                    if (size >= 32) merge(tid, 16, minval, maxval, minloc, maxloc);
                    if (size >= 16) merge(tid, 8, minval, maxval, minloc, maxloc);
                    if (size >= 8) merge(tid, 4, minval, maxval, minloc, maxloc);
                    if (size >= 4) merge(tid, 2, minval, maxval, minloc, maxloc);
                    if (size >= 2) merge(tid, 1, minval, maxval, minloc, maxloc);
                }
            }


            template <int nthreads, typename T, typename Mask>
            __global__ void minMaxLocKernel(const DevMem2Db src, Mask mask, T* minval, T* maxval, 
                                            uint* minloc, uint* maxloc)
            {
                typedef typename MinMaxTypeTraits<T>::best_type best_type;
                __shared__ best_type sminval[nthreads];
                __shared__ best_type smaxval[nthreads];
                __shared__ uint sminloc[nthreads];
                __shared__ uint smaxloc[nthreads];

                uint x0 = blockIdx.x * blockDim.x * ctwidth + threadIdx.x;
                uint y0 = blockIdx.y * blockDim.y * ctheight + threadIdx.y;
                uint tid = threadIdx.y * blockDim.x + threadIdx.x;

                T mymin = numeric_limits<T>::max();
                T mymax = numeric_limits<T>::is_signed ? -numeric_limits<T>::max() : numeric_limits<T>::min(); 
                uint myminloc = 0;
                uint mymaxloc = 0;
                uint y_end = ::min(y0 + (ctheight - 1) * blockDim.y + 1, src.rows);
                uint x_end = ::min(x0 + (ctwidth - 1) * blockDim.x + 1, src.cols);

                for (uint y = y0; y < y_end; y += blockDim.y)
                {
                    const T* ptr = (const T*)src.ptr(y);
                    for (uint x = x0; x < x_end; x += blockDim.x)
                    {
                        if (mask(y, x))
                        {
                            T val = ptr[x];
                            if (val <= mymin) { mymin = val; myminloc = y * src.cols + x; }
                            if (val >= mymax) { mymax = val; mymaxloc = y * src.cols + x; }
                        }
                    }
                }

                sminval[tid] = mymin; 
                smaxval[tid] = mymax;
                sminloc[tid] = myminloc;
                smaxloc[tid] = mymaxloc;
                __syncthreads();

                findMinMaxLocInSmem<nthreads, best_type>(sminval, smaxval, sminloc, smaxloc, tid);

            #if __CUDA_ARCH__ >= 110
		        __shared__ bool is_last;

		        if (tid == 0)
		        {
			        minval[blockIdx.y * gridDim.x + blockIdx.x] = (T)sminval[0];
                    maxval[blockIdx.y * gridDim.x + blockIdx.x] = (T)smaxval[0];
                    minloc[blockIdx.y * gridDim.x + blockIdx.x] = sminloc[0];
                    maxloc[blockIdx.y * gridDim.x + blockIdx.x] = smaxloc[0];
			        __threadfence();

			        uint ticket = atomicInc(&blocks_finished, gridDim.x * gridDim.y);
			        is_last = ticket == gridDim.x * gridDim.y - 1;
		        }

		        __syncthreads();

		        if (is_last)
		        {
                    uint idx = ::min(tid, gridDim.x * gridDim.y - 1);

                    sminval[tid] = minval[idx];
                    smaxval[tid] = maxval[idx];
                    sminloc[tid] = minloc[idx];
                    smaxloc[tid] = maxloc[idx];
                    __syncthreads();

			        findMinMaxLocInSmem<nthreads, best_type>(sminval, smaxval, sminloc, smaxloc, tid);

                    if (tid == 0) 
                    {
                        minval[0] = (T)sminval[0];
                        maxval[0] = (T)smaxval[0];
                        minloc[0] = sminloc[0];
                        maxloc[0] = smaxloc[0];
                        blocks_finished = 0;
                    }
		        }
            #else
                if (tid == 0) 
                {
                    minval[blockIdx.y * gridDim.x + blockIdx.x] = (T)sminval[0];
                    maxval[blockIdx.y * gridDim.x + blockIdx.x] = (T)smaxval[0];
                    minloc[blockIdx.y * gridDim.x + blockIdx.x] = sminloc[0];
                    maxloc[blockIdx.y * gridDim.x + blockIdx.x] = smaxloc[0];
                }
            #endif
            }


            template <typename T>
            void minMaxLocMaskCaller(const DevMem2Db src, const PtrStepb mask, double* minval, double* maxval, 
                                     int minloc[2], int maxloc[2], PtrStepb valbuf, PtrStepb locbuf)
            {
                dim3 threads, grid;
                estimateThreadCfg(src.cols, src.rows, threads, grid);
                setKernelConsts(src.cols, src.rows, threads, grid);

                T* minval_buf = (T*)valbuf.ptr(0);
                T* maxval_buf = (T*)valbuf.ptr(1);
                uint* minloc_buf = (uint*)locbuf.ptr(0);
                uint* maxloc_buf = (uint*)locbuf.ptr(1);

                minMaxLocKernel<256, T, Mask8U><<<grid, threads>>>(src, Mask8U(mask), minval_buf, maxval_buf, 
                                                                   minloc_buf, maxloc_buf);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                T minval_, maxval_;
                cudaSafeCall( hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost) );
                cudaSafeCall( hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost) );
                *minval = minval_;
                *maxval = maxval_;

                uint minloc_, maxloc_;
                cudaSafeCall( hipMemcpy(&minloc_, minloc_buf, sizeof(int), hipMemcpyDeviceToHost) );
                cudaSafeCall( hipMemcpy(&maxloc_, maxloc_buf, sizeof(int), hipMemcpyDeviceToHost) );
                minloc[1] = minloc_ / src.cols; minloc[0] = minloc_ - minloc[1] * src.cols;
                maxloc[1] = maxloc_ / src.cols; maxloc[0] = maxloc_ - maxloc[1] * src.cols;
            }

            template void minMaxLocMaskCaller<uchar>(const DevMem2Db, const PtrStepb, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocMaskCaller<char>(const DevMem2Db, const PtrStepb, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocMaskCaller<ushort>(const DevMem2Db, const PtrStepb, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocMaskCaller<short>(const DevMem2Db, const PtrStepb, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocMaskCaller<int>(const DevMem2Db, const PtrStepb, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocMaskCaller<float>(const DevMem2Db, const PtrStepb, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocMaskCaller<double>(const DevMem2Db, const PtrStepb, double*, double*, int[2], int[2], PtrStepb, PtrStepb);


            template <typename T>
            void minMaxLocCaller(const DevMem2Db src, double* minval, double* maxval, 
                                 int minloc[2], int maxloc[2], PtrStepb valbuf, PtrStepb locbuf)
            {
                dim3 threads, grid;
                estimateThreadCfg(src.cols, src.rows, threads, grid);
                setKernelConsts(src.cols, src.rows, threads, grid);

                T* minval_buf = (T*)valbuf.ptr(0);
                T* maxval_buf = (T*)valbuf.ptr(1);
                uint* minloc_buf = (uint*)locbuf.ptr(0);
                uint* maxloc_buf = (uint*)locbuf.ptr(1);

                minMaxLocKernel<256, T, MaskTrue><<<grid, threads>>>(src, MaskTrue(), minval_buf, maxval_buf, 
                                                                     minloc_buf, maxloc_buf);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                T minval_, maxval_;
                cudaSafeCall(hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost));
                cudaSafeCall(hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost));
                *minval = minval_;
                *maxval = maxval_;

                uint minloc_, maxloc_;
                cudaSafeCall(hipMemcpy(&minloc_, minloc_buf, sizeof(int), hipMemcpyDeviceToHost));
                cudaSafeCall(hipMemcpy(&maxloc_, maxloc_buf, sizeof(int), hipMemcpyDeviceToHost));
                minloc[1] = minloc_ / src.cols; minloc[0] = minloc_ - minloc[1] * src.cols;
                maxloc[1] = maxloc_ / src.cols; maxloc[0] = maxloc_ - maxloc[1] * src.cols;
            }

            template void minMaxLocCaller<uchar>(const DevMem2Db, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocCaller<char>(const DevMem2Db, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocCaller<ushort>(const DevMem2Db, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocCaller<short>(const DevMem2Db, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocCaller<int>(const DevMem2Db, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocCaller<float>(const DevMem2Db, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocCaller<double>(const DevMem2Db, double*, double*, int[2], int[2], PtrStepb, PtrStepb);


            // This kernel will be used only when compute capability is 1.0
            template <int nthreads, typename T>
            __global__ void minMaxLocPass2Kernel(T* minval, T* maxval, uint* minloc, uint* maxloc, int size)
            {
                typedef typename MinMaxTypeTraits<T>::best_type best_type;
                __shared__ best_type sminval[nthreads];
                __shared__ best_type smaxval[nthreads];
                __shared__ uint sminloc[nthreads];
                __shared__ uint smaxloc[nthreads];

                uint tid = threadIdx.y * blockDim.x + threadIdx.x;
                uint idx = ::min(tid, size - 1);

                sminval[tid] = minval[idx];
                smaxval[tid] = maxval[idx];
                sminloc[tid] = minloc[idx];
                smaxloc[tid] = maxloc[idx];
                __syncthreads();

                findMinMaxLocInSmem<nthreads, best_type>(sminval, smaxval, sminloc, smaxloc, tid);

                if (tid == 0) 
                {
                    minval[0] = (T)sminval[0];
                    maxval[0] = (T)smaxval[0];
                    minloc[0] = sminloc[0];
                    maxloc[0] = smaxloc[0];
                }
            }


            template <typename T>
            void minMaxLocMaskMultipassCaller(const DevMem2Db src, const PtrStepb mask, double* minval, double* maxval, 
                                              int minloc[2], int maxloc[2], PtrStepb valbuf, PtrStepb locbuf)
            {
                dim3 threads, grid;
                estimateThreadCfg(src.cols, src.rows, threads, grid);
                setKernelConsts(src.cols, src.rows, threads, grid);

                T* minval_buf = (T*)valbuf.ptr(0);
                T* maxval_buf = (T*)valbuf.ptr(1);
                uint* minloc_buf = (uint*)locbuf.ptr(0);
                uint* maxloc_buf = (uint*)locbuf.ptr(1);

                minMaxLocKernel<256, T, Mask8U><<<grid, threads>>>(src, Mask8U(mask), minval_buf, maxval_buf, 
                                                                   minloc_buf, maxloc_buf);
                cudaSafeCall( hipGetLastError() );
                minMaxLocPass2Kernel<256, T><<<1, 256>>>(minval_buf, maxval_buf, minloc_buf, maxloc_buf, grid.x * grid.y);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                T minval_, maxval_;
                cudaSafeCall(hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost));
                cudaSafeCall(hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost));
                *minval = minval_;
                *maxval = maxval_;

                uint minloc_, maxloc_;
                cudaSafeCall(hipMemcpy(&minloc_, minloc_buf, sizeof(int), hipMemcpyDeviceToHost));
                cudaSafeCall(hipMemcpy(&maxloc_, maxloc_buf, sizeof(int), hipMemcpyDeviceToHost));
                minloc[1] = minloc_ / src.cols; minloc[0] = minloc_ - minloc[1] * src.cols;
                maxloc[1] = maxloc_ / src.cols; maxloc[0] = maxloc_ - maxloc[1] * src.cols;
            }

            template void minMaxLocMaskMultipassCaller<uchar>(const DevMem2Db, const PtrStepb, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocMaskMultipassCaller<char>(const DevMem2Db, const PtrStepb, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocMaskMultipassCaller<ushort>(const DevMem2Db, const PtrStepb, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocMaskMultipassCaller<short>(const DevMem2Db, const PtrStepb, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocMaskMultipassCaller<int>(const DevMem2Db, const PtrStepb, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocMaskMultipassCaller<float>(const DevMem2Db, const PtrStepb, double*, double*, int[2], int[2], PtrStepb, PtrStepb);


            template <typename T>
            void minMaxLocMultipassCaller(const DevMem2Db src, double* minval, double* maxval, 
                                          int minloc[2], int maxloc[2], PtrStepb valbuf, PtrStepb locbuf)
            {
                dim3 threads, grid;
                estimateThreadCfg(src.cols, src.rows, threads, grid);
                setKernelConsts(src.cols, src.rows, threads, grid);

                T* minval_buf = (T*)valbuf.ptr(0);
                T* maxval_buf = (T*)valbuf.ptr(1);
                uint* minloc_buf = (uint*)locbuf.ptr(0);
                uint* maxloc_buf = (uint*)locbuf.ptr(1);

                minMaxLocKernel<256, T, MaskTrue><<<grid, threads>>>(src, MaskTrue(), minval_buf, maxval_buf, 
                                                                     minloc_buf, maxloc_buf);
                cudaSafeCall( hipGetLastError() );
                minMaxLocPass2Kernel<256, T><<<1, 256>>>(minval_buf, maxval_buf, minloc_buf, maxloc_buf, grid.x * grid.y);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                T minval_, maxval_;
                cudaSafeCall(hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost));
                cudaSafeCall(hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost));
                *minval = minval_;
                *maxval = maxval_;

                uint minloc_, maxloc_;
                cudaSafeCall(hipMemcpy(&minloc_, minloc_buf, sizeof(int), hipMemcpyDeviceToHost));
                cudaSafeCall(hipMemcpy(&maxloc_, maxloc_buf, sizeof(int), hipMemcpyDeviceToHost));
                minloc[1] = minloc_ / src.cols; minloc[0] = minloc_ - minloc[1] * src.cols;
                maxloc[1] = maxloc_ / src.cols; maxloc[0] = maxloc_ - maxloc[1] * src.cols;
            }

            template void minMaxLocMultipassCaller<uchar>(const DevMem2Db, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocMultipassCaller<char>(const DevMem2Db, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocMultipassCaller<ushort>(const DevMem2Db, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocMultipassCaller<short>(const DevMem2Db, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocMultipassCaller<int>(const DevMem2Db, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
            template void minMaxLocMultipassCaller<float>(const DevMem2Db, double*, double*, int[2], int[2], PtrStepb, PtrStepb);
        } // namespace minmaxloc

        //////////////////////////////////////////////////////////////////////////////////////////////////////////
        // countNonZero

        namespace countnonzero 
        {
            __constant__ int ctwidth;
            __constant__ int ctheight;

            __device__ uint blocks_finished = 0;

            void estimateThreadCfg(int cols, int rows, dim3& threads, dim3& grid)
            {
                threads = dim3(32, 8);
                grid = dim3(divUp(cols, threads.x * 8), divUp(rows, threads.y * 32));
                grid.x = std::min(grid.x, threads.x);
                grid.y = std::min(grid.y, threads.y);
            }


            void getBufSizeRequired(int cols, int rows, int& bufcols, int& bufrows)
            {
                dim3 threads, grid;
                estimateThreadCfg(cols, rows, threads, grid);
                bufcols = grid.x * grid.y * sizeof(int);
                bufrows = 1;
            }


            void setKernelConsts(int cols, int rows, const dim3& threads, const dim3& grid)
            {        
                int twidth = divUp(divUp(cols, grid.x), threads.x);
                int theight = divUp(divUp(rows, grid.y), threads.y);
                cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctwidth), &twidth, sizeof(twidth))); 
                cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctheight), &theight, sizeof(theight))); 
            }


            template <int nthreads, typename T>
            __global__ void countNonZeroKernel(const DevMem2Db src, volatile uint* count)
            {
                __shared__ uint scount[nthreads];

                uint x0 = blockIdx.x * blockDim.x * ctwidth + threadIdx.x;
                uint y0 = blockIdx.y * blockDim.y * ctheight + threadIdx.y;
                uint tid = threadIdx.y * blockDim.x + threadIdx.x;

		        uint cnt = 0;
                for (uint y = 0; y < ctheight && y0 + y * blockDim.y < src.rows; ++y)
                {
                    const T* ptr = (const T*)src.ptr(y0 + y * blockDim.y);
                    for (uint x = 0; x < ctwidth && x0 + x * blockDim.x < src.cols; ++x)
				        cnt += ptr[x0 + x * blockDim.x] != 0;
		        }

		        scount[tid] = cnt;
		        __syncthreads();

                sumInSmem<nthreads, uint>(scount, tid);

            #if __CUDA_ARCH__ >= 110
		        __shared__ bool is_last;

		        if (tid == 0)
		        {
			        count[blockIdx.y * gridDim.x + blockIdx.x] = scount[0];
			        __threadfence();

			        uint ticket = atomicInc(&blocks_finished, gridDim.x * gridDim.y);
			        is_last = ticket == gridDim.x * gridDim.y - 1;
		        }

		        __syncthreads();

		        if (is_last)
		        {
                    scount[tid] = tid < gridDim.x * gridDim.y ? count[tid] : 0;
                    __syncthreads();

			        sumInSmem<nthreads, uint>(scount, tid);

			        if (tid == 0) 
                    {
                        count[0] = scount[0];
                        blocks_finished = 0;
                    }
		        }
            #else
                if (tid == 0) count[blockIdx.y * gridDim.x + blockIdx.x] = scount[0];
            #endif
            }

           
            template <typename T>
            int countNonZeroCaller(const DevMem2Db src, PtrStepb buf)
            {
                dim3 threads, grid;
                estimateThreadCfg(src.cols, src.rows, threads, grid);
                setKernelConsts(src.cols, src.rows, threads, grid);

                uint* count_buf = (uint*)buf.ptr(0);

                countNonZeroKernel<256, T><<<grid, threads>>>(src, count_buf);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                uint count;
                cudaSafeCall(hipMemcpy(&count, count_buf, sizeof(int), hipMemcpyDeviceToHost));
                
                return count;
            }  

            template int countNonZeroCaller<uchar>(const DevMem2Db, PtrStepb);
            template int countNonZeroCaller<char>(const DevMem2Db, PtrStepb);
            template int countNonZeroCaller<ushort>(const DevMem2Db, PtrStepb);
            template int countNonZeroCaller<short>(const DevMem2Db, PtrStepb);
            template int countNonZeroCaller<int>(const DevMem2Db, PtrStepb);
            template int countNonZeroCaller<float>(const DevMem2Db, PtrStepb);
            template int countNonZeroCaller<double>(const DevMem2Db, PtrStepb);


            template <int nthreads, typename T>
            __global__ void countNonZeroPass2Kernel(uint* count, int size)
            {
                __shared__ uint scount[nthreads];
                uint tid = threadIdx.y * blockDim.x + threadIdx.x;

                scount[tid] = tid < size ? count[tid] : 0;
                __syncthreads();

                sumInSmem<nthreads, uint>(scount, tid);

                if (tid == 0) 
                    count[0] = scount[0];
            }


            template <typename T>
            int countNonZeroMultipassCaller(const DevMem2Db src, PtrStepb buf)
            {
                dim3 threads, grid;
                estimateThreadCfg(src.cols, src.rows, threads, grid);
                setKernelConsts(src.cols, src.rows, threads, grid);

                uint* count_buf = (uint*)buf.ptr(0);

                countNonZeroKernel<256, T><<<grid, threads>>>(src, count_buf);
                cudaSafeCall( hipGetLastError() );
                countNonZeroPass2Kernel<256, T><<<1, 256>>>(count_buf, grid.x * grid.y);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                uint count;
                cudaSafeCall(hipMemcpy(&count, count_buf, sizeof(int), hipMemcpyDeviceToHost));
                
                return count;
            }  

            template int countNonZeroMultipassCaller<uchar>(const DevMem2Db, PtrStepb);
            template int countNonZeroMultipassCaller<char>(const DevMem2Db, PtrStepb);
            template int countNonZeroMultipassCaller<ushort>(const DevMem2Db, PtrStepb);
            template int countNonZeroMultipassCaller<short>(const DevMem2Db, PtrStepb);
            template int countNonZeroMultipassCaller<int>(const DevMem2Db, PtrStepb);
            template int countNonZeroMultipassCaller<float>(const DevMem2Db, PtrStepb);

        } // namespace countnonzero


        //////////////////////////////////////////////////////////////////////////
        // Sum

        namespace sum
        {
            template <typename T> struct SumType {};
            template <> struct SumType<uchar> { typedef uint R; };
            template <> struct SumType<char> { typedef int R; };
            template <> struct SumType<ushort> { typedef uint R; };
            template <> struct SumType<short> { typedef int R; };
            template <> struct SumType<int> { typedef int R; };
            template <> struct SumType<float> { typedef float R; };
            template <> struct SumType<double> { typedef double R; };

            template <typename R> 
            struct IdentityOp { static __device__ __forceinline__ R call(R x) { return x; } };

            template <typename R> 
            struct AbsOp { static __device__ __forceinline__ R call(R x) { return ::abs(x); } };

            template <>
            struct AbsOp<uint> { static __device__ __forceinline__ uint call(uint x) { return x; } };

            template <typename R> 
            struct SqrOp { static __device__ __forceinline__ R call(R x) { return x * x; } };

            __constant__ int ctwidth;
            __constant__ int ctheight;
            __device__ uint blocks_finished = 0;

            const int threads_x = 32;
            const int threads_y = 8;

            void estimateThreadCfg(int cols, int rows, dim3& threads, dim3& grid)
            {
                threads = dim3(threads_x, threads_y);
                grid = dim3(divUp(cols, threads.x * threads.y), 
                            divUp(rows, threads.y * threads.x));
                grid.x = std::min(grid.x, threads.x);
                grid.y = std::min(grid.y, threads.y);
            }


            void getBufSizeRequired(int cols, int rows, int cn, int& bufcols, int& bufrows)
            {
                dim3 threads, grid;
                estimateThreadCfg(cols, rows, threads, grid);
                bufcols = grid.x * grid.y * sizeof(double) * cn;
                bufrows = 1;
            }


            void setKernelConsts(int cols, int rows, const dim3& threads, const dim3& grid)
            {        
                int twidth = divUp(divUp(cols, grid.x), threads.x);
                int theight = divUp(divUp(rows, grid.y), threads.y);
                cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctwidth), &twidth, sizeof(twidth))); 
                cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctheight), &theight, sizeof(theight))); 
            }

            template <typename T, typename R, typename Op, int nthreads>
            __global__ void sumKernel(const DevMem2Db src, R* result)
            {
                __shared__ R smem[nthreads];

                const int x0 = blockIdx.x * blockDim.x * ctwidth + threadIdx.x;
                const int y0 = blockIdx.y * blockDim.y * ctheight + threadIdx.y;
                const int tid = threadIdx.y * blockDim.x + threadIdx.x;
                const int bid = blockIdx.y * gridDim.x + blockIdx.x;

                R sum = 0;
                for (int y = 0; y < ctheight && y0 + y * blockDim.y < src.rows; ++y)
                {
                    const T* ptr = (const T*)src.ptr(y0 + y * blockDim.y);
                    for (int x = 0; x < ctwidth && x0 + x * blockDim.x < src.cols; ++x)
                        sum += Op::call(ptr[x0 + x * blockDim.x]);
                }

                smem[tid] = sum;
                __syncthreads();

                sumInSmem<nthreads, R>(smem, tid);

            #if __CUDA_ARCH__ >= 110
                __shared__ bool is_last;

                if (tid == 0)
                {
                    result[bid] = smem[0];
                    __threadfence();

                    uint ticket = atomicInc(&blocks_finished, gridDim.x * gridDim.y);
                    is_last = (ticket == gridDim.x * gridDim.y - 1);
                }

                __syncthreads();

                if (is_last)
                {
                    smem[tid] = tid < gridDim.x * gridDim.y ? result[tid] : 0;
                    __syncthreads();

                    sumInSmem<nthreads, R>(smem, tid);

                    if (tid == 0) 
                    {
                        result[0] = smem[0];
                        blocks_finished = 0;
                    }
                }
            #else
                if (tid == 0) result[bid] = smem[0];
            #endif
            }


            template <typename T, typename R, int nthreads>
            __global__ void sumPass2Kernel(R* result, int size)
            {
                __shared__ R smem[nthreads];
                int tid = threadIdx.y * blockDim.x + threadIdx.x;

                smem[tid] = tid < size ? result[tid] : 0;
                __syncthreads();

                sumInSmem<nthreads, R>(smem, tid);

                if (tid == 0) 
                    result[0] = smem[0];
            }


            template <typename T, typename R, typename Op, int nthreads>
            __global__ void sumKernel_C2(const DevMem2Db src, typename TypeVec<R, 2>::vec_type* result)
            {
                typedef typename TypeVec<T, 2>::vec_type SrcType;
                typedef typename TypeVec<R, 2>::vec_type DstType;

                __shared__ R smem[nthreads * 2];

                const int x0 = blockIdx.x * blockDim.x * ctwidth + threadIdx.x;
                const int y0 = blockIdx.y * blockDim.y * ctheight + threadIdx.y;
                const int tid = threadIdx.y * blockDim.x + threadIdx.x;
                const int bid = blockIdx.y * gridDim.x + blockIdx.x;

                SrcType val;
                DstType sum = VecTraits<DstType>::all(0);
                for (int y = 0; y < ctheight && y0 + y * blockDim.y < src.rows; ++y)
                {
                    const SrcType* ptr = (const SrcType*)src.ptr(y0 + y * blockDim.y);
                    for (int x = 0; x < ctwidth && x0 + x * blockDim.x < src.cols; ++x)
                    {
                        val = ptr[x0 + x * blockDim.x];
                        sum = sum + VecTraits<DstType>::make(Op::call(val.x), Op::call(val.y));
                    }
                }

                smem[tid] = sum.x;
                smem[tid + nthreads] = sum.y;
                __syncthreads();

                sumInSmem<nthreads, R>(smem, tid);
                sumInSmem<nthreads, R>(smem + nthreads, tid);

            #if __CUDA_ARCH__ >= 110
                __shared__ bool is_last;

                if (tid == 0)
                {
                    DstType res;
                    res.x = smem[0];
                    res.y = smem[nthreads];
                    result[bid] = res;
                    __threadfence();

                    uint ticket = atomicInc(&blocks_finished, gridDim.x * gridDim.y);
                    is_last = (ticket == gridDim.x * gridDim.y - 1);
                }

                __syncthreads();

                if (is_last)
                {
                    DstType res = tid < gridDim.x * gridDim.y ? result[tid] : VecTraits<DstType>::all(0);
                    smem[tid] = res.x;
                    smem[tid + nthreads] = res.y;
                    __syncthreads();

                    sumInSmem<nthreads, R>(smem, tid);
                    sumInSmem<nthreads, R>(smem + nthreads, tid);

                    if (tid == 0) 
                    {
                        res.x = smem[0];
                        res.y = smem[nthreads];
                        result[0] = res;
                        blocks_finished = 0;
                    }
                }
            #else
                if (tid == 0) 
                {
                    DstType res;
                    res.x = smem[0];
                    res.y = smem[nthreads];
                    result[bid] = res;
                }
            #endif
            }


            template <typename T, typename R, int nthreads>
            __global__ void sumPass2Kernel_C2(typename TypeVec<R, 2>::vec_type* result, int size)
            {
                typedef typename TypeVec<R, 2>::vec_type DstType;

                __shared__ R smem[nthreads * 2];

                const int tid = threadIdx.y * blockDim.x + threadIdx.x;

                DstType res = tid < size ? result[tid] : VecTraits<DstType>::all(0);
                smem[tid] = res.x;
                smem[tid + nthreads] = res.y;
                __syncthreads();

                sumInSmem<nthreads, R>(smem, tid);
                sumInSmem<nthreads, R>(smem + nthreads, tid);

                if (tid == 0) 
                {
                    res.x = smem[0];
                    res.y = smem[nthreads];
                    result[0] = res;
                }
            }


            template <typename T, typename R, typename Op, int nthreads>
            __global__ void sumKernel_C3(const DevMem2Db src, typename TypeVec<R, 3>::vec_type* result)
            {
                typedef typename TypeVec<T, 3>::vec_type SrcType;
                typedef typename TypeVec<R, 3>::vec_type DstType;

                __shared__ R smem[nthreads * 3];

                const int x0 = blockIdx.x * blockDim.x * ctwidth + threadIdx.x;
                const int y0 = blockIdx.y * blockDim.y * ctheight + threadIdx.y;
                const int tid = threadIdx.y * blockDim.x + threadIdx.x;
                const int bid = blockIdx.y * gridDim.x + blockIdx.x;

                SrcType val;
                DstType sum = VecTraits<DstType>::all(0);
                for (int y = 0; y < ctheight && y0 + y * blockDim.y < src.rows; ++y)
                {
                    const SrcType* ptr = (const SrcType*)src.ptr(y0 + y * blockDim.y);
                    for (int x = 0; x < ctwidth && x0 + x * blockDim.x < src.cols; ++x)
                    {
                        val = ptr[x0 + x * blockDim.x];
                        sum = sum + VecTraits<DstType>::make(Op::call(val.x), Op::call(val.y), Op::call(val.z));
                    }
                }

                smem[tid] = sum.x;
                smem[tid + nthreads] = sum.y;
                smem[tid + 2 * nthreads] = sum.z;
                __syncthreads();

                sumInSmem<nthreads, R>(smem, tid);
                sumInSmem<nthreads, R>(smem + nthreads, tid);
                sumInSmem<nthreads, R>(smem + 2 * nthreads, tid);

            #if __CUDA_ARCH__ >= 110
                __shared__ bool is_last;

                if (tid == 0)
                {
                    DstType res;
                    res.x = smem[0];
                    res.y = smem[nthreads];
                    res.z = smem[2 * nthreads];
                    result[bid] = res;
                    __threadfence();

                    uint ticket = atomicInc(&blocks_finished, gridDim.x * gridDim.y);
                    is_last = (ticket == gridDim.x * gridDim.y - 1);
                }

                __syncthreads();

                if (is_last)
                {
                    DstType res = tid < gridDim.x * gridDim.y ? result[tid] : VecTraits<DstType>::all(0);
                    smem[tid] = res.x;
                    smem[tid + nthreads] = res.y;
                    smem[tid + 2 * nthreads] = res.z;
                    __syncthreads();

                    sumInSmem<nthreads, R>(smem, tid);
                    sumInSmem<nthreads, R>(smem + nthreads, tid);
                    sumInSmem<nthreads, R>(smem + 2 * nthreads, tid);

                    if (tid == 0) 
                    {
                        res.x = smem[0];
                        res.y = smem[nthreads];
                        res.z = smem[2 * nthreads];
                        result[0] = res;
                        blocks_finished = 0;
                    }
                }
            #else
                if (tid == 0) 
                {
                    DstType res;
                    res.x = smem[0];
                    res.y = smem[nthreads];
                    res.z = smem[2 * nthreads];
                    result[bid] = res;
                }
            #endif
            }


            template <typename T, typename R, int nthreads>
            __global__ void sumPass2Kernel_C3(typename TypeVec<R, 3>::vec_type* result, int size)
            {
                typedef typename TypeVec<R, 3>::vec_type DstType;

                __shared__ R smem[nthreads * 3];

                const int tid = threadIdx.y * blockDim.x + threadIdx.x;

                DstType res = tid < size ? result[tid] : VecTraits<DstType>::all(0);
                smem[tid] = res.x;
                smem[tid + nthreads] = res.y;
                smem[tid + 2 * nthreads] = res.z;
                __syncthreads();

                sumInSmem<nthreads, R>(smem, tid);
                sumInSmem<nthreads, R>(smem + nthreads, tid);
                sumInSmem<nthreads, R>(smem + 2 * nthreads, tid);

                if (tid == 0) 
                {
                    res.x = smem[0];
                    res.y = smem[nthreads];
                    res.z = smem[2 * nthreads];
                    result[0] = res;
                }
            }

            template <typename T, typename R, typename Op, int nthreads>
            __global__ void sumKernel_C4(const DevMem2Db src, typename TypeVec<R, 4>::vec_type* result)
            {
                typedef typename TypeVec<T, 4>::vec_type SrcType;
                typedef typename TypeVec<R, 4>::vec_type DstType;

                __shared__ R smem[nthreads * 4];

                const int x0 = blockIdx.x * blockDim.x * ctwidth + threadIdx.x;
                const int y0 = blockIdx.y * blockDim.y * ctheight + threadIdx.y;
                const int tid = threadIdx.y * blockDim.x + threadIdx.x;
                const int bid = blockIdx.y * gridDim.x + blockIdx.x;

                SrcType val;
                DstType sum = VecTraits<DstType>::all(0);
                for (int y = 0; y < ctheight && y0 + y * blockDim.y < src.rows; ++y)
                {
                    const SrcType* ptr = (const SrcType*)src.ptr(y0 + y * blockDim.y);
                    for (int x = 0; x < ctwidth && x0 + x * blockDim.x < src.cols; ++x)
                    {
                        val = ptr[x0 + x * blockDim.x];
                        sum = sum + VecTraits<DstType>::make(Op::call(val.x), Op::call(val.y), 
                                                             Op::call(val.z), Op::call(val.w));
                    }
                }

                smem[tid] = sum.x;
                smem[tid + nthreads] = sum.y;
                smem[tid + 2 * nthreads] = sum.z;
                smem[tid + 3 * nthreads] = sum.w;
                __syncthreads();

                sumInSmem<nthreads, R>(smem, tid);
                sumInSmem<nthreads, R>(smem + nthreads, tid);
                sumInSmem<nthreads, R>(smem + 2 * nthreads, tid);
                sumInSmem<nthreads, R>(smem + 3 * nthreads, tid);

            #if __CUDA_ARCH__ >= 110
                __shared__ bool is_last;

                if (tid == 0)
                {
                    DstType res;
                    res.x = smem[0];
                    res.y = smem[nthreads];
                    res.z = smem[2 * nthreads];
                    res.w = smem[3 * nthreads];
                    result[bid] = res;
                    __threadfence();

                    uint ticket = atomicInc(&blocks_finished, gridDim.x * gridDim.y);
                    is_last = (ticket == gridDim.x * gridDim.y - 1);
                }

                __syncthreads();

                if (is_last)
                {
                    DstType res = tid < gridDim.x * gridDim.y ? result[tid] : VecTraits<DstType>::all(0);
                    smem[tid] = res.x;
                    smem[tid + nthreads] = res.y;
                    smem[tid + 2 * nthreads] = res.z;
                    smem[tid + 3 * nthreads] = res.w;
                    __syncthreads();

                    sumInSmem<nthreads, R>(smem, tid);
                    sumInSmem<nthreads, R>(smem + nthreads, tid);
                    sumInSmem<nthreads, R>(smem + 2 * nthreads, tid);
                    sumInSmem<nthreads, R>(smem + 3 * nthreads, tid);

                    if (tid == 0) 
                    {
                        res.x = smem[0];
                        res.y = smem[nthreads];
                        res.z = smem[2 * nthreads];
                        res.w = smem[3 * nthreads];
                        result[0] = res;
                        blocks_finished = 0;
                    }
                }
            #else
                if (tid == 0) 
                {
                    DstType res;
                    res.x = smem[0];
                    res.y = smem[nthreads];
                    res.z = smem[2 * nthreads];
                    res.w = smem[3 * nthreads];
                    result[bid] = res;
                }
            #endif
            }


            template <typename T, typename R, int nthreads>
            __global__ void sumPass2Kernel_C4(typename TypeVec<R, 4>::vec_type* result, int size)
            {
                typedef typename TypeVec<R, 4>::vec_type DstType;

                __shared__ R smem[nthreads * 4];

                const int tid = threadIdx.y * blockDim.x + threadIdx.x;

                DstType res = tid < size ? result[tid] : VecTraits<DstType>::all(0);
                smem[tid] = res.x;
                smem[tid + nthreads] = res.y;
                smem[tid + 2 * nthreads] = res.z;
                smem[tid + 3 * nthreads] = res.w;
                __syncthreads();

                sumInSmem<nthreads, R>(smem, tid);
                sumInSmem<nthreads, R>(smem + nthreads, tid);
                sumInSmem<nthreads, R>(smem + 2 * nthreads, tid);
                sumInSmem<nthreads, R>(smem + 3 * nthreads, tid);

                if (tid == 0) 
                {
                    res.x = smem[0];
                    res.y = smem[nthreads];
                    res.z = smem[2 * nthreads];
                    res.w = smem[3 * nthreads];
                    result[0] = res;
                }
            }

            template <typename T>
            void sumMultipassCaller(const DevMem2Db src, PtrStepb buf, double* sum, int cn)
            {
                typedef typename SumType<T>::R R;

                dim3 threads, grid;
                estimateThreadCfg(src.cols, src.rows, threads, grid);
                setKernelConsts(src.cols, src.rows, threads, grid);

                switch (cn)
                {
                case 1:
                    sumKernel<T, R, IdentityOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 1>::vec_type*)buf.ptr(0));
                    cudaSafeCall( hipGetLastError() );

                    sumPass2Kernel<T, R, threads_x * threads_y><<<1, threads_x * threads_y>>>(
                            (typename TypeVec<R, 1>::vec_type*)buf.ptr(0), grid.x * grid.y);
                    cudaSafeCall( hipGetLastError() );

                    break;
                case 2:
                    sumKernel_C2<T, R, IdentityOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 2>::vec_type*)buf.ptr(0));
                    cudaSafeCall( hipGetLastError() );

                    sumPass2Kernel_C2<T, R, threads_x * threads_y><<<1, threads_x * threads_y>>>(
                            (typename TypeVec<R, 2>::vec_type*)buf.ptr(0), grid.x * grid.y);
                    cudaSafeCall( hipGetLastError() );

                    break;
                case 3:
                    sumKernel_C3<T, R, IdentityOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 3>::vec_type*)buf.ptr(0));
                    cudaSafeCall( hipGetLastError() );

                    sumPass2Kernel_C3<T, R, threads_x * threads_y><<<1, threads_x * threads_y>>>(
                            (typename TypeVec<R, 3>::vec_type*)buf.ptr(0), grid.x * grid.y);
                    cudaSafeCall( hipGetLastError() );

                    break;
                case 4:
                    sumKernel_C4<T, R, IdentityOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 4>::vec_type*)buf.ptr(0));
                    cudaSafeCall( hipGetLastError() );

                    sumPass2Kernel_C4<T, R, threads_x * threads_y><<<1, threads_x * threads_y>>>(
                            (typename TypeVec<R, 4>::vec_type*)buf.ptr(0), grid.x * grid.y);
                    cudaSafeCall( hipGetLastError() );

                    break;
                }
                cudaSafeCall( hipDeviceSynchronize() );

                R result[4] = {0, 0, 0, 0};
                cudaSafeCall(hipMemcpy(&result, buf.ptr(0), sizeof(R) * cn, hipMemcpyDeviceToHost));

                sum[0] = result[0];
                sum[1] = result[1];
                sum[2] = result[2];
                sum[3] = result[3];
            }  

            template void sumMultipassCaller<uchar>(const DevMem2Db, PtrStepb, double*, int);
            template void sumMultipassCaller<char>(const DevMem2Db, PtrStepb, double*, int);
            template void sumMultipassCaller<ushort>(const DevMem2Db, PtrStepb, double*, int);
            template void sumMultipassCaller<short>(const DevMem2Db, PtrStepb, double*, int);
            template void sumMultipassCaller<int>(const DevMem2Db, PtrStepb, double*, int);
            template void sumMultipassCaller<float>(const DevMem2Db, PtrStepb, double*, int);


            template <typename T>
            void sumCaller(const DevMem2Db src, PtrStepb buf, double* sum, int cn)
            {
                typedef typename SumType<T>::R R;

                dim3 threads, grid;
                estimateThreadCfg(src.cols, src.rows, threads, grid);
                setKernelConsts(src.cols, src.rows, threads, grid);

                switch (cn)
                {
                case 1:
                    sumKernel<T, R, IdentityOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 1>::vec_type*)buf.ptr(0));
                    break;
                case 2:
                    sumKernel_C2<T, R, IdentityOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 2>::vec_type*)buf.ptr(0));
                    break;
                case 3:
                    sumKernel_C3<T, R, IdentityOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 3>::vec_type*)buf.ptr(0));
                    break;
                case 4:
                    sumKernel_C4<T, R, IdentityOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 4>::vec_type*)buf.ptr(0));
                    break;
                }
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                R result[4] = {0, 0, 0, 0};
                cudaSafeCall(hipMemcpy(&result, buf.ptr(0), sizeof(R) * cn, hipMemcpyDeviceToHost));

                sum[0] = result[0];
                sum[1] = result[1];
                sum[2] = result[2];
                sum[3] = result[3];
            }  

            template void sumCaller<uchar>(const DevMem2Db, PtrStepb, double*, int);
            template void sumCaller<char>(const DevMem2Db, PtrStepb, double*, int);
            template void sumCaller<ushort>(const DevMem2Db, PtrStepb, double*, int);
            template void sumCaller<short>(const DevMem2Db, PtrStepb, double*, int);
            template void sumCaller<int>(const DevMem2Db, PtrStepb, double*, int);
            template void sumCaller<float>(const DevMem2Db, PtrStepb, double*, int);


            template <typename T>
            void absSumMultipassCaller(const DevMem2Db src, PtrStepb buf, double* sum, int cn)
            {
                typedef typename SumType<T>::R R;

                dim3 threads, grid;
                estimateThreadCfg(src.cols, src.rows, threads, grid);
                setKernelConsts(src.cols, src.rows, threads, grid);

                switch (cn)
                {
                case 1:
                    sumKernel<T, R, AbsOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 1>::vec_type*)buf.ptr(0));
                    cudaSafeCall( hipGetLastError() );

                    sumPass2Kernel<T, R, threads_x * threads_y><<<1, threads_x * threads_y>>>(
                            (typename TypeVec<R, 1>::vec_type*)buf.ptr(0), grid.x * grid.y);
                    cudaSafeCall( hipGetLastError() );

                    break;
                case 2:
                    sumKernel_C2<T, R, AbsOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 2>::vec_type*)buf.ptr(0));
                    cudaSafeCall( hipGetLastError() );

                    sumPass2Kernel_C2<T, R, threads_x * threads_y><<<1, threads_x * threads_y>>>(
                            (typename TypeVec<R, 2>::vec_type*)buf.ptr(0), grid.x * grid.y);
                    cudaSafeCall( hipGetLastError() );

                    break;
                case 3:
                    sumKernel_C3<T, R, AbsOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 3>::vec_type*)buf.ptr(0));
                    cudaSafeCall( hipGetLastError() );

                    sumPass2Kernel_C3<T, R, threads_x * threads_y><<<1, threads_x * threads_y>>>(
                            (typename TypeVec<R, 3>::vec_type*)buf.ptr(0), grid.x * grid.y);
                    cudaSafeCall( hipGetLastError() );

                    break;
                case 4:
                    sumKernel_C4<T, R, AbsOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 4>::vec_type*)buf.ptr(0));
                    cudaSafeCall( hipGetLastError() );

                    sumPass2Kernel_C4<T, R, threads_x * threads_y><<<1, threads_x * threads_y>>>(
                            (typename TypeVec<R, 4>::vec_type*)buf.ptr(0), grid.x * grid.y);
                    cudaSafeCall( hipGetLastError() );

                    break;
                }
                cudaSafeCall( hipDeviceSynchronize() );

                R result[4] = {0, 0, 0, 0};
                cudaSafeCall(hipMemcpy(result, buf.ptr(0), sizeof(R) * cn, hipMemcpyDeviceToHost));

                sum[0] = result[0];
                sum[1] = result[1];
                sum[2] = result[2];
                sum[3] = result[3];
            }  

            template void absSumMultipassCaller<uchar>(const DevMem2Db, PtrStepb, double*, int);
            template void absSumMultipassCaller<char>(const DevMem2Db, PtrStepb, double*, int);
            template void absSumMultipassCaller<ushort>(const DevMem2Db, PtrStepb, double*, int);
            template void absSumMultipassCaller<short>(const DevMem2Db, PtrStepb, double*, int);
            template void absSumMultipassCaller<int>(const DevMem2Db, PtrStepb, double*, int);
            template void absSumMultipassCaller<float>(const DevMem2Db, PtrStepb, double*, int);


            template <typename T>
            void absSumCaller(const DevMem2Db src, PtrStepb buf, double* sum, int cn)
            {
                typedef typename SumType<T>::R R;

                dim3 threads, grid;
                estimateThreadCfg(src.cols, src.rows, threads, grid);
                setKernelConsts(src.cols, src.rows, threads, grid);

                switch (cn)
                {
                case 1:
                    sumKernel<T, R, AbsOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 1>::vec_type*)buf.ptr(0));
                    break;
                case 2:
                    sumKernel_C2<T, R, AbsOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 2>::vec_type*)buf.ptr(0));
                    break;
                case 3:
                    sumKernel_C3<T, R, AbsOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 3>::vec_type*)buf.ptr(0));
                    break;
                case 4:
                    sumKernel_C4<T, R, AbsOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 4>::vec_type*)buf.ptr(0));
                    break;
                }
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                R result[4] = {0, 0, 0, 0};
                cudaSafeCall(hipMemcpy(result, buf.ptr(0), sizeof(R) * cn, hipMemcpyDeviceToHost));

                sum[0] = result[0];
                sum[1] = result[1];
                sum[2] = result[2];
                sum[3] = result[3];
            }

            template void absSumCaller<uchar>(const DevMem2Db, PtrStepb, double*, int);
            template void absSumCaller<char>(const DevMem2Db, PtrStepb, double*, int);
            template void absSumCaller<ushort>(const DevMem2Db, PtrStepb, double*, int);
            template void absSumCaller<short>(const DevMem2Db, PtrStepb, double*, int);
            template void absSumCaller<int>(const DevMem2Db, PtrStepb, double*, int);
            template void absSumCaller<float>(const DevMem2Db, PtrStepb, double*, int);


            template <typename T>
            void sqrSumMultipassCaller(const DevMem2Db src, PtrStepb buf, double* sum, int cn)
            {
                typedef typename SumType<T>::R R;

                dim3 threads, grid;
                estimateThreadCfg(src.cols, src.rows, threads, grid);
                setKernelConsts(src.cols, src.rows, threads, grid);

                switch (cn)
                {
                case 1:
                    sumKernel<T, R, SqrOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 1>::vec_type*)buf.ptr(0));
                    cudaSafeCall( hipGetLastError() );

                    sumPass2Kernel<T, R, threads_x * threads_y><<<1, threads_x * threads_y>>>(
                            (typename TypeVec<R, 1>::vec_type*)buf.ptr(0), grid.x * grid.y);
                    cudaSafeCall( hipGetLastError() );

                    break;
                case 2:
                    sumKernel_C2<T, R, SqrOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 2>::vec_type*)buf.ptr(0));
                    cudaSafeCall( hipGetLastError() );

                    sumPass2Kernel_C2<T, R, threads_x * threads_y><<<1, threads_x * threads_y>>>(
                            (typename TypeVec<R, 2>::vec_type*)buf.ptr(0), grid.x * grid.y);
                    cudaSafeCall( hipGetLastError() );

                    break;
                case 3:
                    sumKernel_C3<T, R, SqrOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 3>::vec_type*)buf.ptr(0));
                    cudaSafeCall( hipGetLastError() );

                    sumPass2Kernel_C3<T, R, threads_x * threads_y><<<1, threads_x * threads_y>>>(
                            (typename TypeVec<R, 3>::vec_type*)buf.ptr(0), grid.x * grid.y);
                    cudaSafeCall( hipGetLastError() );

                    break;
                case 4:
                    sumKernel_C4<T, R, SqrOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 4>::vec_type*)buf.ptr(0));
                    cudaSafeCall( hipGetLastError() );

                    sumPass2Kernel_C4<T, R, threads_x * threads_y><<<1, threads_x * threads_y>>>(
                            (typename TypeVec<R, 4>::vec_type*)buf.ptr(0), grid.x * grid.y);
                    cudaSafeCall( hipGetLastError() );

                    break;
                }
                cudaSafeCall( hipDeviceSynchronize() );

                R result[4] = {0, 0, 0, 0};
                cudaSafeCall(hipMemcpy(result, buf.ptr(0), sizeof(R) * cn, hipMemcpyDeviceToHost));

                sum[0] = result[0];
                sum[1] = result[1];
                sum[2] = result[2];
                sum[3] = result[3];
            }  

            template void sqrSumMultipassCaller<uchar>(const DevMem2Db, PtrStepb, double*, int);
            template void sqrSumMultipassCaller<char>(const DevMem2Db, PtrStepb, double*, int);
            template void sqrSumMultipassCaller<ushort>(const DevMem2Db, PtrStepb, double*, int);
            template void sqrSumMultipassCaller<short>(const DevMem2Db, PtrStepb, double*, int);
            template void sqrSumMultipassCaller<int>(const DevMem2Db, PtrStepb, double*, int);
            template void sqrSumMultipassCaller<float>(const DevMem2Db, PtrStepb, double*, int);


            template <typename T>
            void sqrSumCaller(const DevMem2Db src, PtrStepb buf, double* sum, int cn)
            {
                typedef double R;

                dim3 threads, grid;
                estimateThreadCfg(src.cols, src.rows, threads, grid);
                setKernelConsts(src.cols, src.rows, threads, grid);

                switch (cn)
                {
                case 1:
                    sumKernel<T, R, SqrOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 1>::vec_type*)buf.ptr(0));
                    break;
                case 2:
                    sumKernel_C2<T, R, SqrOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 2>::vec_type*)buf.ptr(0));
                    break;
                case 3:
                    sumKernel_C3<T, R, SqrOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 3>::vec_type*)buf.ptr(0));
                    break;
                case 4:
                    sumKernel_C4<T, R, SqrOp<R>, threads_x * threads_y><<<grid, threads>>>(
                            src, (typename TypeVec<R, 4>::vec_type*)buf.ptr(0));
                    break;
                }
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                R result[4] = {0, 0, 0, 0};
                cudaSafeCall(hipMemcpy(result, buf.ptr(0), sizeof(R) * cn, hipMemcpyDeviceToHost));

                sum[0] = result[0];
                sum[1] = result[1];
                sum[2] = result[2];
                sum[3] = result[3];
            }

            template void sqrSumCaller<uchar>(const DevMem2Db, PtrStepb, double*, int);
            template void sqrSumCaller<char>(const DevMem2Db, PtrStepb, double*, int);
            template void sqrSumCaller<ushort>(const DevMem2Db, PtrStepb, double*, int);
            template void sqrSumCaller<short>(const DevMem2Db, PtrStepb, double*, int);
            template void sqrSumCaller<int>(const DevMem2Db, PtrStepb, double*, int);
            template void sqrSumCaller<float>(const DevMem2Db, PtrStepb, double*, int);
        } // namespace sum

        //////////////////////////////////////////////////////////////////////////////
        // reduce

        template <typename S> struct SumReductor
        {
            __device__ __forceinline__ S startValue() const
            {
                return 0;
            }

            __device__ __forceinline__ S operator ()(volatile S a, volatile S b) const
            {
                return a + b;
            }

            __device__ __forceinline__ S result(S r, double) const
            {
                return r;
            }
        };

        template <typename S> struct AvgReductor
        {
            __device__ __forceinline__ S startValue() const
            {
                return 0;
            }

            __device__ __forceinline__ S operator ()(volatile S a, volatile S b) const
            {
                return a + b;
            }

            __device__ __forceinline__ double result(S r, double sz) const
            {
                return r / sz;
            }
        };

        template <typename S> struct MinReductor
        {
            __device__ __forceinline__ S startValue() const
            {
                return numeric_limits<S>::max();
            }

            template <typename T> __device__ __forceinline__ T operator ()(volatile T a, volatile T b) const
            {
                return saturate_cast<T>(::min(a, b));
            }
            __device__ __forceinline__ float operator ()(volatile float a, volatile float b) const
            {
                return ::fmin(a, b);
            }

            __device__ __forceinline__ S result(S r, double) const
            {
                return r;
            }
        };

        template <typename S> struct MaxReductor
        {
            __device__ __forceinline__ S startValue() const
            {
                return numeric_limits<S>::min();
            }

            template <typename T> __device__ __forceinline__ int operator ()(volatile T a, volatile T b) const
            {
                return ::max(a, b);
            }
            __device__ __forceinline__ float operator ()(volatile float a, volatile float b) const
            {
                return ::fmax(a, b);
            }

            __device__ __forceinline__ S result(S r, double) const
            {
                return r;
            }
        };

        template <class Op, typename T, typename S, typename D> __global__ void reduceRows(const DevMem2D_<T> src, D* dst, const Op op)
        {
            __shared__ S smem[16 * 16];

            const int x = blockIdx.x * 16 + threadIdx.x;

            S myVal = op.startValue();

            if (x < src.cols)
            {
                for (int y = threadIdx.y; y < src.rows; y += 16)
                    myVal = op(myVal, src.ptr(y)[x]);
            }        

            smem[threadIdx.x * 16 + threadIdx.y] = myVal;
            __syncthreads();

            if (threadIdx.x < 8)
            {
                volatile S* srow = smem + threadIdx.y * 16;
                srow[threadIdx.x] = op(srow[threadIdx.x], srow[threadIdx.x + 8]);
                srow[threadIdx.x] = op(srow[threadIdx.x], srow[threadIdx.x + 4]);
                srow[threadIdx.x] = op(srow[threadIdx.x], srow[threadIdx.x + 2]);
                srow[threadIdx.x] = op(srow[threadIdx.x], srow[threadIdx.x + 1]);
            }
            __syncthreads();

            if (threadIdx.y == 0 && x < src.cols)
                dst[x] = saturate_cast<D>(op.result(smem[threadIdx.x * 16], src.rows));
        }

        template <template <typename> class Op, typename T, typename S, typename D> void reduceRows_caller(const DevMem2D_<T>& src, DevMem2D_<D> dst, hipStream_t stream)
        {
            const dim3 block(16, 16);
            const dim3 grid(divUp(src.cols, block.x));

            Op<S> op;
            reduceRows<Op<S>, T, S, D><<<grid, block, 0, stream>>>(src, dst.data, op);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );

        }

        template <typename T, typename S, typename D> void reduceRows_gpu(const DevMem2Db& src, const DevMem2Db& dst, int reduceOp, hipStream_t stream)
        {
            typedef void (*caller_t)(const DevMem2D_<T>& src, DevMem2D_<D> dst, hipStream_t stream);

            static const caller_t callers[] = 
            {
                reduceRows_caller<SumReductor, T, S, D>, 
                reduceRows_caller<AvgReductor, T, S, D>, 
                reduceRows_caller<MaxReductor, T, S, D>, 
                reduceRows_caller<MinReductor, T, S, D>
            };

            callers[reduceOp](static_cast< DevMem2D_<T> >(src), static_cast< DevMem2D_<D> >(dst), stream);
        }

        template void reduceRows_gpu<uchar, int, uchar>(const DevMem2Db& src, const DevMem2Db& dst, int reduceOp, hipStream_t stream);
        template void reduceRows_gpu<uchar, int, int>(const DevMem2Db& src, const DevMem2Db& dst, int reduceOp, hipStream_t stream);
        template void reduceRows_gpu<uchar, int, float>(const DevMem2Db& src, const DevMem2Db& dst, int reduceOp, hipStream_t stream);  

        template void reduceRows_gpu<ushort, int, ushort>(const DevMem2Db& src, const DevMem2Db& dst, int reduceOp, hipStream_t stream);
        template void reduceRows_gpu<ushort, int, int>(const DevMem2Db& src, const DevMem2Db& dst, int reduceOp, hipStream_t stream);
        template void reduceRows_gpu<ushort, int, float>(const DevMem2Db& src, const DevMem2Db& dst, int reduceOp, hipStream_t stream); 

        template void reduceRows_gpu<short, int, short>(const DevMem2Db& src, const DevMem2Db& dst, int reduceOp, hipStream_t stream);
        template void reduceRows_gpu<short, int, int>(const DevMem2Db& src, const DevMem2Db& dst, int reduceOp, hipStream_t stream);
        template void reduceRows_gpu<short, int, float>(const DevMem2Db& src, const DevMem2Db& dst, int reduceOp, hipStream_t stream); 

        template void reduceRows_gpu<int, int, int>(const DevMem2Db& src, const DevMem2Db& dst, int reduceOp, hipStream_t stream);
        template void reduceRows_gpu<int, int, float>(const DevMem2Db& src, const DevMem2Db& dst, int reduceOp, hipStream_t stream);

        template void reduceRows_gpu<float, float, float>(const DevMem2Db& src, const DevMem2Db& dst, int reduceOp, hipStream_t stream);



        template <int cn, class Op, typename T, typename S, typename D> __global__ void reduceCols(const DevMem2D_<T> src, D* dst, const Op op)
        {
            __shared__ S smem[256 * cn];

            const int y = blockIdx.x;

            const T* src_row = src.ptr(y);

            S myVal[cn];

            #pragma unroll
            for (int c = 0; c < cn; ++c)
                myVal[c] = op.startValue();

        #if __CUDA_ARCH__ >= 200

            // For cc >= 2.0 prefer L1 cache
            for (int x = threadIdx.x; x < src.cols; x += 256)
            {
                #pragma unroll
                for (int c = 0; c < cn; ++c)
                    myVal[c] = op(myVal[c], src_row[x * cn + c]);
            }

        #else // __CUDA_ARCH__ >= 200

            // For older arch use shared memory for cache
            for (int x = 0; x < src.cols; x += 256)
            {
                #pragma unroll
                for (int c = 0; c < cn; ++c)
                {
                    smem[c * 256 + threadIdx.x] = op.startValue();
                    const int load_x = x * cn + c * 256 + threadIdx.x;
                    if (load_x < src.cols * cn)
                        smem[c * 256 + threadIdx.x] = src_row[load_x];
                }
                __syncthreads();

                #pragma unroll
                for (int c = 0; c < cn; ++c)
                    myVal[c] = op(myVal[c], smem[threadIdx.x * cn + c]);
                __syncthreads();
            }

        #endif // __CUDA_ARCH__ >= 200

            #pragma unroll
            for (int c = 0; c < cn; ++c)
                smem[c * 256 + threadIdx.x] = myVal[c];
            __syncthreads();

            if (threadIdx.x < 128)
            {
                #pragma unroll
                for (int c = 0; c < cn; ++c)
                    smem[c * 256 + threadIdx.x] = op(smem[c * 256 + threadIdx.x], smem[c * 256 + threadIdx.x + 128]);
            }
            __syncthreads();

            if (threadIdx.x < 64)
            {
                #pragma unroll
                for (int c = 0; c < cn; ++c)
                    smem[c * 256 + threadIdx.x] = op(smem[c * 256 + threadIdx.x], smem[c * 256 + threadIdx.x + 64]);
            }
            __syncthreads();

            volatile S* sdata = smem;

            if (threadIdx.x < 32)
            {
                #pragma unroll
                for (int c = 0; c < cn; ++c)
                {
                    sdata[c * 256 + threadIdx.x] = op(sdata[c * 256 + threadIdx.x], sdata[c * 256 + threadIdx.x + 32]);
                    sdata[c * 256 + threadIdx.x] = op(sdata[c * 256 + threadIdx.x], sdata[c * 256 + threadIdx.x + 16]);
                    sdata[c * 256 + threadIdx.x] = op(sdata[c * 256 + threadIdx.x], sdata[c * 256 + threadIdx.x + 8]);
                    sdata[c * 256 + threadIdx.x] = op(sdata[c * 256 + threadIdx.x], sdata[c * 256 + threadIdx.x + 4]);
                    sdata[c * 256 + threadIdx.x] = op(sdata[c * 256 + threadIdx.x], sdata[c * 256 + threadIdx.x + 2]);
                    sdata[c * 256 + threadIdx.x] = op(sdata[c * 256 + threadIdx.x], sdata[c * 256 + threadIdx.x + 1]);
                }
            }
            __syncthreads();

            if (threadIdx.x < cn)
                dst[y * cn + threadIdx.x] = saturate_cast<D>(op.result(smem[threadIdx.x * 256], src.cols));
        }

        template <int cn, template <typename> class Op, typename T, typename S, typename D> void reduceCols_caller(const DevMem2D_<T>& src, DevMem2D_<D> dst, hipStream_t stream)
        {
            const dim3 block(256);
            const dim3 grid(src.rows);

            Op<S> op;
            reduceCols<cn, Op<S>, T, S, D><<<grid, block, 0, stream>>>(src, dst.data, op);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );

        }

        template <typename T, typename S, typename D> void reduceCols_gpu(const DevMem2Db& src, int cn, const DevMem2Db& dst, int reduceOp, hipStream_t stream)
        {
            typedef void (*caller_t)(const DevMem2D_<T>& src, DevMem2D_<D> dst, hipStream_t stream);

            static const caller_t callers[4][4] = 
            {
                {reduceCols_caller<1, SumReductor, T, S, D>, reduceCols_caller<1, AvgReductor, T, S, D>, reduceCols_caller<1, MaxReductor, T, S, D>, reduceCols_caller<1, MinReductor, T, S, D>},
                {reduceCols_caller<2, SumReductor, T, S, D>, reduceCols_caller<2, AvgReductor, T, S, D>, reduceCols_caller<2, MaxReductor, T, S, D>, reduceCols_caller<2, MinReductor, T, S, D>},
                {reduceCols_caller<3, SumReductor, T, S, D>, reduceCols_caller<3, AvgReductor, T, S, D>, reduceCols_caller<3, MaxReductor, T, S, D>, reduceCols_caller<3, MinReductor, T, S, D>},
                {reduceCols_caller<4, SumReductor, T, S, D>, reduceCols_caller<4, AvgReductor, T, S, D>, reduceCols_caller<4, MaxReductor, T, S, D>, reduceCols_caller<4, MinReductor, T, S, D>},
            };

            callers[cn - 1][reduceOp](static_cast< DevMem2D_<T> >(src), static_cast< DevMem2D_<D> >(dst), stream);
        }

        template void reduceCols_gpu<uchar, int, uchar>(const DevMem2Db& src, int cn, const DevMem2Db& dst, int reduceOp, hipStream_t stream);
        template void reduceCols_gpu<uchar, int, int>(const DevMem2Db& src, int cn, const DevMem2Db& dst, int reduceOp, hipStream_t stream);
        template void reduceCols_gpu<uchar, int, float>(const DevMem2Db& src, int cn, const DevMem2Db& dst, int reduceOp, hipStream_t stream);

        template void reduceCols_gpu<ushort, int, ushort>(const DevMem2Db& src, int cn, const DevMem2Db& dst, int reduceOp, hipStream_t stream); 
        template void reduceCols_gpu<ushort, int, int>(const DevMem2Db& src, int cn, const DevMem2Db& dst, int reduceOp, hipStream_t stream);                  
        template void reduceCols_gpu<ushort, int, float>(const DevMem2Db& src, int cn, const DevMem2Db& dst, int reduceOp, hipStream_t stream);

        template void reduceCols_gpu<short, int, short>(const DevMem2Db& src, int cn, const DevMem2Db& dst, int reduceOp, hipStream_t stream);  
        template void reduceCols_gpu<short, int, int>(const DevMem2Db& src, int cn, const DevMem2Db& dst, int reduceOp, hipStream_t stream);                  
        template void reduceCols_gpu<short, int, float>(const DevMem2Db& src, int cn, const DevMem2Db& dst, int reduceOp, hipStream_t stream);  

        template void reduceCols_gpu<int, int, int>(const DevMem2Db& src, int cn, const DevMem2Db& dst, int reduceOp, hipStream_t stream);                  
        template void reduceCols_gpu<int, int, float>(const DevMem2Db& src, int cn, const DevMem2Db& dst, int reduceOp, hipStream_t stream);

        template void reduceCols_gpu<float, float, float>(const DevMem2Db& src, int cn, const DevMem2Db& dst, int reduceOp, hipStream_t stream);
    } // namespace mattrix_reductions
}}} // namespace cv { namespace gpu { namespace device
