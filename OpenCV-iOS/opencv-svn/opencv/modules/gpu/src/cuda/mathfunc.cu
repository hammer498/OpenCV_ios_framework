#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"

namespace cv { namespace gpu { namespace device 
{
    namespace mathfunc 
    {
        //////////////////////////////////////////////////////////////////////////////////////
        // Cart <-> Polar

        struct Nothing
        {
            static __device__ __forceinline__ void calc(int, int, float, float, float*, size_t, float)
            {
            }
        };
        struct Magnitude
        {
            static __device__ __forceinline__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float)
            {
                dst[y * dst_step + x] = ::sqrtf(x_data * x_data + y_data * y_data);
            }
        };
        struct MagnitudeSqr
        {
            static __device__ __forceinline__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float)
            {
                dst[y * dst_step + x] = x_data * x_data + y_data * y_data;
            }
        };
        struct Atan2
        {
            static __device__ __forceinline__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float scale)
            {
                float angle = ::atan2f(y_data, x_data);
                angle += (angle < 0) * 2.0 * CV_PI;
                dst[y * dst_step + x] = scale * angle;
            }
        };
        template <typename Mag, typename Angle>
        __global__ void cartToPolar(const float* xptr, size_t x_step, const float* yptr, size_t y_step, 
                                    float* mag, size_t mag_step, float* angle, size_t angle_step, float scale, int width, int height)
        {
	        const int x = blockDim.x * blockIdx.x + threadIdx.x;
	        const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < width && y < height)
            {
                float x_data = xptr[y * x_step + x];
                float y_data = yptr[y * y_step + x];

                Mag::calc(x, y, x_data, y_data, mag, mag_step, scale);
                Angle::calc(x, y, x_data, y_data, angle, angle_step, scale);
            }
        }

        struct NonEmptyMag
        {
            static __device__ __forceinline__ float get(const float* mag, size_t mag_step, int x, int y)
            {
                return mag[y * mag_step + x];
            }
        };
        struct EmptyMag
        {
            static __device__ __forceinline__ float get(const float*, size_t, int, int)
            {
                return 1.0f;
            }
        };
        template <typename Mag>
        __global__ void polarToCart(const float* mag, size_t mag_step, const float* angle, size_t angle_step, float scale,
            float* xptr, size_t x_step, float* yptr, size_t y_step, int width, int height)
        {
	        const int x = blockDim.x * blockIdx.x + threadIdx.x;
	        const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < width && y < height)
            {
                float mag_data = Mag::get(mag, mag_step, x, y);
                float angle_data = angle[y * angle_step + x];
                float sin_a, cos_a;

                ::sincosf(scale * angle_data, &sin_a, &cos_a);

                xptr[y * x_step + x] = mag_data * cos_a;
                yptr[y * y_step + x] = mag_data * sin_a;
            }
        }

        template <typename Mag, typename Angle>
        void cartToPolar_caller(DevMem2Df x, DevMem2Df y, DevMem2Df mag, DevMem2Df angle, bool angleInDegrees, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(x.cols, threads.x);
            grid.y = divUp(x.rows, threads.y);
            
            const float scale = angleInDegrees ? (float)(180.0f / CV_PI) : 1.f;

            cartToPolar<Mag, Angle><<<grid, threads, 0, stream>>>(
                x.data, x.step/x.elemSize(), y.data, y.step/y.elemSize(), 
                mag.data, mag.step/mag.elemSize(), angle.data, angle.step/angle.elemSize(), scale, x.cols, x.rows);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void cartToPolar_gpu(DevMem2Df x, DevMem2Df y, DevMem2Df mag, bool magSqr, DevMem2Df angle, bool angleInDegrees, hipStream_t stream)
        {
            typedef void (*caller_t)(DevMem2Df x, DevMem2Df y, DevMem2Df mag, DevMem2Df angle, bool angleInDegrees, hipStream_t stream);
            static const caller_t callers[2][2][2] = 
            {
                {
                    {
                        cartToPolar_caller<Magnitude, Atan2>,
                        cartToPolar_caller<Magnitude, Nothing>
                    },
                    {
                        cartToPolar_caller<MagnitudeSqr, Atan2>,
                        cartToPolar_caller<MagnitudeSqr, Nothing>,
                    }
                },
                {
                    {
                        cartToPolar_caller<Nothing, Atan2>,
                        cartToPolar_caller<Nothing, Nothing>
                    },
                    {
                        cartToPolar_caller<Nothing, Atan2>,
                        cartToPolar_caller<Nothing, Nothing>,
                    }
                }
            };

            callers[mag.data == 0][magSqr][angle.data == 0](x, y, mag, angle, angleInDegrees, stream);
        }

        template <typename Mag>
        void polarToCart_caller(DevMem2Df mag, DevMem2Df angle, DevMem2Df x, DevMem2Df y, bool angleInDegrees, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(mag.cols, threads.x);
            grid.y = divUp(mag.rows, threads.y);
            
            const float scale = angleInDegrees ? (float)(CV_PI / 180.0f) : 1.0f;

            polarToCart<Mag><<<grid, threads, 0, stream>>>(mag.data, mag.step/mag.elemSize(), 
                angle.data, angle.step/angle.elemSize(), scale, x.data, x.step/x.elemSize(), y.data, y.step/y.elemSize(), mag.cols, mag.rows);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void polarToCart_gpu(DevMem2Df mag, DevMem2Df angle, DevMem2Df x, DevMem2Df y, bool angleInDegrees, hipStream_t stream)
        {
            typedef void (*caller_t)(DevMem2Df mag, DevMem2Df angle, DevMem2Df x, DevMem2Df y, bool angleInDegrees, hipStream_t stream);
            static const caller_t callers[2] = 
            {
                polarToCart_caller<NonEmptyMag>,
                polarToCart_caller<EmptyMag>
            };

            callers[mag.data == 0](mag, angle, x, y, angleInDegrees, stream);
        }
    } // namespace mathfunc
}}} // namespace cv { namespace gpu { namespace device
