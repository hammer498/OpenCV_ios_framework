#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/gpu/device/saturate_cast.hpp"
#include "opencv2/gpu/device/transform.hpp"
#include "opencv2/gpu/device/functional.hpp"

namespace cv { namespace gpu { namespace device 
{
    template <typename T> struct shift_and_sizeof;
    template <> struct shift_and_sizeof<signed char> { enum { shift = 0 }; };
    template <> struct shift_and_sizeof<unsigned char> { enum { shift = 0 }; };
    template <> struct shift_and_sizeof<short> { enum { shift = 1 }; };
    template <> struct shift_and_sizeof<unsigned short> { enum { shift = 1 }; };
    template <> struct shift_and_sizeof<int> { enum { shift = 2 }; };
    template <> struct shift_and_sizeof<float> { enum { shift = 2 }; };
    template <> struct shift_and_sizeof<double> { enum { shift = 3 }; };

    ///////////////////////////////////////////////////////////////////////////
    ////////////////////////////////// CopyTo /////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////

    template <typename T> void copyToWithMask(DevMem2Db src, DevMem2Db dst, int cn, DevMem2Db mask, bool colorMask, hipStream_t stream)
    {
        if (colorMask)
            cv::gpu::device::transform((DevMem2D_<T>)src, (DevMem2D_<T>)dst, identity<T>(), SingleMask(mask), stream);
        else
            cv::gpu::device::transform((DevMem2D_<T>)src, (DevMem2D_<T>)dst, identity<T>(), SingleMaskChannels(mask, cn), stream);
    }

    void copyToWithMask_gpu(DevMem2Db src, DevMem2Db dst, size_t elemSize1, int cn, DevMem2Db mask, bool colorMask, hipStream_t stream)
    {
        typedef void (*func_t)(DevMem2Db src, DevMem2Db dst, int cn, DevMem2Db mask, bool colorMask, hipStream_t stream);

        static func_t tab[] =
        {
            0,
            copyToWithMask<unsigned char>,
            copyToWithMask<unsigned short>,
            0,
            copyToWithMask<int>,
            0,
            0,
            0,
            copyToWithMask<double>
        };

        tab[elemSize1](src, dst, cn, mask, colorMask, stream);
    }

    ///////////////////////////////////////////////////////////////////////////
    ////////////////////////////////// SetTo //////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////

    __constant__ uchar scalar_8u[4];
    __constant__ schar scalar_8s[4];
    __constant__ ushort scalar_16u[4];
    __constant__ short scalar_16s[4];
    __constant__ int scalar_32s[4];
    __constant__ float scalar_32f[4]; 
    __constant__ double scalar_64f[4];

    template <typename T> __device__ __forceinline__ T readScalar(int i);
    template <> __device__ __forceinline__ uchar readScalar<uchar>(int i) {return scalar_8u[i];}
    template <> __device__ __forceinline__ schar readScalar<schar>(int i) {return scalar_8s[i];}
    template <> __device__ __forceinline__ ushort readScalar<ushort>(int i) {return scalar_16u[i];}
    template <> __device__ __forceinline__ short readScalar<short>(int i) {return scalar_16s[i];}
    template <> __device__ __forceinline__ int readScalar<int>(int i) {return scalar_32s[i];}
    template <> __device__ __forceinline__ float readScalar<float>(int i) {return scalar_32f[i];}
    template <> __device__ __forceinline__ double readScalar<double>(int i) {return scalar_64f[i];}

    void writeScalar(const uchar* vals)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_8u), vals, sizeof(uchar) * 4) );
    }
    void writeScalar(const schar* vals)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_8s), vals, sizeof(schar) * 4) );
    }
    void writeScalar(const ushort* vals)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_16u), vals, sizeof(ushort) * 4) );
    }
    void writeScalar(const short* vals)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_16s), vals, sizeof(short) * 4) );
    }
    void writeScalar(const int* vals)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_32s), vals, sizeof(int) * 4) );
    }
    void writeScalar(const float* vals)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_32f), vals, sizeof(float) * 4) );
    }
    void writeScalar(const double* vals)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_64f), vals, sizeof(double) * 4) );
    }

    template<typename T>
    __global__ void set_to_without_mask(T* mat, int cols, int rows, size_t step, int channels)
    {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;

        if ((x < cols * channels ) && (y < rows))
        {
            size_t idx = y * ( step >> shift_and_sizeof<T>::shift ) + x;
            mat[idx] = readScalar<T>(x % channels);
        }
    }

    template<typename T>
    __global__ void set_to_with_mask(T* mat, const uchar* mask, int cols, int rows, size_t step, int channels, size_t step_mask)
    {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;

        if ((x < cols * channels ) && (y < rows))
            if (mask[y * step_mask + x / channels] != 0)
            {
                size_t idx = y * ( step >> shift_and_sizeof<T>::shift ) + x;
                mat[idx] = readScalar<T>(x % channels);
            }
    }
    template <typename T>
    void set_to_gpu(DevMem2Db mat, const T* scalar, DevMem2Db mask, int channels, hipStream_t stream)
    {
        writeScalar(scalar);

        dim3 threadsPerBlock(32, 8, 1);
        dim3 numBlocks (mat.cols * channels / threadsPerBlock.x + 1, mat.rows / threadsPerBlock.y + 1, 1);

        set_to_with_mask<T><<<numBlocks, threadsPerBlock, 0, stream>>>((T*)mat.data, (uchar*)mask.data, mat.cols, mat.rows, mat.step, channels, mask.step);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall ( hipDeviceSynchronize() );
    }

    template void set_to_gpu<uchar >(DevMem2Db mat, const uchar*  scalar, DevMem2Db mask, int channels, hipStream_t stream);
    template void set_to_gpu<schar >(DevMem2Db mat, const schar*  scalar, DevMem2Db mask, int channels, hipStream_t stream);
    template void set_to_gpu<ushort>(DevMem2Db mat, const ushort* scalar, DevMem2Db mask, int channels, hipStream_t stream);
    template void set_to_gpu<short >(DevMem2Db mat, const short*  scalar, DevMem2Db mask, int channels, hipStream_t stream);
    template void set_to_gpu<int   >(DevMem2Db mat, const int*    scalar, DevMem2Db mask, int channels, hipStream_t stream);
    template void set_to_gpu<float >(DevMem2Db mat, const float*  scalar, DevMem2Db mask, int channels, hipStream_t stream);
    template void set_to_gpu<double>(DevMem2Db mat, const double* scalar, DevMem2Db mask, int channels, hipStream_t stream);

    template <typename T>
    void set_to_gpu(DevMem2Db mat, const T* scalar, int channels, hipStream_t stream)
    {
        writeScalar(scalar);

        dim3 threadsPerBlock(32, 8, 1);
        dim3 numBlocks (mat.cols * channels / threadsPerBlock.x + 1, mat.rows / threadsPerBlock.y + 1, 1);

        set_to_without_mask<T><<<numBlocks, threadsPerBlock, 0, stream>>>((T*)mat.data, mat.cols, mat.rows, mat.step, channels);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall ( hipDeviceSynchronize() );
    }

    template void set_to_gpu<uchar >(DevMem2Db mat, const uchar*  scalar, int channels, hipStream_t stream);
    template void set_to_gpu<schar >(DevMem2Db mat, const schar*  scalar, int channels, hipStream_t stream);
    template void set_to_gpu<ushort>(DevMem2Db mat, const ushort* scalar, int channels, hipStream_t stream);
    template void set_to_gpu<short >(DevMem2Db mat, const short*  scalar, int channels, hipStream_t stream);
    template void set_to_gpu<int   >(DevMem2Db mat, const int*    scalar, int channels, hipStream_t stream);
    template void set_to_gpu<float >(DevMem2Db mat, const float*  scalar, int channels, hipStream_t stream);
    template void set_to_gpu<double>(DevMem2Db mat, const double* scalar, int channels, hipStream_t stream);

    ///////////////////////////////////////////////////////////////////////////
    //////////////////////////////// ConvertTo ////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////

    template <typename T, typename D> struct Convertor : unary_function<T, D>
    {
        Convertor(double alpha_, double beta_) : alpha(alpha_), beta(beta_) {}

        __device__ __forceinline__ D operator()(const T& src) const
        {
            return saturate_cast<D>(alpha * src + beta);
        }

        double alpha, beta;
    };

    namespace detail
    {
        template <size_t src_size, size_t dst_size, typename F> struct ConvertTraitsDispatcher : DefaultTransformFunctorTraits<F>
        {
        };
        template <typename F> struct ConvertTraitsDispatcher<1, 1, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_shift = 8 };
        };
        template <typename F> struct ConvertTraitsDispatcher<1, 2, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_shift = 4 };
        };
        template <typename F> struct ConvertTraitsDispatcher<1, 4, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_block_dim_y = 8 };
            enum { smart_shift = 4 };
        };

        template <typename F> struct ConvertTraitsDispatcher<2, 2, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_shift = 4 };
        };
        template <typename F> struct ConvertTraitsDispatcher<2, 4, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_shift = 2 };
        };

        template <typename F> struct ConvertTraitsDispatcher<4, 2, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_block_dim_y = 8 };
            enum { smart_shift = 4 };
        };
        template <typename F> struct ConvertTraitsDispatcher<4, 4, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_block_dim_y = 8 };
            enum { smart_shift = 2 };
        };

        template <typename F> struct ConvertTraits : ConvertTraitsDispatcher<sizeof(typename F::argument_type), sizeof(typename F::result_type), F>
        {
        };
    }

    template <typename T, typename D> struct TransformFunctorTraits< Convertor<T, D> > : detail::ConvertTraits< Convertor<T, D> >
    {
    };
        
    template<typename T, typename D>
    void cvt_(DevMem2Db src, DevMem2Db dst, double alpha, double beta, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&alpha) );
        cudaSafeCall( cudaSetDoubleForDevice(&beta) );
        Convertor<T, D> op(alpha, beta);
        cv::gpu::device::transform((DevMem2D_<T>)src, (DevMem2D_<D>)dst, op, WithOutMask(), stream);
    }

    void convert_gpu(DevMem2Db src, int sdepth, DevMem2Db dst, int ddepth, double alpha, double beta, hipStream_t stream)
    {
        typedef void (*caller_t)(DevMem2Db src, DevMem2Db dst, double alpha, double beta, hipStream_t stream);

        static const caller_t tab[8][8] =
        {
            {cvt_<uchar, uchar>, cvt_<uchar, schar>, cvt_<uchar, ushort>, cvt_<uchar, short>,
            cvt_<uchar, int>, cvt_<uchar, float>, cvt_<uchar, double>, 0},

            {cvt_<schar, uchar>, cvt_<schar, schar>, cvt_<schar, ushort>, cvt_<schar, short>,
            cvt_<schar, int>, cvt_<schar, float>, cvt_<schar, double>, 0},

            {cvt_<ushort, uchar>, cvt_<ushort, schar>, cvt_<ushort, ushort>, cvt_<ushort, short>,
            cvt_<ushort, int>, cvt_<ushort, float>, cvt_<ushort, double>, 0},

            {cvt_<short, uchar>, cvt_<short, schar>, cvt_<short, ushort>, cvt_<short, short>,
            cvt_<short, int>, cvt_<short, float>, cvt_<short, double>, 0},

            {cvt_<int, uchar>, cvt_<int, schar>, cvt_<int, ushort>,
            cvt_<int, short>, cvt_<int, int>, cvt_<int, float>, cvt_<int, double>, 0},

            {cvt_<float, uchar>, cvt_<float, schar>, cvt_<float, ushort>,
            cvt_<float, short>, cvt_<float, int>, cvt_<float, float>, cvt_<float, double>, 0},

            {cvt_<double, uchar>, cvt_<double, schar>, cvt_<double, ushort>,
            cvt_<double, short>, cvt_<double, int>, cvt_<double, float>, cvt_<double, double>, 0},

            {0,0,0,0,0,0,0,0}
        };

        caller_t func = tab[sdepth][ddepth];
        if (!func)
            cv::gpu::error("Unsupported convert operation", __FILE__, __LINE__, "convert_gpu");

        func(src, dst, alpha, beta, stream);
    }
}}} // namespace cv { namespace gpu { namespace device
